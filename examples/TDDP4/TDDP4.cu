#include "hip/hip_runtime.h"
// ROOT
#include <TFile.h>
#include <TTree.h>

// GooFit stuff
#include <fstream>
#include <goofit/Application.h>
#include <goofit/PDFs/basic/PolynomialPdf.h>
#include <goofit/PDFs/combine/AddPdf.h>
#include <goofit/PDFs/physics/DP4Pdf.h>
#include <goofit/PDFs/physics/Tddp4Pdf.h>
#include <goofit/PDFs/physics/TruthResolution_Aux.h>
#include <goofit/UnbinnedDataSet.h>
#include <goofit/Variable.h>
#include <thrust/count.h>

using namespace std;
using namespace GooFit;

// Constants used in more than one PDF component.
const fptype _mD0       = 1.8645;
const fptype piPlusMass = 0.13957018;
const fptype KmMass     = .493677;

int main(int argc, char **argv) {
    GooFit::Application app("Time dependent Dalitz plot, 4 particles", argc, argv);

    TString output = "test_10_15.output";
    app.add_option("-o,--output,output", output, "File to output", true)->check(GooFit::NonexistentPath);

    int trials = 100;
    app.add_option("-t,--trials,output", trials, "Number of trials", true);

    GOOFIT_PARSE(app);

    DecayInfo4t DK3P_DI{Variable("tau", 0.4101, 0.001, 0.300, 0.500),
                        Variable("xmixing", 0.005, 0.001, 0, 0),
                        Variable("ymixing", 0.01, 0.001, 0, 0),
                        Variable("SqWStoRSrate", 1.0 / sqrt(300.0))};

    DK3P_DI.meson_radius = 1.5;
    DK3P_DI.particle_masses.push_back(_mD0);
    DK3P_DI.particle_masses.push_back(piPlusMass);
    DK3P_DI.particle_masses.push_back(piPlusMass);
    DK3P_DI.particle_masses.push_back(KmMass);
    DK3P_DI.particle_masses.push_back(piPlusMass);

    Variable RhoMass{"rho_mass", 0.77526, 0.01, 0.7, 0.8};
    Variable RhoWidth{"rho_width", 0.1478, 0.01, 0.1, 0.2};
    Variable KstarM{"KstarM", 0.89581, 0.01, 0.9, 0.1};
    Variable KstarW{"KstarW", 0.0474, 0.01, 0.1, 0.2};

    // Variable* f600M  = new Variable("f600M", 0.519, 0.01, 0.75, 0.85);
    // Variable* f600W  = new Variable("f600W", 0.454, 0.01, 0.75, 0.85);
    // Variable* a1M  = new Variable("a1M", 1.23, 0.01, 1.2, 1.3);
    // Variable* a1W  = new Variable("a1W", 0.42, 0.01, 0.37, 0.47);
    // Variable* K1M  = new Variable("K1M", 1.272, 0.01, 1.2, 1.3);
    // Variable* K1W  = new Variable("K1W", 0.09, 0.01, 0.08, 0.1);
    // Variable* K1430M  = new Variable("K1430M", 1.414, 0.01, 1.4, 1.5);
    // Variable* K1430W  = new Variable("K1430W", .29, 0.01, 0.25, 0.35);

    // Spin factors: we have two due to the bose symmetrization of the two pi+
    std::vector<SpinFactor *> SFKRS = {new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_S, _mD0, 0, 1, 2, 3),
                                       new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_S, _mD0, 3, 1, 2, 0)};

    std::vector<SpinFactor *> SFKRP;
    SFKRP.push_back(new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_P, _mD0, 0, 1, 2, 3));
    SFKRP.push_back(new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_P, _mD0, 3, 1, 2, 0));

    std::vector<SpinFactor *> SFKRD;
    SFKRD.push_back(new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_D, _mD0, 0, 1, 2, 3));
    SFKRD.push_back(new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_D, _mD0, 3, 1, 2, 0));

    std::vector<SpinFactor *> SFKF;
    SFKF.push_back(new SpinFactor("SF", SF_4Body::DtoVS_VtoP1P2_StoP3P4, _mD0, 2, 3, 0, 1));
    SFKF.push_back(new SpinFactor("SF", SF_4Body::DtoVS_VtoP1P2_StoP3P4, _mD0, 2, 0, 3, 1));

    std::vector<SpinFactor *> SFKK;
    SFKK.push_back(new SpinFactor("SF", SF_4Body::DtoAP1_AtoSP2_StoP3P4, _mD0, 0, 1, 3, 2));
    SFKK.push_back(new SpinFactor("SF", SF_4Body::DtoAP1_AtoSP2_StoP3P4, _mD0, 3, 1, 0, 2));

    std::vector<SpinFactor *> SFK1R;
    SFK1R.push_back(new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2_VtoP3P4, _mD0, 3, 2, 0, 1));
    SFK1R.push_back(new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2_VtoP3P4, _mD0, 0, 2, 3, 1));

    std::vector<SpinFactor *> SFA1R;
    SFA1R.push_back(new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2_VtoP3P4, _mD0, 2, 3, 0, 1));
    SFA1R.push_back(new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2_VtoP3P4, _mD0, 2, 0, 3, 1));

    std::vector<SpinFactor *> SFA1RD;
    SFA1RD.push_back(new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2Dwave_VtoP3P4, _mD0, 2, 3, 0, 1));
    SFA1RD.push_back(new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2Dwave_VtoP3P4, _mD0, 2, 0, 3, 1));

    // Lineshapes, also for both pi+ configurations
    std::vector<Lineshape *> LSKRS = {new Lineshapes::RBW("rho(770)", RhoMass, RhoWidth, 1, M_12, FF::BL2),
                                      new Lineshapes::RBW("K*(892)bar", KstarM, KstarW, 1, M_34, FF::BL2),
                                      new Lineshapes::RBW("rho(770)", RhoMass, RhoWidth, 1, M_24, FF::BL2),
                                      new Lineshapes::RBW("K*(892)bar", KstarM, KstarW, 1, M_13, FF::BL2)};

    std::vector<Lineshape *> LSKRP = {new Lineshapes::RBW("rho(770)", RhoMass, RhoWidth, 1, M_12, FF::BL2),
                                      new Lineshapes::RBW("K*(892)bar", KstarM, KstarW, 1, M_34, FF::BL2),
                                      new Lineshapes::RBW("rho(770)", RhoMass, RhoWidth, 1, M_24, FF::BL2),
                                      new Lineshapes::RBW("K*(892)bar", KstarM, KstarW, 1, M_13, FF::BL2)};

    std::vector<Lineshape *> LSKRD = {new Lineshapes::RBW("rho(770)", RhoMass, RhoWidth, 1, M_12, FF::BL2),
                                      new Lineshapes::RBW("K*(892)bar", KstarM, KstarW, 1, M_34, FF::BL2),
                                      new Lineshapes::RBW("rho(770)", RhoMass, RhoWidth, 1, M_24, FF::BL2),
                                      new Lineshapes::RBW("K*(892)bar", KstarM, KstarW, 1, M_13, FF::BL2)};

    // the very last parameter means that we have two permutations. so the first half of the Lineshapes
    // and the first half of the spinfactors are amplitude 1, rest is amplitude 2
    // This means that it is important for symmetrized amplitudes that the spinfactors and lineshapes are in the "right"
    // order

    Amplitude Bose_symmetrized_AMP_S{
        "K*(892)rho(770)_S", Variable("amp_real1", 1.0), Variable("amp_imag1", 0.0), LSKRS, SFKRS, 2};
    Amplitude Bose_symmetrized_AMP_P{
        "K*(892)rho(770)_P", Variable("amp_real2", 0.526), Variable("amp_imag2", -0.626), LSKRP, SFKRP, 2};
    Amplitude Bose_symmetrized_AMP_D{
        "K*(892)rho(770)_D", Variable("amp_real3", 26.537), Variable("amp_imag3", 12.284), LSKRD, SFKRD, 2};

    Amplitude Bose_symmetrized_AMP_S_B{
        "B_K*(892)rho(770)_S", Variable("amp_real1", 1.0), Variable("amp_imag1", 0), LSKRS, SFKRS, 2};
    Amplitude Bose_symmetrized_AMP_P_B{
        "B_K*(892)rho(770)_P", Variable("amp_real2", -0.145), Variable("amp_imag2", 0.86), LSKRP, SFKRP, 2};
    Amplitude Bose_symmetrized_AMP_D_B{
        "B_K*(892)rho(770)_D", Variable("amp_real3", 24.343), Variable("amp_imag3", 5.329), LSKRD, SFKRD, 2};

    DK3P_DI.amplitudes_B.push_back(&Bose_symmetrized_AMP_S);
    DK3P_DI.amplitudes_B.push_back(&Bose_symmetrized_AMP_P);
    DK3P_DI.amplitudes_B.push_back(&Bose_symmetrized_AMP_D);

    DK3P_DI.amplitudes.push_back(&Bose_symmetrized_AMP_S_B);
    DK3P_DI.amplitudes.push_back(&Bose_symmetrized_AMP_P_B);
    DK3P_DI.amplitudes.push_back(&Bose_symmetrized_AMP_D_B);

    Observable m12{"m12", 0, 3};
    Observable m34{"m34", 0, 3};
    Observable cos12{"cos12", -1, 1};
    Observable cos34{"m12", -1, 1};
    Observable phi{"phi", -3.5, 3.5};
    EventNumber eventNumber{"eventNumber"};
    Observable dtime{"dtime", 0, 10};
    Observable sigmat{"sigmat", -3, 3};
    Variable constantOne{"constantOne", 1};
    Variable constantZero{"constantZero", 0};

    vector<Observable> observables{m12, m34, cos12, cos34, phi, eventNumber, dtime, sigmat};
    vector<Variable> offsets{constantZero, constantZero};
    vector<Variable> coefficients{constantOne};

    TruthResolution dat;
    PolynomialPdf eff{"constantEff", observables, coefficients, offsets, 0};
    TDDP4 dp{"test", observables, DK3P_DI, &dat, &eff, 0, 1};

    TFile *file = new TFile(output, "RECREATE");
    TTree *tree = new TTree("events", "events");

    double tm12, tm34, tc12, tc34, tphi, tdtime, D0_E, D0_Px, D0_Py, D0_Pz, Kplus_E, Kplus_Px, Kplus_Py, Kplus_Pz,
        Piminus1_E, Piminus1_Px, Piminus1_Py, Piminus1_Pz, Piminus2_E, Piminus2_Px, Piminus2_Py, Piminus2_Pz, Piplus_E,
        Piplus_Px, Piplus_Py, Piplus_Pz;
    int D0_pdg, Kplus_pdg, Piminus1_pdg, Piminus2_pdg, Piplus_pdg;

    tree->Branch("m12", &tm12, "m12/D");
    tree->Branch("m34", &tm34, "m34/D");
    tree->Branch("c12", &tc12, "c12/D");
    tree->Branch("c34", &tc34, "c34/D");
    tree->Branch("phi", &tphi, "phi/D");
    tree->Branch("dtime", &tdtime, "dtime/D");
    tree->Branch("D0_E", &D0_E, "D0_E/D");
    tree->Branch("D0_Px", &D0_Px, "D0_Px/D");
    tree->Branch("D0_Py", &D0_Py, "D0_Py/D");
    tree->Branch("D0_Pz", &D0_Pz, "D0_Pz/D");
    tree->Branch("D0_pdg", &D0_pdg, "D0_pdg/I");
    tree->Branch("Kplus_E", &Kplus_E, "Kplus_E/D");
    tree->Branch("Kplus_Px", &Kplus_Px, "Kplus_Px/D");
    tree->Branch("Kplus_Py", &Kplus_Py, "Kplus_Py/D");
    tree->Branch("Kplus_Pz", &Kplus_Pz, "Kplus_Pz/D");
    tree->Branch("Kplus_pdg", &Kplus_pdg, "Kplus_pdg/I");
    tree->Branch("Piminus1_E", &Piminus1_E, "Piminus1_E/D");
    tree->Branch("Piminus1_Px", &Piminus1_Px, "Piminus1_Px/D");
    tree->Branch("Piminus1_Py", &Piminus1_Py, "Piminus1_Py/D");
    tree->Branch("Piminus1_Pz", &Piminus1_Pz, "Piminus1_Pz/D");
    tree->Branch("Piminus1_pdg", &Piminus1_pdg, "Piminus1_pdg/I");
    tree->Branch("Piminus2_E", &Piminus2_E, "Piminus2_E/D");
    tree->Branch("Piminus2_Px", &Piminus2_Px, "Piminus2_Px/D");
    tree->Branch("Piminus2_Py", &Piminus2_Py, "Piminus2_Py/D");
    tree->Branch("Piminus2_Pz", &Piminus2_Pz, "Piminus2_Pz/D");
    tree->Branch("Piminus2_pdg", &Piminus2_pdg, "Piminus2_pdg/I");
    tree->Branch("Piplus_E", &Piplus_E, "Piplus_E/D");
    tree->Branch("Piplus_Px", &Piplus_Px, "Piplus_Px/D");
    tree->Branch("Piplus_Py", &Piplus_Py, "Piplus_Py/D");
    tree->Branch("Piplus_Pz", &Piplus_Pz, "Piplus_Pz/D");
    tree->Branch("Piplus_pdg", &Piplus_pdg, "Piplus_pdg/I");

    int total_accepted = 0;

    for(int k = 0; k < trials; ++k) {
        int numEvents = 800000;
        dp.setGenerationOffset(k * numEvents);

        mcbooster::ParticlesSet_h particles; // typedef for std::vector<Particles_h *>
        mcbooster::VariableSet_h variables;
        mcbooster::RealVector_h weights;
        mcbooster::BoolVector_h flags;

        std::tie(particles, variables, weights, flags) = dp.GenerateSig(numEvents);

        int accepted = thrust::count_if(flags.begin(), flags.end(), thrust::identity<bool>());
        total_accepted += accepted;

        GOOFIT_INFO(
            "Run #{}: Using accept-reject method would leave you with {} out of {} events", k, accepted, numEvents);

        for(int i = 0; i < weights.size(); ++i) {
            if(flags[i] == 1) {
                // printf("%.5g %.5g %.5g %.5g %.5g %.5g %.5g\n", (*(variables[0]))[i], (*(variables[1]))[i],
                // (*(variables[2]))[i], (*(variables[3]))[i], (*(variables[4]))[i], weights[i], flags[i]);
                tm12         = (*(variables[0]))[i];
                tm34         = (*(variables[1]))[i];
                tc12         = (*(variables[2]))[i];
                tc34         = (*(variables[3]))[i];
                tphi         = (*(variables[4]))[i];
                tdtime       = (*(variables[5]))[i];
                D0_E         = 1864;
                D0_Px        = 0.0;
                D0_Py        = 0.0;
                D0_Pz        = 0.0;
                D0_pdg       = 421;
                Kplus_E      = 1000 * (*(particles[2]))[i].get(0);
                Kplus_Px     = 1000 * (*(particles[2]))[i].get(1);
                Kplus_Py     = 1000 * (*(particles[2]))[i].get(2);
                Kplus_Pz     = 1000 * (*(particles[2]))[i].get(3);
                Kplus_pdg    = -321;
                Piminus1_E   = 1000 * (*(particles[3]))[i].get(0);
                Piminus1_Px  = 1000 * (*(particles[3]))[i].get(1);
                Piminus1_Py  = 1000 * (*(particles[3]))[i].get(2);
                Piminus1_Pz  = 1000 * (*(particles[3]))[i].get(3);
                Piminus1_pdg = 211;
                Piminus2_E   = 1000 * (*(particles[0]))[i].get(0);
                Piminus2_Px  = 1000 * (*(particles[0]))[i].get(1);
                Piminus2_Py  = 1000 * (*(particles[0]))[i].get(2);
                Piminus2_Pz  = 1000 * (*(particles[0]))[i].get(3);
                Piminus2_pdg = 211;
                Piplus_E     = 1000 * (*(particles[1]))[i].get(0);
                Piplus_Px    = 1000 * (*(particles[1]))[i].get(1);
                Piplus_Py    = 1000 * (*(particles[1]))[i].get(2);
                Piplus_Pz    = 1000 * (*(particles[1]))[i].get(3);
                Piplus_pdg   = -211;

                tree->Fill();
            }
        }

        delete variables[0];
        delete variables[1];
        delete variables[2];
        delete variables[3];
        delete variables[4];
        delete variables[5];

        delete particles[0];
        delete particles[1];
        delete particles[2];
        delete particles[3];
    }

    tree->Write();
    file->Close();

    if(total_accepted > 0)
        return 0;
    else {
        GOOFIT_ERROR("Total accepted was 0! Something is wrong.");
        return 1;
    }
}
