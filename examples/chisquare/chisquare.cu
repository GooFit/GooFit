#include "hip/hip_runtime.h"
#include "goofit/Application.h"
#include "goofit/Variable.h"
#include "goofit/FitManager.h"
#include "goofit/BinnedDataSet.h"
#include "goofit/UnbinnedDataSet.h"
#include "goofit/FitControl.h"

#include "goofit/PDFs/basic/PolynomialPdf.h"
#include "TMinuit.h"
#include "TRandom.h"
#include "TH1F.h"
#include "TCanvas.h"
#include "TLatex.h"

#include <sys/time.h>
#include <sys/times.h>

TCanvas foo;
timeval startTime, stopTime, totalTime;

#include <vector>
#include <iostream>
#include <string>

using namespace std;
using namespace GooFit;

Variable *decayTime  = 0;
Variable *constaCoef = 0;
Variable *linearCoef = 0;
Variable *secondCoef = 0;

double integralExpCon(double lo, double hi) { return (exp(-lo) - exp(-hi)); }

double integralExpLin(double lo, double hi) { return ((lo + 1) * exp(-lo) - (hi + 1) * exp(-hi)); }

double integralExpSqu(double lo, double hi) {
    return ((lo * lo + 2 * lo + 2) * exp(-lo) - (hi * hi + 2 * hi + 2) * exp(-hi));
}

void generateEvents(vector<int> &rsEvtVec,
                    vector<int> &wsEvtVec,
                    Variable const *const decayTime,
                    double conCoef,
                    double linCoef,
                    double squCoef,
                    int eventsToGenerate) {
    static TRandom donram(24);
    double totalRSintegral = integralExpCon(0, 100);
    double step            = (decayTime->getUpperLimit() - decayTime->getLowerLimit()) / decayTime->getNumBins();

    for(int i = 0; i < decayTime->getNumBins(); ++i) {
        double binStart = i * step;
        binStart += decayTime->getLowerLimit();
        double binFinal = binStart + step;

        double rsIntegral = integralExpCon(binStart, binFinal);
        double wsIntegral = conCoef * integralExpCon(binStart, binFinal);
        wsIntegral += linCoef * integralExpLin(binStart, binFinal);
        wsIntegral += squCoef * integralExpSqu(binStart, binFinal);

        double expectedRSevts = eventsToGenerate * rsIntegral / totalRSintegral;
        double expectedWSevts = eventsToGenerate * wsIntegral / totalRSintegral;

        int rsEvts  = donram.Poisson(expectedRSevts);
        int wsEvts  = donram.Poisson(expectedWSevts);
        rsEvtVec[i] = rsEvts;
        wsEvtVec[i] = wsEvts;

        if(0 == (i % 10))
            std::cout << "Events in bin " << i << " : " << rsEvts << " (" << expectedRSevts << ") " << wsEvts << " ("
                      << expectedWSevts << ")\n";
    }
}

int fitRatio(vector<int> &rsEvts, vector<int> &wsEvts, std::string plotName = "") {
    TH1D *ratioHist
        = new TH1D("ratioHist", "", decayTime->getNumBins(), decayTime->getLowerLimit(), decayTime->getUpperLimit());

    BinnedDataSet *ratioData = new BinnedDataSet(decayTime);

    for(unsigned int i = 0; i < wsEvts.size(); ++i) {
        double ratio = wsEvts[i];

        if(0 == rsEvts[i])
            rsEvts[i] = 1; // Cheating to avoid div by zero.

        ratio /= rsEvts[i];

        if(0 == wsEvts[i])
            wsEvts[i] = 1; // Avoid zero errors

        double error = wsEvts[i] / pow(rsEvts[i], 2);
        error += pow(wsEvts[i], 2) / pow(rsEvts[i], 3);
        error = sqrt(error);

        ratioData->setBinContent(i, ratio);
        ratioData->setBinError(i, error);
        ratioHist->SetBinContent(i + 1, ratio);
        ratioHist->SetBinError(i + 1, error);
    }

    if(0 == constaCoef) {
        constaCoef = new Variable("constaCoef", 0.03, 0.01, -1, 1);
        constaCoef->setValue(0.03);
        constaCoef->setError(0.01);
        linearCoef = new Variable("linearCoef", 0, 0.01, -1, 1);
        linearCoef->setValue(0.00);
        linearCoef->setError(0.01);
        secondCoef = new Variable("secondCoef", 0, 0.01, -1, 1);
        secondCoef->setValue(0.00);
        secondCoef->setError(0.01);
    }

    vector<Variable *> weights;
    weights.push_back(constaCoef);
    weights.push_back(linearCoef);
    weights.push_back(secondCoef);

    PolynomialPdf *poly = new PolynomialPdf("poly", decayTime, weights);
    poly->setFitControl(new BinnedErrorFit());
    poly->setData(ratioData);
    FitManager datapdf{poly};

    gettimeofday(&startTime, nullptr);
    datapdf.fit();
    gettimeofday(&stopTime, nullptr);

    vector<fptype> values = poly->evaluateAtPoints(decayTime);
    TH1D pdfHist("pdfHist", "", decayTime->getNumBins(), decayTime->getLowerLimit(), decayTime->getUpperLimit());

    for(int i = 0; i < values.size(); ++i) {
        pdfHist.SetBinContent(i + 1, values[i]);
    }

    ratioHist->SetMarkerStyle(8);
    ratioHist->SetMarkerSize(0.5);
    ratioHist->SetStats(false);
    ratioHist->Draw("p");

    char strbuffer[1000];
    sprintf(
        strbuffer, "Constant [10^{-2}] : %.3f #pm %.3f", 1e2 * constaCoef->getValue(), constaCoef->getError() * 1e2);
    TLatex res1(0.14, 0.83, strbuffer);
    res1.SetNDC(true);
    sprintf(
        strbuffer, "Linear [10^{-4}]   : %.3f #pm %.3f", 1e4 * linearCoef->getValue(), linearCoef->getError() * 1e4);
    TLatex res2(0.14, 0.73, strbuffer);
    res2.SetNDC(true);
    sprintf(
        strbuffer, "Quadratic [10^{-6}]: %.3f #pm %.3f", 1e6 * secondCoef->getValue(), secondCoef->getError() * 1e6);
    TLatex res3(0.14, 0.63, strbuffer);
    res3.SetNDC(true);

    res1.Draw();
    res2.Draw();
    res3.Draw();

    pdfHist.SetLineColor(kBlue);
    pdfHist.SetLineWidth(3);
    pdfHist.SetStats(false);
    pdfHist.Draw("lsame");
    foo.SaveAs(plotName.c_str());

    std::cout << "Polynomial function: " << poly->getCoefficient(2) << " * t^2 + " << poly->getCoefficient(1)
              << " * t + " << poly->getCoefficient(0) << std::endl;

    delete ratioHist;
    delete ratioData;
    delete poly;

    return datapdf;
}

double dzero_con     = 0;
double dzero_lin     = 0;
double dzero_qua     = 0;
double dzero_con_err = 0;
double dzero_lin_err = 0;
double dzero_qua_err = 0;
double d0bar_con     = 0;
double d0bar_lin     = 0;
double d0bar_qua     = 0;
double d0bar_con_err = 0;
double d0bar_lin_err = 0;
double d0bar_qua_err = 0;

vector<double> ratios;
vector<double> errors;

void cpvFitFcn(int &npar, double *gin, double &fun, double *fp, int iflag) {
    double conCoef = fp[0];
    double linCoef = fp[1];
    double squCoef = fp[2];

    double chisq = 0;
    double step  = (decayTime->getUpperLimit() - decayTime->getLowerLimit()) / decayTime->getNumBins();

    for(unsigned int i = 0; i < ratios.size(); ++i) {
        double currDTime = decayTime->getLowerLimit() + (i + 0.5) * step;
        double pdfval    = conCoef + linCoef * currDTime + squCoef * currDTime * currDTime;
        chisq += pow((pdfval - ratios[i]) / errors[i], 2);
    }

    fun = chisq;
}

void fitRatioCPU(vector<int> &rsEvts, vector<int> &wsEvts) {
    TH1D *ratioHist
        = new TH1D("ratioHist", "", decayTime->getNumBins(), decayTime->getLowerLimit(), decayTime->getUpperLimit());

    ratios.resize(wsEvts.size());
    errors.resize(wsEvts.size());

    for(unsigned int i = 0; i < wsEvts.size(); ++i) {
        double ratio = wsEvts[i];

        if(0 == rsEvts[i])
            rsEvts[i] = 1; // Cheating to avoid div by zero.

        ratio /= rsEvts[i];

        if(0 == wsEvts[i])
            wsEvts[i] = 1; // Avoid zero errors

        double error = wsEvts[i] / pow(rsEvts[i], 2);
        error += pow(wsEvts[i], 2) / pow(rsEvts[i], 3);
        error = sqrt(error);

        ratios[i] = ratio;
        errors[i] = error;
        ratioHist->SetBinContent(i + 1, ratio);
        ratioHist->SetBinError(i + 1, error);
    }

    TMinuit *minuit = new TMinuit(3);
    minuit->DefineParameter(0, "constaCoef", 0.03, 0.01, -1, 1);
    minuit->DefineParameter(1, "linearCoef", 0, 0.01, -1, 1);
    minuit->DefineParameter(2, "secondCoef", 0, 0.01, -1, 1);
    minuit->SetFCN(cpvFitFcn);

    gettimeofday(&startTime, nullptr);
    minuit->Migrad();
    gettimeofday(&stopTime, nullptr);
}

int main(int argc, char **argv) {
    GooFit::Application app("Chi-square example", argc, argv);

    int numbins = 100;
    app.add_option("-n,--numbins", numbins, "Number of bins", true);

    try {
        app.run();
    } catch(const GooFit::ParseError &e) {
        return app.exit(e);
    }

    // Time is in units of lifetime
    Variable decayTime{"decayTime", 100, 0, 10};
    decayTime.setNumBins(numbins);

    double rSubD = 0.03;
    double rBarD = 0.03;
    double delta = 0;
    double wpPhi = 0;
    double x_mix = 0.0016;
    double y_mix = 0.0055;
    double magPQ = 1.0;
    double magQP = 1.0 / magPQ;

    int eventsToGenerate = 10000000;

    vector<int> dZeroEvtsWS(decayTime.getNumBins());
    vector<int> dZeroEvtsRS(decayTime.getNumBins());
    vector<int> d0barEvtsWS(decayTime.getNumBins());
    vector<int> d0barEvtsRS(decayTime.getNumBins());

    double dZeroLinearCoef = magPQ * sqrt(rSubD) * (y_mix * cos(delta + wpPhi) - x_mix * sin(delta + wpPhi));
    double d0barLinearCoef = magQP * sqrt(rBarD) * (y_mix * cos(delta - wpPhi) - x_mix * sin(delta - wpPhi));

    double dZeroSecondCoef = 0.25 * magPQ * magPQ * (x_mix * x_mix + y_mix * y_mix);
    double d0barSecondCoef = 0.25 * magQP * magQP * (x_mix * x_mix + y_mix * y_mix);

    generateEvents(dZeroEvtsRS, dZeroEvtsWS, &decayTime, rSubD, dZeroLinearCoef, dZeroSecondCoef, eventsToGenerate);
    generateEvents(d0barEvtsRS, d0barEvtsWS, &decayTime, rBarD, d0barLinearCoef, d0barSecondCoef, eventsToGenerate);

    double gpuTime = 0;
    double cpuTime = 0;

    int retval;
    retval = fitRatio(dZeroEvtsRS, dZeroEvtsWS, "dzeroEvtRatio.png");

    if(retval != 0)
        return retval;

    timersub(&stopTime, &startTime, &totalTime);
    gpuTime += totalTime.tv_sec + totalTime.tv_usec / 1000000.0;
    retval = fitRatio(d0barEvtsRS, d0barEvtsWS, "dzbarEvtRatio.png");
    if(retval != 0)
        return retval;
    timersub(&stopTime, &startTime, &totalTime);
    gpuTime += totalTime.tv_sec + totalTime.tv_usec / 1000000.0;

    fitRatioCPU(dZeroEvtsRS, dZeroEvtsWS);
    timersub(&stopTime, &startTime, &totalTime);
    cpuTime += totalTime.tv_sec + totalTime.tv_usec / 1000000.0;
    fitRatioCPU(d0barEvtsRS, d0barEvtsWS);
    timersub(&stopTime, &startTime, &totalTime);
    cpuTime += totalTime.tv_sec + totalTime.tv_usec / 1000000.0;

    std::cout << "GPU time [seconds] : " << gpuTime << "\nCPU time [seconds] : " << cpuTime << std::endl;

    return 0;
}
