#include "hip/hip_runtime.h"
#include "goofit/Application.h"
#include "goofit/FitManager.h"
#include "goofit/UnbinnedDataSet.h"
#include "goofit/PDFs/LandauPdf.h"
#include "goofit/PDFs/NovosibirskPdf.h"
#include "goofit/PDFs/BifurGaussPdf.h"

#include "goofit/Variable.h"
#include "TH1F.h"
#include "TStyle.h"
#include "TCanvas.h"
#include "TRandom.h"

#include <sys/time.h>
#include <sys/times.h>
#include <iostream>

using namespace std;
using namespace GooFit;

// CPU-side Novosibirsk evaluation for use in generating toy MC.
double novosib(double x, double peak, double width, double tail) {
    double qa=0, qb=0, qc=0, qx=0, qy=0;

    if(fabs(tail) < 1.e-7)
        qc = 0.5*pow(((x-peak)/width), 2);
    else {
        qa = tail*sqrt(log(4.));
        qb = sinh(qa)/qa;
        qx = (x-peak)/width*qb;
        qy = 1.+tail*qx;

        //---- Cutting curve from right side

        if(qy > 1.E-7)
            qc = 0.5*(pow((log(qy)/tail), 2) + tail*tail);
        else
            qc = 15.0;
    }

    //---- Normalize the result

    return exp(-qc);
}

TCanvas* foo = 0;

void fitAndPlot(GooPdf* total, UnbinnedDataSet* data, TH1F& dataHist, Variable* xvar, const char* fname) {
    total->setData(data);
    FitManager fitter(total);
    fitter.fit();
    
    if(!fitter)
        std::exit(fitter);

    TH1F pdfHist("pdfHist", "", xvar->getNumBins(), xvar->getLowerLimit(), xvar->getUpperLimit());
    pdfHist.SetStats(false);

    UnbinnedDataSet grid(xvar);
    double step = (xvar->getUpperLimit() - xvar->getLowerLimit())/xvar->getNumBins();

    for(int i = 0; i < xvar->getNumBins(); ++i) {
        xvar->setValue(xvar->getLowerLimit() + (i + 0.5) * step);
        grid.addEvent();
    }

    total->setData(&grid);
    std::vector<std::vector<double>> pdfVals =  total->getCompProbsAtDataPoints();

    double totalPdf = 0;

    for(int i = 0; i < grid.getNumEvents(); ++i) {
        grid.loadEvent(i);
        pdfHist.Fill(xvar->getValue(), pdfVals[0][i]);
        totalPdf += pdfVals[0][i];
    }

    for(int i = 0; i < xvar->getNumBins(); ++i) {
        double val = pdfHist.GetBinContent(i+1);
        val /= totalPdf;
        val *= data->getNumEvents();
        pdfHist.SetBinContent(i+1, val);
    }

    //foo->SetLogy(true);
    dataHist.SetMarkerStyle(8);
    dataHist.SetMarkerSize(0.5);
    dataHist.Draw("p");
    pdfHist.SetLineColor(kBlue);
    pdfHist.SetLineWidth(3);
    pdfHist.Draw("lsame");
    foo->SaveAs(fname);
}

int main(int argc, char** argv) {
    GooFit::Application app("Simple fit example", argc, argv);

    size_t numevents = 100000;
    app.add_option("-n,--num", numevents, "Number of events", true);
    
    try {
        app.run();
    } catch (const GooFit::ParseError &e) {
        return app.exit(e);
    }

    gStyle->SetCanvasBorderMode(0);
    gStyle->SetCanvasColor(10);
    gStyle->SetFrameFillColor(10);
    gStyle->SetFrameBorderMode(0);
    gStyle->SetPadColor(0);
    gStyle->SetTitleColor(1);
    gStyle->SetStatColor(0);
    gStyle->SetFillColor(0);
    gStyle->SetFuncWidth(1);
    gStyle->SetLineWidth(1);
    gStyle->SetLineColor(1);
    gStyle->SetPalette(1, 0);

    // Independent variable.
    Variable* xvar = new Variable("xvar", -100, 100);
    xvar->setNumBins(1000); // For such a large range, want more bins for better accuracy in normalisation.

    // Data sets for the three fits.
    UnbinnedDataSet landdata(xvar);
    UnbinnedDataSet bifgdata(xvar);
    UnbinnedDataSet novodata(xvar);

    // Histograms for showing the fit.
    TH1F landHist("landHist", "", xvar->getNumBins(), xvar->getLowerLimit(), xvar->getUpperLimit());
    TH1F bifgHist("bifgHist", "", xvar->getNumBins(), xvar->getLowerLimit(), xvar->getUpperLimit());
    TH1F novoHist("novoHist", "", xvar->getNumBins(), xvar->getLowerLimit(), xvar->getUpperLimit());
    landHist.SetStats(false);
    bifgHist.SetStats(false);
    novoHist.SetStats(false);

    TRandom donram(42);

    double maxNovo = 0;

    for(double x = xvar->getLowerLimit(); x < xvar->getUpperLimit(); x += 0.01) {
        double curr = novosib(x, 0.3, 0.5, 1.0);

        if(curr < maxNovo)
            continue;

        maxNovo = curr;
    }

    double leftSigma = 13;
    double rightSigma = 29;
    double leftIntegral = 0.5 / (leftSigma * sqrt(2*M_PI));
    double rightIntegral = 0.5 / (rightSigma * sqrt(2*M_PI));
    double totalIntegral = leftIntegral + rightIntegral;
    double bifpoint = -10;

    // Generating three sets of toy MC.
    while(landdata.getNumEvents() < numevents) {
        // Landau
        try {
            xvar->setValue(donram.Landau(20, 1));
            landdata.addEvent();
            landHist.Fill(xvar->getValue());
        } catch (const GooFit::OutOfRange &) {}
    }
    
    while (bifgdata.getNumEvents() < numevents) {
        // Bifurcated Gaussian
        double val;
        if(donram.Uniform() < (leftIntegral / totalIntegral)) {
            do {
                val = donram.Gaus(bifpoint, rightSigma);
            } while(val < bifpoint || val > xvar->getUpperLimit());
            xvar->setValue(val);

        } else {
            do {
                val = donram.Gaus(bifpoint, leftSigma);
            } while(val > bifpoint || val < xvar->getLowerLimit());
            xvar->setValue(val);
        }

        bifgdata.addEvent();
        bifgHist.Fill(xvar->getValue());
    }
    
    while (novodata.getNumEvents() < numevents) {

        // And Novosibirsk.
        while(true) {
            xvar->setValue(donram.Uniform(xvar->getLowerLimit(), xvar->getUpperLimit()));
            double y = donram.Uniform(0, maxNovo);

            if(y < novosib(xvar->getValue(), 0.3, 0.5, 1.0))
                break;
        }

        novodata.addEvent();
        novoHist.Fill(xvar->getValue());
    }

    foo = new TCanvas();

    Variable* mpv            = new Variable("mpv", 40, 0, 150);
    Variable* sigma          = new Variable("sigma", 5, 0, 30);
    GooPdf* landau = new LandauPdf("landau", xvar, mpv, sigma);
    fitAndPlot(landau, &landdata, landHist, xvar, "landau.png");


    Variable* nmean = new Variable("nmean", 0.4, -10.0, 10.0);
    Variable* nsigm = new Variable("nsigm", 0.6, 0.0, 1.0);
    Variable* ntail = new Variable("ntail", 1.1, 0.1, 0.0, 3.0);
    GooPdf* novo = new NovosibirskPdf("novo", xvar, nmean, nsigm, ntail);
    fitAndPlot(novo, &novodata, novoHist, xvar, "novo.png");

    Variable* gmean = new Variable("gmean", 3.0, 1, -15, 15);
    Variable* lsigm = new Variable("lsigm", 10, 1, 10, 20);
    Variable* rsigm = new Variable("rsigm", 20, 1, 10, 40);
    GooPdf* bifur = new BifurGaussPdf("bifur", xvar, gmean, lsigm, rsigm);
    fitAndPlot(bifur, &bifgdata, bifgHist, xvar, "bifur.png");

    return 0;
}
