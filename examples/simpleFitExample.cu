#include "hip/hip_runtime.h"
#include "Variable.hh" 
#include "PdfFunctor.hh"
#include "UnbinnedDataSet.hh" 
#include "LandauThrustFunctor.hh" 
#include "NovosibirskThrustFunctor.hh"
#include "BifurGaussThrustFunctor.hh" 

#include "TRandom.hh" 
#include "TH1F.h"
#include "TH2F.h"
#include "TStyle.h" 
#include "TCanvas.h" 

#include <sys/time.h>
#include <sys/times.h>
#include <iostream>

using namespace std; 

// CPU-side Novosibirsk evaluation for use in generating toy MC. 
double novosib (double x, double peak, double width, double tail) {
  double qa=0,qb=0,qc=0,qx=0,qy=0;

  if(fabs(tail) < 1.e-7) 
    qc = 0.5*pow(((x-peak)/width),2);
  else {
    qa = tail*sqrt(log(4.));
    qb = sinh(qa)/qa;
    qx = (x-peak)/width*qb;
    qy = 1.+tail*qx;
  
    //---- Cutting curve from right side

    if( qy > 1.E-7) 
      qc = 0.5*(pow((log(qy)/tail),2) + tail*tail);
    else
      qc = 15.0;
  }

  //---- Normalize the result

  return exp(-qc);
}

TCanvas* foo = 0;

void fitAndPlot (ThrustPdfFunctor* total, UnbinnedDataSet* data, TH1F& dataHist, Variable* xvar, const char* fname) {
  total->setData(data);
  PdfFunctor fitter(total);
  fitter.fit(); 
  fitter.getMinuitValues(); 

  TH1F pdfHist("pdfHist", "", xvar->numbins, xvar->lowerlimit, xvar->upperlimit);
  pdfHist.SetStats(false);

  UnbinnedDataSet grid(xvar);
  double step = (xvar->upperlimit - xvar->lowerlimit)/xvar->numbins;
  for (int i = 0; i < xvar->numbins; ++i) {
    xvar->value = xvar->lowerlimit + (i + 0.5) * step;
    grid.addEvent(); 
  }

  total->setData(&grid);
  vector<vector<double> > pdfVals;
  total->getCompProbsAtDataPoints(pdfVals); 

  double totalPdf = 0; 
  for (int i = 0; i < grid.getNumEvents(); ++i) {
    grid.loadEvent(i); 
    pdfHist.Fill(xvar->value, pdfVals[0][i]);
    totalPdf += pdfVals[0][i]; 
  }

  for (int i = 0; i < xvar->numbins; ++i) {
    double val = pdfHist.GetBinContent(i+1); 
    val /= totalPdf; 
    val *= data->getNumEvents(); 
    pdfHist.SetBinContent(i+1, val); 
  }
  foo->SetLogy(true); 
  dataHist.SetMarkerStyle(8);
  dataHist.SetMarkerSize(0.5);
  dataHist.Draw("p"); 
  pdfHist.SetLineColor(kBlue);
  pdfHist.SetLineWidth(3); 
  pdfHist.Draw("lsame"); 
  foo->SaveAs(fname); 
}

int main (int argc, char** argv) {
  gStyle->SetCanvasBorderMode(0);
  gStyle->SetCanvasColor(10);
  gStyle->SetFrameFillColor(10);
  gStyle->SetFrameBorderMode(0);
  gStyle->SetPadColor(0);
  gStyle->SetTitleColor(1);
  gStyle->SetStatColor(0);
  gStyle->SetFillColor(0);
  gStyle->SetFuncWidth(1);
  gStyle->SetLineWidth(1);
  gStyle->SetLineColor(1);
  gStyle->SetPalette(1, 0);

  // Independent variable. 
  Variable* xvar = new Variable("xvar", -100, 100); 
  xvar->numbins = 1000; // For such a large range, want more bins for better accuracy in normalisation. 

  // Data sets for the three fits. 
  UnbinnedDataSet landdata(xvar);
  UnbinnedDataSet bifgdata(xvar);
  UnbinnedDataSet novodata(xvar);

  // Histograms for showing the fit. 
  TH1F landHist("landHist", "", xvar->numbins, xvar->lowerlimit, xvar->upperlimit);
  TH1F bifgHist("bifgHist", "", xvar->numbins, xvar->lowerlimit, xvar->upperlimit);
  TH1F novoHist("novoHist", "", xvar->numbins, xvar->lowerlimit, xvar->upperlimit);
  landHist.SetStats(false); 
  bifgHist.SetStats(false); 
  novoHist.SetStats(false); 

  TRandom donram(42); 

  double maxNovo = 0; 
  for (double x = xvar->lowerlimit; x < xvar->upperlimit; x += 0.01) {
    double curr = novosib(x, 0.3, 0.5, 1.0);
    if (curr < maxNovo) continue;
    maxNovo = curr; 
  }

  double leftSigma = 13;
  double rightSigma = 29;
  double leftIntegral = 0.5 / (leftSigma * sqrt(2*M_PI));
  double rightIntegral = 0.5 / (rightSigma * sqrt(2*M_PI));
  double totalIntegral = leftIntegral + rightIntegral; 
  double bifpoint = -10; 

  // Generating three sets of toy MC. 
  for (int i = 0; i < 100000; ++i) {
    // Landau
    xvar->value = xvar->upperlimit + 1; 
    while ((xvar->value > xvar->upperlimit) || (xvar->value < xvar->lowerlimit)) {
      xvar->value = donram.Landau(20, 1); 
    }
    landdata.addEvent(); 
    landHist.Fill(xvar->value); 

    // Bifurcated Gaussian
    if (donram.Uniform() < (leftIntegral / totalIntegral)) {
      xvar->value = bifpoint - 1;
      while ((xvar->value < bifpoint) || (xvar->value > xvar->upperlimit)) xvar->value = donram.Gaus(bifpoint, rightSigma);
    }
    else {
      xvar->value = bifpoint + 1;
      while ((xvar->value > bifpoint) || (xvar->value < xvar->lowerlimit)) xvar->value = donram.Gaus(bifpoint, leftSigma);
    }
    bifgdata.addEvent(); 
    bifgHist.Fill(xvar->value); 

    // And Novosibirsk. 
    while (true) {
      xvar->value = donram.Uniform(xvar->lowerlimit, xvar->upperlimit);
      double y = donram.Uniform(0, maxNovo); 
      if (y < novosib(xvar->value, 0.3, 0.5, 1.0)) break;
    }
    novodata.addEvent(); 
    novoHist.Fill(xvar->value); 
  }

  foo = new TCanvas(); 

  Variable* mpv            = new Variable("mpv", 40, 0, 150);
  Variable* sigma          = new Variable("sigma", 5, 0, 30);
  ThrustPdfFunctor* landau = new LandauThrustFunctor("landau", xvar, mpv, sigma); 
  fitAndPlot(landau, &landdata, landHist, xvar, "landau.eps"); 

  
  Variable* nmean = new Variable("nmean", 0.4, -10.0, 10.0);
  Variable* nsigm = new Variable("nsigm", 0.6, 0.0, 1.0);
  Variable* ntail = new Variable("ntail", 1.1, 0.1, 0.0, 3.0);
  ThrustPdfFunctor* novo = new NovosibirskThrustFunctor("novo", xvar, nmean, nsigm, ntail);
  fitAndPlot(novo, &novodata, novoHist, xvar, "novo.eps"); 

  Variable* gmean = new Variable("gmean", 3.0, 1, -15, 15); 
  Variable* lsigm = new Variable("lsigm", 10, 1, 10, 20); 
  Variable* rsigm = new Variable("rsigm", 20, 1, 10, 40); 
  ThrustPdfFunctor* bifur = new BifurGaussThrustFunctor("bifur", xvar, gmean, lsigm, rsigm); 
  fitAndPlot(bifur, &bifgdata, bifgHist, xvar, "bifur.eps"); 
   
  return 0;
}
