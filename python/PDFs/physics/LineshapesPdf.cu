#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

#include <goofit/PDFs/physics/LineshapesPdf.h>
#include <goofit/PDFs/physics/ResonancePdf.h>
#include <goofit/Variable.h>

using namespace GooFit;
namespace py = pybind11;

void init_LineshapesPdf(py::module &m) {
    py::class_<Lineshape, GooPdf>(m, "Lineshape")
        .def(py::init<std::string,
                      Variable *,
                      Variable *,
                      unsigned int,
                      unsigned int,
                      LS,
                      FF,
                      fptype,
                      std::vector<Variable *>>())

        ;
}
