#include "hip/hip_runtime.h"
#include "goofit/PDFs/basic/ArgusPdf.h"
#include "goofit/Variable.h"

namespace GooFit {

__device__ fptype device_Argus_Upper(fptype *evt, ParameterContainer &pc) {
    int id = RO_CACHE(pc.observables[pc.observableIdx + 1]);

    fptype x  = evt[id];
    fptype m0 = RO_CACHE(pc.parameters[pc.parameterIdx + 1]);

    double t = x / m0;

    fptype slope = RO_CACHE(pc.parameters[pc.parameterIdx + 2]);
    fptype power = RO_CACHE(pc.parameters[pc.parameterIdx + 3]);

    pc.incrementIndex(1, 3, 0, 1, 1);

    if(t >= 1)
        return 0;

    t = 1 - t * t;

    return x * pow(t, power) * exp(slope * t);
}

__device__ fptype device_Argus_Lower(fptype *evt, ParameterContainer &pc) {
    int id = RO_CACHE(pc.observables[pc.observableIdx + 1]);

    fptype x  = evt[id];
    fptype m0 = RO_CACHE(pc.parameters[pc.parameterIdx + 1]);

    fptype t = x / m0;

    fptype slope = RO_CACHE(pc.parameters[pc.parameterIdx + 2]);
    fptype power = RO_CACHE(pc.parameters[pc.parameterIdx + 3]);

    pc.incrementIndex(1, 3, 0, 1, 1);

    if(t <= 1)
        return 0;

    t *= t;
    t -= 1;

    fptype ret = x * pow(t, power) * exp(slope * t);

    return ret;
}

__device__ device_function_ptr ptr_to_Argus_Upper = device_Argus_Upper;
__device__ device_function_ptr ptr_to_Argus_Lower = device_Argus_Lower;

__host__ ArgusPdf::ArgusPdf(std::string n, Variable *_x, Variable *m0, Variable *slope, bool upper, Variable *power)
    : GooPdf(_x, n) {
    registerParameter(m0);
    registerParameter(slope);

    if(!power)
        power = new Variable(n + "powervar", 0.5);

    registerParameter(power);

    std::vector<unsigned int> pindices;

    if(upper) {
        ArgusType = 1;
        GET_FUNCTION_ADDR(ptr_to_Argus_Upper);
    } else {
        ArgusType = 0;
        GET_FUNCTION_ADDR(ptr_to_Argus_Lower);
    }

    initialize(pindices);
}

__host__ void ArgusPdf::recursiveSetIndices() {
    if(ArgusType == 1) {
        GOOFIT_TRACE("host_function_table[{}] = {}({})", num_device_functions, getName(), "ptr_to_Argus_Upper");
        GET_FUNCTION_ADDR(ptr_to_Argus_Upper);
    } else if(ArgusType == 0) {
        GOOFIT_TRACE("host_function_table[{}] = {}({})", num_device_functions, getName(), "ptr_to_Argus_Lower");
        GET_FUNCTION_ADDR(ptr_to_Argus_Lower);
    }

    host_function_table[num_device_functions] = host_fcn_ptr;
    functionIdx                               = num_device_functions++;

    populateArrays();
}

fptype argus_lower_helper(fptype x, fptype m0, fptype slope, fptype power) {
    fptype t = x / m0;

    if(t <= 1)
        return 0;

    t *= t;
    t -= 1;

    fptype ret = x * pow(t, power) * exp(slope * t);

    return ret;
}

__host__ double ArgusPdf::integrate(fptype lo, fptype hi) const {
    double norm  = 0;
    fptype m0    = host_parameters[parametersIdx + 1];
    fptype slope = host_parameters[parametersIdx + 2];
    fptype power = host_parameters[parametersIdx + 3];

    for(int j = 0; j < integrationBins; ++j) {
        double x = hi;
        x -= lo;
        x /= integrationBins;
        x *= j;
        x += lo;
        norm += argus_lower_helper(x, m0, slope, power);
    }

    norm *= ((hi - lo) / integrationBins);
    return norm;
}
} // namespace GooFit
