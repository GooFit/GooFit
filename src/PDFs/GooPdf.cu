#include "hip/hip_runtime.h"
#include <goofit/BinnedDataSet.h>
#include <goofit/Error.h>
#include <goofit/FitControl.h>
#include <goofit/GlobalCudaDefines.h>
#include <goofit/Log.h>
#include <goofit/PDFs/GooPdf.h>
#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/UnbinnedDataSet.h>
#include <goofit/Variable.h>
#include <goofit/Version.h>
#include <goofit/detail/SmartVector.h>
#include <goofit/detail/ThrustOverride.h>

#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>

#ifdef GOOFIT_MPI
#include <mpi.h>
#endif

namespace GooFit {

// These variables are either function-pointer related (thus specific to this implementation)
// or constrained to be in the CUDAglob translation unit by nvcc limitations; otherwise they
// would be in PdfBase.

// For debugging

__constant__ int gpuDebug;
__constant__ unsigned int debugParamIndex;

int cpuDebug = 0;

GooPdf::~GooPdf() { cleanup(); }

// Reduce the PDFs to a single value based on metric taker
// numVars will be different for binned or unbinned fit
// This does NOT normalize!
__host__ double GooPdf::reduce_with_metric() const {
    double ret;

    double start = 0.0;

    thrust::constant_iterator<int> eventSize(get_event_size());
    thrust::constant_iterator<fptype *> arrayAddress(dev_event_array);
    thrust::counting_iterator<int> eventIndex(0);

#ifdef GOOFIT_MPI
    size_t entries_to_process = m_iEventsPerTask;
#else
    size_t entries_to_process = numEntries;
#endif

    // Calls and sums in parallel:
    // logger(0, arrayAddress, eventSize) +
    // logger(1, arrayAddress, eventSize) +
    // ...

    ret = thrust::transform_reduce(
        thrust::make_zip_iterator(thrust::make_tuple(eventIndex, arrayAddress, eventSize)),
        thrust::make_zip_iterator(thrust::make_tuple(eventIndex + entries_to_process, arrayAddress, eventSize)),
        *logger,
        start,
        thrust::plus<double>());

#ifdef GOOFIT_MPI
    double r = ret;
    MPI_Allreduce(&r, &ret, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
#endif

    return ret;
}

// Reduce the PDFs to a single value based on metric taker
// numVars will be different for binned or unbinned fit
// This does NOT normalize!
__host__ double GooPdf::reduce_with_bins() const {
    double sum;
    double start = 0.0;

    thrust::constant_iterator<int> eventSize(observablesList.size());
    thrust::constant_iterator<fptype *> arrayAddress(normRanges);
    thrust::counting_iterator<int> binIndex(0);
    logger->setFunctionIndex(functionIdx);

    size_t totalBins = get_bin_grid_size();

    // Calls and sums in parallel:
    // logger(0, eventSize, arrayAddress) +
    // logger(1, eventSize, arrayAddress) +
    // ...
    // Note that these are *reversed*!

    sum = thrust::transform_reduce(
        thrust::make_zip_iterator(thrust::make_tuple(binIndex, eventSize, arrayAddress)),
        thrust::make_zip_iterator(thrust::make_tuple(binIndex + totalBins, eventSize, arrayAddress)),
        *logger,
        start,
        thrust::plus<double>());

    return sum;
}

/// This evaluates the current function over the data. Does *not* prepare
/// or normalize
__host__ void GooPdf::evaluate_with_metric(thrust::device_vector<fptype> &results) const {
    // if(results.size() != numEntries)
    //    results.resize(numEntries);

    thrust::constant_iterator<int> eventSize(observablesList.size());
    thrust::constant_iterator<fptype *> arrayAddress(dev_event_array);
    thrust::counting_iterator<int> eventIndex(0);

#ifdef GOOFIT_MPI
    size_t entries_to_process = m_iEventsPerTask;
#else
    size_t entries_to_process = numEntries;
#endif

    results.resize(entries_to_process);

    // Calls in parallel:
    // logger(0, arrayAddress, eventSize)
    // logger(1, arrayAddress, eventSize)
    // ...

    thrust::transform(
        thrust::make_zip_iterator(thrust::make_tuple(eventIndex, arrayAddress, eventSize)),
        thrust::make_zip_iterator(thrust::make_tuple(eventIndex + entries_to_process, arrayAddress, eventSize)),
        results.begin(),
        *logger);

    // We need to copy each 'results' buffer to each other
#ifdef GOOFIT_MPI
    // copy our local device buffer to a temporary host_vector
    thrust::host_vector<fptype> local_results = results;

    int myId, numProcs;
    MPI_Comm_size(MPI_COMM_WORLD, &numProcs);
    MPI_Comm_rank(MPI_COMM_WORLD, &myId);

    int counts[numProcs];
    int displs[numProcs];

    // gather all our counts.
    MPI_Allgather(&entries_to_process, 1, MPI_INT, &counts[0], 1, MPI_INT, MPI_COMM_WORLD);

    // calculate our displs.
    displs[0] = 0;
    for(int i = 1; i < numProcs; i++)
        displs[i] = displs[i - 1] + counts[i - 1];

    thrust::host_vector<fptype> total_results;
    total_results.resize(numEntries);

    MPI_Allgatherv(&local_results[0],
                   local_results.size(),
                   MPI_DOUBLE,
                   &total_results[0],
                   &counts[0],
                   &displs[0],
                   MPI_DOUBLE,
                   MPI_COMM_WORLD);

    // copy our results back to our device_vector.
    results = total_results;
#endif
}

__host__ thrust::host_vector<fptype> GooPdf::evaluate_with_metric() const {
    thrust::device_vector<fptype> results;
    results.resize(numEntries);
    evaluate_with_metric(results);
    return thrust::host_vector<fptype>(results);
}

__host__ void GooPdf::setIndices() {
    // If not set, perform unbinned Nll fit!
    if(!fitControl)
        setFitControl(std::make_shared<UnbinnedNllFit>());

    // Ensure that we properly populate *logger with the correct metric
    setMetrics();

    GOOFIT_TRACE("GooPdf::setIndices!");
    PdfBase::setIndices();

    GOOFIT_DEBUG("host_function_table[{}] = {} (fitControl)", host_function_table.size(), fitControl->getName());
    host_function_table.push_back(getMetricPointer(fitControl->getMetric()));

    // copy all the device functions over:
    pre_run();
}

__host__ int GooPdf::findFunctionIdx(void *dev_functionPtr) {
    // Code specific to function-pointer implementation
    auto localPos = functionAddressToDeviceIndexMap.find(dev_functionPtr);

    if(localPos != functionAddressToDeviceIndexMap.end()) {
        return (*localPos).second;
    }

    int fIdx = host_function_table.size();
    host_function_table.push_back(dev_functionPtr);
    functionAddressToDeviceIndexMap[dev_functionPtr] = fIdx;
    host_function_table.sync(d_function_table);

    return fIdx;
}

__host__ void GooPdf::setDebugMask(int mask, bool setSpecific) const {
    cpuDebug = mask;
#if THRUST_DEVICE_SYSTEM != THRUST_DEVICE_SYSTEM_CUDA
    gpuDebug = cpuDebug;

    if(setSpecific)
        debugParamIndex = parameters;

#else
    MEMCPY_TO_SYMBOL(gpuDebug, &cpuDebug, sizeof(int), 0, hipMemcpyHostToDevice);

    if(setSpecific)
        MEMCPY_TO_SYMBOL(debugParamIndex, &parameters, sizeof(unsigned int), 0, hipMemcpyHostToDevice);

#endif
}

__host__ double GooPdf::calculateNLL() {
    GOOFIT_MAYBE_UNUSED fptype norm = normalize();
    GOOFIT_TRACE("GooPdf::calculateNLL calling normalize: {} (host_norm should be 1: {})",
                 norm,
                 host_normalizations.at(normalIdx + 1));

    if(host_normalizations[normalIdx + 1] <= 0)
        GooFit::abort(__FILE__, __LINE__, getName() + " non-positive normalization", this);

    // make this memcpy async
    host_normalizations.sync(d_normalizations);

    fptype ret = reduce_with_metric();
    GOOFIT_TRACE("GooPdf::calculateNLL calling sumOfNll = {}", ret);

    if(0.0 == ret)
        GooFit::abort(__FILE__, __LINE__, getName() + " zero NLL", this);

    return 2.0 * ret;
}

__host__ std::vector<fptype> GooPdf::evaluateAtPoints(Observable var) {
    setFitControl(std::make_shared<EvalFit>());

    setIndices();

    normalize();
    host_normalizations.sync(d_normalizations);

    UnbinnedDataSet tempdata(observablesList);

    double step = var.getBinSize();

    for(int i = 0; i < var.getNumBins(); ++i) {
        var.setValue(var.getLowerLimit() + (i + 0.5) * step);
        tempdata.addEvent();
    }

    auto old = getData();
    setData(&tempdata);

    normalize();

    host_normalizations.sync(d_normalizations);

    thrust::host_vector<fptype> h_results = evaluate_with_metric();
    std::vector<fptype> res;
    res.resize(var.getNumBins());

    for(int i = 0; i < var.getNumBins(); ++i) {
        fptype n = host_normalizations.at(normalIdx + 1);
        fptype v = h_results[i];
        res[i]   = v * n;
    }

    setData(old);

    return res;
}

__host__ fptype GooPdf::getValue(EvalFunc evalfunc) {
    if(evalfunc == EvalFunc::Prob)
        setFitControl(std::make_shared<ProbFit>());
    else if(evalfunc == EvalFunc::Eval)
        setFitControl(std::make_shared<EvalFit>());
    else if(evalfunc == EvalFunc::NLL)
        setFitControl(std::make_shared<UnbinnedNllFit>());
    else
        throw GeneralError("That EvalFunc is not supported");

    setIndices();
    normalize();
    host_normalizations.sync(d_normalizations);

    UnbinnedDataSet point(observablesList);
    point.addEvent();
    auto old = getData();
    setData(&point);

    auto results = evaluate_with_metric();

    setData(old);

    return results[0];
}

__host__ fptype GooPdf::normalize() {
    if(!fitControl->metricIsPdf()) {
        GOOFIT_TRACE("{}: metricIsPdf, returning 1", getName());
        host_normalizations.at(normalIdx + 1) = 1.0;
        cachedNormalization                   = 1.0;
        return 1.0;
    }

    fptype ret = 1;

    if(hasAnalyticIntegral()) {
        // Loop goes only over observables of this PDF.
        for(const Observable &v : observablesList) {
            GOOFIT_TRACE("{}: Analytically integrating over {}", getName(), v.getName());
            ret *= integrate(v.getLowerLimit(), v.getUpperLimit());
        }

        host_normalizations.at(normalIdx + 1) = 1.0 / ret;
        cachedNormalization                   = 1.0 / ret;
        GOOFIT_TRACE("{}: Param {} integral is = {}", getName(), parameters, ret);

        return ret;
    }

    GOOFIT_TRACE("{}, Computing integral without analytic help", getName());

    size_t totalBins = get_bin_grid_size();

    GOOFIT_TRACE("Total Bins: {}", totalBins);

    ret *= get_bin_grid_volume();

    GOOFIT_TRACE("Bin volume: {}", ret);

    ret /= totalBins;

    GOOFIT_TRACE("Total bin volume: {}", ret);

    fptype sum = reduce_with_bins();

    GOOFIT_TRACE("Sum: {}", sum);

    if(std::isnan(sum)) {
        GooFit::abort(__FILE__, __LINE__, getName() + " NaN in normalization", this);
    } else if(0 >= sum) {
        GooFit::abort(__FILE__, __LINE__, "Non-positive normalization", this);
    }

    ret *= sum;

    if(0 == ret)
        GooFit::abort(__FILE__, __LINE__, "Zero integral");

    GOOFIT_TRACE("{}: Param {} integral is ~= {}", getName(), normalIdx, ret);
    host_normalizations.at(normalIdx + 1) = 1.0 / ret;
    cachedNormalization                   = 1.0 / ret;
    return (fptype)ret;
}

__device__ fptype callFunction(fptype *eventAddress, ParameterContainer &pc) {
    return (*(reinterpret_cast<device_function_ptr>(d_function_table[pc.funcIdx])))(eventAddress, pc);
}

__host__ std::vector<std::vector<fptype>> GooPdf::getCompProbsAtDataPoints() {
    // note, we need to overwrite what our metric operator is going to do, and restore previous
    auto fc = fitControl;
    setFitControl(std::make_shared<ProbFit>());

    normalize();
    host_normalizations.sync(d_normalizations);

    thrust::host_vector<fptype> host_results = evaluate_with_metric();

    std::vector<std::vector<fptype>> values;
    values.resize(components.size() + 1);
    values[0] = std::vector<fptype>(host_results.begin(), host_results.end());

    for(unsigned int i = 0; i < components.size(); ++i) {
        // we need to recreate the indexing for each component
        // components[i]->setFitControl(std::make_shared<ProbFit>());
        components[i]->setIndices();
        components[i]->normalize();

        GOOFIT_TRACE("host_function_table[{}] = {}", host_function_table.size(), fitControl->getName());
        host_function_table.push_back(getMetricPointer(fitControl->getMetric()));

        // copy all the device functions over:
        GOOFIT_DEBUG("Copying all host side parameters to device (normalizations too)");
        pre_run();

        auto result   = evaluate_with_metric();
        values[1 + i] = std::vector<fptype>(result.begin(), result.end());
    }

    // restore previous fit control
    setFitControl(fc);

    return values;
}

} // namespace GooFit
