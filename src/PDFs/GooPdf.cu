#include "hip/hip_runtime.h"
#include <goofit/GlobalCudaDefines.h>
#include <goofit/PDFs/GooPdf.h>
#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/detail/ThrustOverride.h>

#include <goofit/BinnedDataSet.h>
#include <goofit/Error.h>
#include <goofit/FitControl.h>
#include <goofit/Log.h>
#include <goofit/UnbinnedDataSet.h>
#include <goofit/Variable.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>

#ifdef ROOT_FOUND
#include <TH1D.h>
#endif

#ifdef GOOFIT_MPI
#include <mpi.h>
#endif

namespace GooFit {

// These variables are either function-pointer related (thus specific to this implementation)
// or constrained to be in the CUDAglob translation unit by nvcc limitations; otherwise they
// would be in PdfBase.

// Device-side, translation-unit constrained.  These were constant, removing const.
// The reason is that this will make it much more difficult to fetch memory, since
// it has less memory to work with limiting the constant amount.
__device__ fptype d_parameters[maxParams];
__device__ fptype d_constants[maxParams];
__device__ fptype d_observables[maxParams];
__device__ fptype d_normalisations[maxParams];

__constant__ unsigned int c_totalEvents;
__constant__ fptype c_motherMass;
__constant__ fptype c_daug1Mass;
__constant__ fptype c_daug2Mass;
__constant__ fptype c_daug3Mass;
__constant__ fptype c_meson_radius;

// For debugging

__constant__ int callnumber;
__constant__ int gpuDebug;
__constant__ unsigned int debugParamIndex;
__device__ int internalDebug1 = -1;
__device__ int internalDebug2 = -1;
__device__ int internalDebug3 = -1;
int cpuDebug                  = 0;

#ifdef PROFILING
__device__ fptype timeHistogram[10000];
fptype host_timeHist[10000];
#endif

// Function-pointer related.
__device__ void *device_function_table[200];
// Not clear why this cannot be __constant__, but it causes crashes to declare it so.

void *host_function_table[200];
unsigned int num_device_functions = 0;
std::map<void *, int> functionAddressToDeviceIndexMap;

// For use in debugging memory issues
void printMemoryStatus(std::string file, int line) {
    size_t memfree  = 0;
    size_t memtotal = 0;
    hipDeviceSynchronize();

#if THRUST_DEVICE_SYSTEM == THRUST_DEVICE_SYSTEM_CUDA
    hipMemGetInfo(&memfree, &memtotal);
#endif
    hipDeviceSynchronize();
    std::cout << "Memory status " << file << " " << line << " Free " << memfree << " Total " << memtotal << " Used "
              << (memtotal - memfree) << std::endl;
}

__device__ fptype calculateEval(fptype rawPdf, fptype *evtVal, fptype norm) {
    // Just return the raw PDF value, for use in (eg) normalisation.
    return rawPdf;
}

__device__ fptype calculateNLL(fptype rawPdf, fptype *evtVal, fptype norm) {
    // if ((10 > callnumber) && (THREADIDX < 10) && (BLOCKIDX == 0)) cuPrintf("calculateNll %i %f %f %f\n", callnumber,
    // rawPdf, normalisationFactors[par], rawPdf*normalisationFactors[par]);  if (THREADIDX < 50) printf("Thread %i %f
    // %f\n", THREADIDX, rawPdf, normalisationFactors[par]);
    rawPdf *= norm;
    return rawPdf > 0.0 ? -log(rawPdf) : 0.0;
}

__device__ fptype calculateProb(fptype rawPdf, fptype *evtVal, fptype norm) {
    // Return probability, ie normalized PDF value.
    return rawPdf * norm;
}

__device__ fptype calculateBinAvg(fptype rawPdf, fptype *evtVal, fptype norm) {
    // TODO:(brad) address these metric devices later
    rawPdf *= norm;
    rawPdf *= evtVal[1]; // Bin volume

    // Log-likelihood of numEvents with expectation of exp is (-exp + numEvents*ln(exp) - ln(numEvents!)).
    // The last is constant, so we drop it; and then multiply by minus one to get the negative log-likelihood.
    if(rawPdf > 0) {
        fptype expEvents = c_totalEvents * rawPdf;
        return (expEvents - evtVal[0] * log(expEvents));
    }

    return 0;
}

__device__ fptype calculateBinWithError(fptype rawPdf, fptype *evtVal, fptype norm) {
    // TODO:(brad) address these metric devices later

    // In this case interpret the rawPdf as just a number, not a number of events.
    // Do not divide by integral over phase space, do not multiply by bin volume,
    // and do not collect 200 dollars. evtVal should have the structure (bin entry, bin error).
    // printf("[%i, %i] ((%f - %f) / %f)^2 = %f\n", BLOCKIDX, THREADIDX, rawPdf, evtVal[0], evtVal[1], pow((rawPdf -
    // evtVal[0]) / evtVal[1], 2));
    rawPdf -= evtVal[0]; // Subtract observed value.
    rawPdf /= evtVal[1]; // Divide by error.
    rawPdf *= rawPdf;
    return rawPdf;
}

__device__ fptype calculateChisq(fptype rawPdf, fptype *evtVal, fptype norm) {
    // TODO:(brad) address these metric devices later
    rawPdf *= norm;
    rawPdf *= evtVal[1]; // Bin volume

    return POW2(rawPdf * c_totalEvents - evtVal[0]) / (evtVal[0] > 1 ? evtVal[0] : 1);
}

__device__ device_metric_ptr ptr_to_Eval         = calculateEval;
__device__ device_metric_ptr ptr_to_NLL          = calculateNLL;
__device__ device_metric_ptr ptr_to_Prob         = calculateProb;
__device__ device_metric_ptr ptr_to_BinAvg       = calculateBinAvg;
__device__ device_metric_ptr ptr_to_BinWithError = calculateBinWithError;
__device__ device_metric_ptr ptr_to_Chisq        = calculateChisq;

void *host_fcn_ptr = nullptr;

void *getMetricPointer(std::string name) {
#define CHOOSE_PTR(ptrname)                                                                                            \
    if(name == #ptrname)                                                                                               \
        GET_FUNCTION_ADDR(ptrname);
    host_fcn_ptr = nullptr;
    CHOOSE_PTR(ptr_to_Eval);
    CHOOSE_PTR(ptr_to_NLL);
    CHOOSE_PTR(ptr_to_Prob);
    CHOOSE_PTR(ptr_to_BinAvg);
    CHOOSE_PTR(ptr_to_BinWithError);
    CHOOSE_PTR(ptr_to_Chisq);

    if(host_fcn_ptr == nullptr)
        throw GooFit::GeneralError("host_fcn_ptr is nullptr");

    return host_fcn_ptr;
#undef CHOOSE_PTR
}

void *getMetricPointer(EvalFunc val) { return getMetricPointer(evalfunc_to_string(val)); }

__host__ void GooPdf::setIndices() {
    // If not set, perform unbinned Nll fit!
    if(!fitControl)
        setFitControl(std::make_shared<UnbinnedNllFit>());

    // Ensure that we properly populate *logger with the correct metric
    setMetrics();

    GOOFIT_DEBUG("GooPdf::setIndices!");
    PdfBase::setIndices();

    GOOFIT_TRACE("host_function_table[{}] = {}", num_device_functions, fitControl->getMetric());
    host_function_table[num_device_functions] = getMetricPointer(fitControl->getMetric());
    num_device_functions++;

    // copy all the device functions over:
    GOOFIT_DEBUG("Copying all host side parameters to device");
    MEMCPY_TO_SYMBOL(
        device_function_table, &host_function_table, num_device_functions * sizeof(fptype), 0, hipMemcpyHostToDevice);
    MEMCPY_TO_SYMBOL(d_parameters, &host_parameters, totalParameters * sizeof(fptype), 0, hipMemcpyHostToDevice);
    MEMCPY_TO_SYMBOL(d_constants, &host_constants, totalConstants * sizeof(fptype), 0, hipMemcpyHostToDevice);
    MEMCPY_TO_SYMBOL(d_observables, &host_observables, totalObservables * sizeof(fptype), 0, hipMemcpyHostToDevice);
}

__host__ int GooPdf::findFunctionIdx(void *dev_functionPtr) {
    // Code specific to function-pointer implementation
    auto localPos = functionAddressToDeviceIndexMap.find(dev_functionPtr);

    if(localPos != functionAddressToDeviceIndexMap.end()) {
        return (*localPos).second;
    }

    int fIdx                                         = num_device_functions;
    host_function_table[num_device_functions]        = dev_functionPtr;
    functionAddressToDeviceIndexMap[dev_functionPtr] = num_device_functions;
    num_device_functions++;
    MEMCPY_TO_SYMBOL(
        device_function_table, host_function_table, num_device_functions * sizeof(void *), 0, hipMemcpyHostToDevice);

#ifdef PROFILING
    host_timeHist[fIdx] = 0;
    MEMCPY_TO_SYMBOL(timeHistogram, host_timeHist, 10000 * sizeof(fptype), 0);
#endif

    return fIdx;
}

__host__ void GooPdf::initialize() {
    if(!fitControl)
        setFitControl(std::make_shared<UnbinnedNllFit>());

    // MetricTaker must be created after PdfBase initialisation is done.
    PdfBase::initializeIndices();

    setMetrics();
}

__host__ void GooPdf::setDebugMask(int mask, bool setSpecific) const {
    cpuDebug = mask;
#if THRUST_DEVICE_SYSTEM != THRUST_DEVICE_SYSTEM_CUDA
    gpuDebug = cpuDebug;

    if(setSpecific)
        debugParamIndex = parameters;

#else
    MEMCPY_TO_SYMBOL(gpuDebug, &cpuDebug, sizeof(int), 0, hipMemcpyHostToDevice);

    if(setSpecific)
        MEMCPY_TO_SYMBOL(debugParamIndex, &parameters, sizeof(unsigned int), 0, hipMemcpyHostToDevice);

#endif
}

__host__ void GooPdf::setMetrics() {
    logger = std::make_shared<MetricTaker>(this, getMetricPointer(fitControl->getMetric()));
}

__host__ double GooPdf::sumOfNll(int numVars) const {
    static thrust::plus<double> cudaPlus;
    thrust::constant_iterator<int> eventSize(numVars);
    thrust::constant_iterator<fptype *> arrayAddress(dev_event_array);
    double dummy = 0;

    // if (host_callnumber >= 2) GooFit::abort(__FILE__, __LINE__, getName() + " debug abort", this);
    thrust::counting_iterator<int> eventIndex(0);

    double ret;
#ifdef GOOFIT_MPI
    double r = thrust::transform_reduce(
        thrust::make_zip_iterator(thrust::make_tuple(eventIndex, arrayAddress, eventSize)),
        thrust::make_zip_iterator(thrust::make_tuple(eventIndex + m_iEventsPerTask, arrayAddress, eventSize)),
        *logger,
        dummy,
        cudaPlus);

    MPI_Allreduce(&r, &ret, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
#else
    ret = thrust::transform_reduce(
        thrust::make_zip_iterator(thrust::make_tuple(eventIndex, arrayAddress, eventSize)),
        thrust::make_zip_iterator(thrust::make_tuple(eventIndex + numEntries, arrayAddress, eventSize)),
        *logger,
        dummy,
        cudaPlus);
#endif
    return ret;
}

__host__ double GooPdf::calculateNLL() const {
    GOOFIT_DEBUG("GooPdf::calculateNLL calling normalize");
    normalize();

    if(host_normalisations[normalIdx + 1] <= 0)
        GooFit::abort(__FILE__, __LINE__, getName() + " non-positive normalisation", this);

    // make this memcpy async
    MEMCPY_TO_SYMBOL(
        d_normalisations, host_normalisations, totalNormalisations * sizeof(fptype), 0, hipMemcpyHostToDevice);
    // hipDeviceSynchronize(); // Ensure normalisation integrals are finished

    int numVars = observablesList.size();

    if(fitControl->binnedFit()) {
        numVars += 2;
        numVars *= -1;
    }

    GOOFIT_DEBUG("GooPdf::calculateNLL calling sumOfNll");
    fptype ret = sumOfNll(numVars);

    if(0.0 == ret)
        GooFit::abort(__FILE__, __LINE__, getName() + " zero NLL", this);

    return 2.0 * ret;
}

__host__ std::vector<fptype> GooPdf::evaluateAtPoints(Observable var) {
    setFitControl(std::make_shared<EvalFit>());

    setIndices();

    normalize();

    MEMCPY(d_normalisations, host_normalisations, totalNormalisations * sizeof(fptype), hipMemcpyHostToDevice);
    UnbinnedDataSet tempdata(observablesList);

    double step = var.getBinSize();

    for(int i = 0; i < var.getNumBins(); ++i) {
        var.setValue(var.getLowerLimit() + (i + 0.5) * step);
        tempdata.addEvent();
    }

    auto old = getData();
    setData(&tempdata);

    normalize();

    MEMCPY(d_normalisations, host_normalisations, totalNormalisations * sizeof(fptype), hipMemcpyHostToDevice);

    thrust::counting_iterator<int> eventIndex(0);
    thrust::constant_iterator<int> eventSize(observablesList.size());
    thrust::constant_iterator<fptype *> arrayAddress(dev_event_array);
    thrust::device_vector<fptype> results(var.getNumBins());

#ifdef GOOFIT_MPI
    thrust::transform(
        thrust::make_zip_iterator(thrust::make_tuple(eventIndex, arrayAddress, eventSize)),
        thrust::make_zip_iterator(thrust::make_tuple(eventIndex + m_iEventsPerTask, arrayAddress, eventSize)),
        results.begin(),
        *logger);
#else
    thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(eventIndex, arrayAddress, eventSize)),
                      thrust::make_zip_iterator(thrust::make_tuple(eventIndex + numEntries, arrayAddress, eventSize)),
                      results.begin(),
                      *logger);
#endif

    // Note, This is not fully realized with MPI.  We need to copy each 'results' buffer to each other 'MPI_Scatterv',
    // then we can do the rest.
    thrust::host_vector<fptype> h_results = results;
    std::vector<fptype> res;
    res.resize(var.getNumBins());

    for(int i = 0; i < var.getNumBins(); ++i) {
        fptype n = host_normalisations[normalIdx + 1];
        fptype v = h_results[i];
        res[i]   = v * n;
    }

    if(old != nullptr)
        setData(old);

    return res;
}

__host__ void GooPdf::scan(Observable var, std::vector<fptype> &values) {
    fptype step = var.getUpperLimit();
    step -= var.getLowerLimit();
    step /= var.getNumBins();
    values.clear();

    for(fptype v = var.getLowerLimit() + 0.5 * step; v < var.getUpperLimit(); v += step) {
        var.setValue(v);
        copyParams();
        fptype curr = calculateNLL();
        values.push_back(curr);
    }
}

// TODO: is this needed?
__host__ void GooPdf::setParameterConstantness(bool constant) {
    std::vector<Variable> pars = getParameters();

    for(Variable &p : pars) {
        p.setFixed(constant);
    }
}

__host__ fptype GooPdf::getValue(EvalFunc evalfunc) {
    if(evalfunc == EvalFunc::Prob)
        setFitControl(std::make_shared<ProbFit>());
    if(evalfunc == EvalFunc::Eval)
        setFitControl(std::make_shared<EvalFit>());
    if(evalfunc == EvalFunc::NLL)
        setFitControl(std::make_shared<UnbinnedNllFit>());

    setIndices();

    // Returns the value of the PDF at a single point.
    // Execute redundantly in all threads for OpenMP multiGPU case
    // copyParams();
    normalize();
    // MEMCPY_TO_SYMBOL(normalisationFactors, host_normalisation, totalParams*sizeof(fptype), 0,
    // hipMemcpyHostToDevice);
    MEMCPY_TO_SYMBOL(
        d_normalisations, host_normalisations, totalNormalisations * sizeof(fptype), 0, hipMemcpyHostToDevice);

    UnbinnedDataSet point(observablesList);
    point.addEvent();
    auto old = getData();
    setData(&point);

    thrust::counting_iterator<int> eventIndex(0);
    thrust::constant_iterator<int> eventSize(observablesList.size());
    thrust::constant_iterator<fptype *> arrayAddress(dev_event_array);
    thrust::device_vector<fptype> results(1);

    // MetricTaker evalor(this, getMetricPointer(evalfunc));
    thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(eventIndex, arrayAddress, eventSize)),
                      thrust::make_zip_iterator(thrust::make_tuple(eventIndex + 1, arrayAddress, eventSize)),
                      results.begin(),
                      *logger);

    // if (old != nullptr)
    //    setData(old);

    return results[0];
}

__host__ fptype GooPdf::normalize() const {
    if(!fitControl->metricIsPdf()) {
        GOOFIT_TRACE("{}: metricIsPdf, returning 1", getName());
        host_normalisations[normalIdx + 1] = 1.0;
        return 1.0;
    }

    fptype ret = 1;

    if(hasAnalyticIntegral()) {
        // Loop goes only over observables of this PDF.
        for(const Observable &v : observablesList) {
            GOOFIT_TRACE("{}: Analytically integrating over {}", getName(), v.getName());
            ret *= integrate(v.getLowerLimit(), v.getUpperLimit());
        }

        host_normalisations[normalIdx + 1] = 1.0 / ret;
        GOOFIT_TRACE("{}: Param {} integral is = {}", getName(), parameters, ret);

        return ret;
    }

    GOOFIT_TRACE("{}, Computing integral without analytic help", getName());

    int totalBins = 1;

    for(const Observable &v : observablesList) {
        ret *= v.getUpperLimit() - v.getLowerLimit();
        totalBins *= integrationBins > 0 ? integrationBins : v.getNumBins();

        GOOFIT_TRACE("Total bins {} due to {} {} {}", totalBins, v.getName(), integrationBins, v.getNumBins());
    }

    ret /= totalBins;

    fptype dummy = 0;
    static thrust::plus<fptype> cudaPlus;
    thrust::constant_iterator<fptype *> arrayAddress(normRanges);
    thrust::constant_iterator<int> eventSize(observablesList.size());
    thrust::counting_iterator<int> binIndex(0);
    // thrust::constant_iterator<int> funcIdx (functionIdx);
    logger->setFunctionIndex(functionIdx);

    fptype sum;
#ifdef GOOFIT_MPI
    fptype s = thrust::transform_reduce(
        thrust::make_zip_iterator(thrust::make_tuple(binIndex, eventSize, arrayAddress)),
        thrust::make_zip_iterator(thrust::make_tuple(binIndex + totalBins, eventSize, arrayAddress)),
        *logger,
        dummy,
        cudaPlus);

    MPI_Allreduce(&s, &sum, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
#else
    sum = thrust::transform_reduce(
        thrust::make_zip_iterator(thrust::make_tuple(binIndex, eventSize, arrayAddress)),
        thrust::make_zip_iterator(thrust::make_tuple(binIndex + totalBins, eventSize, arrayAddress)),
        *logger,
        dummy,
        cudaPlus);

    GOOFIT_TRACE("sum = {}", sum);
#endif

    if(std::isnan(sum)) {
        GooFit::abort(__FILE__, __LINE__, getName() + " NaN in normalisation", this);
    } else if(0 >= sum) {
        GooFit::abort(__FILE__, __LINE__, "Non-positive normalisation", this);
    }

    ret *= sum;

    if(0 == ret)
        GooFit::abort(__FILE__, __LINE__, "Zero integral");

    GOOFIT_TRACE("{}: Param {} integral is ~= {}", getName(), normalIdx, ret);
    host_normalisations[normalIdx + 1] = 1.0 / ret;
    return (fptype)ret;
}

#ifdef PROFILING
__constant__ fptype conversion = (1.0 / CLOCKS_PER_SEC);
__device__ fptype callFunction(fptype *eventAddress, unsigned int functionIdx, unsigned int paramIdx) {
    clock_t start = clock();
    fptype ret    = (*(reinterpret_cast<device_function_ptr>(device_function_table[functionIdx])))(
        eventAddress, hipArray, paramIndices + paramIdx);
    clock_t stop = clock();

    if((0 == THREADIDX + BLOCKIDX) && (stop > start)) {
        // Avoid issue when stop overflows and start doesn't.
        timeHistogram[functionIdx * 100 + paramIdx] += ((stop - start) * conversion);
        // printf("Clock: %li %li %li | %u %f\n", (long) start, (long) stop, (long) (stop - start), functionIdx,
        // timeHistogram[functionIdx]);
    }

    return ret;
}
#else
__device__ fptype callFunction(fptype *eventAddress, ParameterContainer &pc) {
    return (*(reinterpret_cast<device_function_ptr>(device_function_table[pc.funcIdx])))(eventAddress, pc);
}
#endif

__host__ std::vector<std::vector<fptype>> GooPdf::getCompProbsAtDataPoints() {
    // note, we need to overwrite what our metric operator is going to do, and restore previous
    auto fc = fitControl;
    setFitControl(std::make_shared<ProbFit>());

    // copyParams();
    // double overall =
    normalize();
    // MEMCPY_TO_SYMBOL(normalisationFactors, host_normalisations, totalParams*sizeof(fptype), 0,
    // hipMemcpyHostToDevice);
    MEMCPY_TO_SYMBOL(
        d_normalisations, host_normalisations, totalNormalisations * sizeof(fptype), 0, hipMemcpyHostToDevice);

    int numVars = observablesList.size();

    if(fitControl->binnedFit()) {
        numVars += 2;
        numVars *= -1;
    }

    thrust::device_vector<fptype> results(numEntries);
    thrust::constant_iterator<int> eventSize(numVars);
    thrust::constant_iterator<fptype *> arrayAddress(dev_event_array);
    thrust::counting_iterator<int> eventIndex(0);
    // MetricTaker evalor(this, getMetricPointer("ptr_to_Prob"));
    thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(eventIndex, arrayAddress, eventSize)),
                      thrust::make_zip_iterator(thrust::make_tuple(eventIndex + numEntries, arrayAddress, eventSize)),
                      results.begin(),
                      *logger);
    std::vector<std::vector<fptype>> values;
    values.resize(components.size() + 1);
    thrust::host_vector<fptype> host_results = results;

    //

    for(unsigned int i = 0; i < host_results.size(); ++i) {
        values[0].push_back(host_results[i]);
    }

    for(unsigned int i = 0; i < components.size(); ++i) {
        // we need to recreate the indexing for each component
        // components[i]->setFitControl(std::make_shared<ProbFit>());
        components[i]->setIndices();
        components[i]->normalize();

        GOOFIT_TRACE("host_function_table[{}] = {}", num_device_functions, fitControl->getMetric());
        host_function_table[num_device_functions] = getMetricPointer(fitControl->getMetric());
        num_device_functions++;

        // copy all the device functions over:
        GOOFIT_DEBUG("Copying all host side parameters to device");
        MEMCPY_TO_SYMBOL(device_function_table,
                         &host_function_table,
                         num_device_functions * sizeof(fptype),
                         0,
                         hipMemcpyHostToDevice);
        MEMCPY_TO_SYMBOL(d_parameters, &host_parameters, totalParameters * sizeof(fptype), 0, hipMemcpyHostToDevice);
        MEMCPY_TO_SYMBOL(d_constants, &host_constants, totalConstants * sizeof(fptype), 0, hipMemcpyHostToDevice);
        MEMCPY_TO_SYMBOL(
            d_observables, &host_observables, totalObservables * sizeof(fptype), 0, hipMemcpyHostToDevice);
        MEMCPY_TO_SYMBOL(
            d_normalisations, host_normalisations, totalNormalisations * sizeof(fptype), 0, hipMemcpyHostToDevice);

        thrust::counting_iterator<int> ceventIndex(0);
        thrust::transform(
            thrust::make_zip_iterator(thrust::make_tuple(ceventIndex, arrayAddress, eventSize)),
            thrust::make_zip_iterator(thrust::make_tuple(ceventIndex + numEntries, arrayAddress, eventSize)),
            results.begin(),
            *logger);
        host_results = results;

        for(unsigned int j = 0; j < host_results.size(); ++j) {
            values[1 + i].push_back(host_results[j]);
        }
    }

    // restore previous fit control
    setFitControl(fc);

    return values;
}

__host__ UnbinnedDataSet GooPdf::makeGrid() {
    std::vector<Observable> ret = getObservables();

    UnbinnedDataSet grid{ret};
    grid.fillWithGrid();

    return grid;
}

// still need to add OpenMP/multi-GPU code here
__host__ void GooPdf::transformGrid(fptype *host_output) {
    generateNormRange();
    // normalize();
    int totalBins = 1;

    for(const Observable &v : observablesList) {
        totalBins *= v.getNumBins();
    }

    thrust::constant_iterator<fptype *> arrayAddress(normRanges);
    thrust::constant_iterator<int> eventSize(observablesList.size());
    thrust::counting_iterator<int> binIndex(0);
    thrust::device_vector<fptype> d_vec;
    d_vec.resize(totalBins);

    thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(binIndex, eventSize, arrayAddress)),
                      thrust::make_zip_iterator(thrust::make_tuple(binIndex + totalBins, eventSize, arrayAddress)),
                      d_vec.begin(),
                      *logger);

    thrust::host_vector<fptype> h_vec = d_vec;

    for(unsigned int i = 0; i < totalBins; ++i)
        host_output[i] = h_vec[i];
}

__host__ void GooPdf::setFitControl(std::shared_ptr<FitControl> fc) {
    for(auto &component : components) {
        component->setFitControl(fc);
    }

    fitControl = fc;

    setMetrics();

    setIndices();
}

#ifdef ROOT_FOUND
__host__ TH1D *GooPdf::plotToROOT(Observable var, double normFactor, std::string name) {
    if(name.empty())
        name = getName() + "_hist";

    auto ret = new TH1D(name.c_str(), "", var.getNumBins(), var.getLowerLimit(), var.getUpperLimit());
    std::vector<fptype> binValues = evaluateAtPoints(var);

    double pdf_int = 0;

    for(int i = 0; i < var.getNumBins(); ++i) {
        pdf_int += binValues[i];
    }

    for(int i = 0; i < var.getNumBins(); ++i)
        ret->SetBinContent(i + 1, binValues[i] * normFactor / pdf_int / var.getBinSize());
    return ret;
}
#endif
} // namespace GooFit
