#include "hip/hip_runtime.h"
#include "goofit/PDFs/basic/InterHistPdf.h"
#include "goofit/Variable.h"
#include <algorithm>

namespace GooFit {

__constant__ fptype *dev_base_interhists[100]; // Multiple histograms for the case of multiple PDFs
#define OBS_CODE 4242424242
// This number is presumably so high that it will never collide
// with an actual parameter index. It indicates that this dimension
// is an event observable.

// dev_powi is implemented in SmoothHistogramPdf.cu.

__device__ fptype device_InterHistogram(fptype *evt, ParameterContainer &pc) {
    // Structure is
    // nP totalHistograms (idx1 limit1 step1 bins1) (idx2 limit2 step2 bins2) nO o1 o2
    // where limit and step are indices into functorConstants.

    int numVars          = int(RO_CACHE(pc.constants[pc.constantIdx + 2]) - 1) / 4; //(indices[0] - 1) / 4;
    int globalBin        = 0;
    int previous         = 1;
    int myHistogramIndex = RO_CACHE(pc.constants[pc.constantIdx + 1]); // indices[1];
    fptype binDistances[10];                                           // Ten dimensions should be more than enough!
    // Distance from bin center in units of bin width in each dimension.

    unsigned int observablesSeen = 0;

    for(int i = 0; i < numVars; ++i) {
        fptype currVariable = 0;
        unsigned int varIndex = RO_CACHE(pc.constants[pc.constantIdx + 3 + i * 4]); // constantindices[2 + 4 * i];

        // check where we get our value
        if(varIndex == OBS_CODE) {
            // Interpret this number as observable index.
            // Notice that this if does not cause a fork
            // - all threads will hit the same index and
            // make the same decision.
            int id = pc.observables[pc.observableIdx + 1 + i];
            currVariable = evt[id]; // evt[indices[indices[0] + 2 + observablesSeen++]];
        } else {
            // Interpret as parameter index.
            currVariable = pc.parameters[pc.parameterIdx + 1 + varIndex];
        }

        int lowerBoundIdx = 3 + 4 * i;
        fptype lowerBound
            = pc.parameters[pc.parameterIdx + 3 + i * 4 + 1];         // functorConstants[indices[lowerBoundIdx + 0]];
        fptype step = pc.parameters[pc.parameterIdx + 3 + i * 4 + 2]; // functorConstants[indices[lowerBoundIdx + 1]];

        currVariable -= lowerBound;
        currVariable /= step;

        auto localBin   = static_cast<int>(floor(currVariable));
        binDistances[i] = currVariable - localBin - fptype(0.5);
        globalBin += previous * localBin;
        previous *= pc.constants[pc.constantIdx + lowerBoundIdx + 1]; // indices[lowerBoundIdx + 2];

        if(0 == THREADIDX + BLOCKIDX)
            printf("Variable %i: %f %f %i\n", i, currVariable, currVariable * step + lowerBound, localBin);
    }

    fptype *myHistogram = dev_base_interhists[myHistogramIndex];
    fptype ret          = 0;

    //------------------
    //     |     |     |
    //  3  |  4  |  5  |
    //     |     |     |
    //------------------
    //    x|     |     |
    //  0  |  1  |  2  |
    //     |     |     |
    //------------------

    fptype totalWeight = 0;
    int totalBins      = dev_powi(3, numVars);

    for(int i = 0; i < totalBins; ++i) {
        int currBin          = globalBin;
        int localPrevious    = 1;
        int trackingBin      = globalBin;
        bool offSomeAxis     = false;
        fptype currentWeight = 0;

        // Loop over vars to get offset for each one.
        for(int v = 0; v < numVars; ++v) {
            int localNumBins = pc.parameters[pc.parameterIdx + 1 + v]; // indices[4 * (v + 1) + 1];
            int offset       = ((i / dev_powi(3, v)) % 3) - 1;

            currBin += offset * localPrevious;
            localPrevious *= localNumBins;

            int currVarBin = trackingBin % localNumBins;
            trackingBin /= localNumBins;

            if(currVarBin + offset < 0)
                offSomeAxis = true;

            if(currVarBin + offset >= localNumBins)
                offSomeAxis = true;

            fptype currDist = binDistances[v];
            currDist -= offset;
            currentWeight += currDist * currDist;

            if(0 == THREADIDX + BLOCKIDX)
                printf("%i, %i: %f %f %f %i %s\n",
                       i,
                       v,
                       currDist,
                       binDistances[v],
                       currentWeight,
                       offset,
                       offSomeAxis ? "off" : "on");
        }

        // Only interpolate the four closest boxes (in two dimensions; more in three dimensions).
        currentWeight = currentWeight > 0
                            ? (currentWeight <= sqrt(static_cast<fptype>(numVars)) ? 1 / sqrt(currentWeight) : 0)
                            : 0;
        fptype currentEntry = offSomeAxis ? 0 : myHistogram[currBin];
        ret += currentWeight * currentEntry;
        totalWeight += currentWeight;

        if(0 == THREADIDX + BLOCKIDX)
            printf(
                "Adding bin content %i %f with weight %f for total %f.\n", currBin, currentEntry, currentWeight, ret);
    }

    // if(0 == THREADIDX + BLOCKIDX)
    //    printf("%f %f %f %i %f\n", ret, totalWeight, evt[0], indices[6], p[indices[6]]);

    ret /= totalWeight;
    return ret;
}

__device__ device_function_ptr ptr_to_InterHistogram = device_InterHistogram;

__host__ InterHistPdf::InterHistPdf(std::string n,
                                    BinnedDataSet *x,
                                    std::vector<Variable *> params,
                                    std::vector<Variable *> obses)
    : GooPdf(nullptr, n)
    , numVars(x->numVariables()) {
    int numConstants = 2 * numVars;
    // registerConstants(numConstants);
    static unsigned int totalHistograms = 0;
    // host_constants                      = new fptype[numConstants];
    totalEvents = 0;

    std::vector<unsigned int> pindices;
    pindices.push_back(totalHistograms);

    // push on the histogram index and number of variables.
    constantsList.push_back(totalHistograms);
    constantsList.push_back(numVars);

    int varIndex = 0;

    for(Variable *var : x->getVariables()) {
        // push back an out-of-range value for differencing observables with parameters
        if(std::find(obses.begin(), obses.end(), var) != obses.end()) {
            registerObservable(var);
            pindices.push_back(OBS_CODE);
            constantsList.push_back(OBS_CODE);
        } else {
            pindices.push_back(registerParameter(var));
            constantsList.push_back(parametersList.size() - 1);
        }

        pindices.push_back(cIndex + 2 * varIndex + 0);
        pindices.push_back(cIndex + 2 * varIndex + 1);
        pindices.push_back(var->getNumBins());

        constantsList.push_back(var->getLowerLimit());
        constantsList.push_back(var->getBinSize());
        constantsList.push_back(var->getNumBins());

        // NB, do not put cIndex here, it is accounted for by the offset in MEMCPY_TO_SYMBOL below.
        // host_constants[2 * varIndex + 0] = var->getLowerLimit();
        // host_constants[2 * varIndex + 1] = var->getBinSize();
        varIndex++;
    }

    unsigned int numbins = x->getNumBins();
    thrust::host_vector<fptype> host_histogram;

    for(unsigned int i = 0; i < numbins; ++i) {
        fptype curr = x->getBinContent(i);
        host_histogram.push_back(curr);
        totalEvents += curr;
    }

    // MEMCPY_TO_SYMBOL(functorConstants,
    //                 host_constants,
    //                 numConstants * sizeof(fptype),
    //                 cIndex * sizeof(fptype),
    //                 hipMemcpyHostToDevice);

    dev_base_histogram = new thrust::device_vector<fptype>(host_histogram);
    static fptype *dev_address[1];
    dev_address[0] = (&((*dev_base_histogram)[0])).get();
    MEMCPY_TO_SYMBOL(
        dev_base_interhists, dev_address, sizeof(fptype *), totalHistograms * sizeof(fptype *), hipMemcpyHostToDevice);
    GET_FUNCTION_ADDR(ptr_to_InterHistogram);
    initialize(pindices);

    totalHistograms++;
}

void InterHistPdf::recursiveSetIndices() {
    GET_FUNCTION_ADDR(ptr_to_InterHistogram);

    GOOFIT_TRACE("host_function_table[{}] = {}({})", num_device_functions, getName(), "ptr_to_InterHistogram");
    host_function_table[num_device_functions] = host_fcn_ptr;
    functionIdx                               = num_device_functions++;

    populateArrays();
}

} // namespace GooFit
