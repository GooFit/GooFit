#include "hip/hip_runtime.h"
#include "goofit/PDFs/basic/JohnsonSUPdf.h"

namespace GooFit {

const fptype SQRT2PI = 2.506628;

__device__ fptype device_JohnsonSU(fptype *evt, ParameterContainer &pc) {
    int id     = RO_CACHE(pc.observables[pc.observableIdx + 1]);
    fptype _Jm = RO_CACHE(pc.parameters[pc.parameterIdx + 1]);
    fptype _Js = RO_CACHE(pc.parameters[pc.parameterIdx + 2]);
    fptype _Jg = RO_CACHE(pc.parameters[pc.parameterIdx + 3]);
    fptype _Jd = RO_CACHE(pc.parameters[pc.parameterIdx + 4]);

    // we are using index 0.  If we need a different idx, we need to pass that information along.
    fptype x = evt[id];

    pc.incrementIndex(1, 4, 0, 1, 1);

    fptype px       = (x - _Jm) / _Js;
    fptype px2      = px * px;
    fptype sqrt_arg = sqrt(1 + px2);
    fptype inv_sinh = log(px + sqrt_arg);
    fptype gaus_arg = _Jg + _Jd * inv_sinh;
    // if ((gpuDebug & 1) && (0 == BLOCKIDX) && (0 == THREADIDX))
    // if (gpuDebug & 1)
    // printf("Johnson SU: %f %f %f %f | %f %f %i\n", _Jm, _Js, _Jg, _Jd, x, _Jd / (_Js * SQRT2PI * sqrt_arg) * exp(-0.5
    // * gaus_arg * gaus_arg), indices[2 + indices[0]]);
    // printf("Johnson SU: %f %f %f %f | %f %f %f %f\n", _Jm, _Js, _Jg, _Jd, x, _Jd / (_Js * SQRT2PI * sqrt_arg) *
    // exp(-0.5 * gaus_arg * gaus_arg), hipArray[indices[1]], hipArray[indices[2]]);
    return _Jd / (_Js * SQRT2PI * sqrt_arg) * exp(-0.5 * gaus_arg * gaus_arg);
}

__device__ device_function_ptr ptr_to_JohnsonSU = device_JohnsonSU;

__host__ JohnsonSUPdf::JohnsonSUPdf(
    std::string n, Variable *_x, Variable *mean, Variable *sigma, Variable *gamma, Variable *delta)
    : GooPdf(_x, n) {
    std::vector<unsigned int> pindices;
    pindices.push_back(registerParameter(mean));
    pindices.push_back(registerParameter(sigma));
    pindices.push_back(registerParameter(gamma));
    pindices.push_back(registerParameter(delta));
    GET_FUNCTION_ADDR(ptr_to_JohnsonSU);
    initialize(pindices);
}

__host__ void JohnsonSUPdf::recursiveSetIndices() {
    GET_FUNCTION_ADDR(ptr_to_JohnsonSU);

    GOOFIT_TRACE("host_function_table[{}] = {}({})", num_device_functions, getName(), "ptr_to_JohnsonSU");
    host_function_table[num_device_functions] = host_fcn_ptr;
    functionIdx                               = num_device_functions++;

    populateArrays();
}

__host__ fptype JohnsonSUPdf::integrate(fptype lo, fptype hi) const {
    return 1.0; // Analytic integral included in device function! (Correct for minus to plus inf.)
}
} // namespace GooFit
