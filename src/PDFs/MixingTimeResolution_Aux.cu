#include "goofit/PDFs/physics/MixingTimeResolution_Aux.h"
#include "goofit/PDFs/GooPdf.h"

namespace GooFit {

MixingTimeResolution::MixingTimeResolution()
    : GooPdf(NULL, "mixing_resolution") {}
MixingTimeResolution::~MixingTimeResolution() = default;

void MixingTimeResolution::initIndex(void *dev_fcn_ptr) { resFunctionIdx = GooPdf::findFunctionIdx(dev_fcn_ptr); }

void MixingTimeResolution::recursiveSetIndices() {}
} // namespace GooFit
