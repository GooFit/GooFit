#include "hip/hip_runtime.h"
#include "goofit/PDFs/basic/PolynomialPdf.h"
#include "goofit/Variable.h"
#include "goofit/Log.h"

namespace GooFit {

__device__ fptype device_Polynomial(fptype *evt, ParameterContainer &pc) {
    int id = RO_CACHE(pc.observables[pc.observableIdx + 1]);
    // Structure is nP lowestdegree c1 c2 c3 nO o1

    int numParams    = RO_CACHE(pc.parameters[pc.parameterIdx]);
    int lowestDegree = RO_CACHE(pc.constants[pc.constantIdx + 1]);

    fptype x   = evt[id];
    fptype ret = 0;

    // unsure why this starts at i=2...
    for(int i = 0; i < numParams; ++i) {
        fptype param = RO_CACHE(pc.parameters[pc.parameterIdx + i + 1]);
        ret += param * pow(x, lowestDegree + i);
    }

    pc.incrementIndex(1, numParams, 3, 1, 1);

    return ret;
}

__device__ fptype device_OffsetPolynomial(fptype *evt, ParameterContainer &pc) {
    int id = RO_CACHE(pc.observables[pc.observableIdx + 1]);

    int numParams    = RO_CACHE(pc.parameters[pc.parameterIdx]);
    int lowestDegree = RO_CACHE(pc.constants[pc.constantIdx + 1]);

    fptype x = evt[id];
    // TODO: Not sure where this is pointing...
    // x -= RO_CACHE(p[RO_CACHE(indices[numParams])]);
    fptype ret = 0;

    for(int i = 2; i < numParams; ++i) {
        ret += RO_CACHE(pc.parameters[pc.parameterIdx + i]) * pow(x, lowestDegree + i - 2);
    }

    pc.incrementIndex(1, numParams, 1, 1, 1);

    return ret;
}

__device__ fptype device_MultiPolynomial(fptype *evt, ParameterContainer &pc) {
    int num_constants  = RO_CACHE(pc.constants[pc.constantIdx]);
    int num_parameters = RO_CACHE(pc.parameters[pc.parameterIdx]);

    // Structure is nP, maxDegree, offset1, offset2, ..., coeff1, coeff2, ..., nO, o1, o2, ...

    int num_observables = RO_CACHE(pc.observables[pc.observableIdx]);
    int maxDegree       = RO_CACHE(pc.constants[pc.constantIdx + 1]) + 1;
    // Only appears in construction (maxDegree + 1) or (x > maxDegree), so
    // may as well add the one and use >= instead.

    // Technique is to iterate over the full n-dimensional box, skipping matrix elements
    // whose sum of indices is greater than maxDegree. Notice that this is increasingly
    // inefficient as n grows, since a larger proportion of boxes will be skipped.
    int numBoxes = 1;

    for(int i = 0; i < num_observables; ++i)
        numBoxes *= maxDegree;

    int coeffNumber = num_observables; // Index of first coefficient is 2 + nO, not 1 + nO, due to maxDegree. (nO comes
                                       // from offsets.)
    fptype ret = RO_CACHE(pc.parameters[pc.parameterIdx + 1 + coeffNumber]); // Coefficient of constant term.
    coeffNumber++;

    for(int i = 1; i < numBoxes;
        ++i) { // Notice skip of inmost 'box' in the pyramid, corresponding to all powers zero, already accounted for.
        fptype currTerm  = 1;
        int currIndex    = i;
        int sumOfIndices = 0;

        // if ((gpuDebug & 1) && (THREADIDX == 50) && (BLOCKIDX == 3))
        // if ((BLOCKIDX == internalDebug1) && (THREADIDX == internalDebug2))
        // if ((1 > (int) floor(0.5 + evt[8])) && (gpuDebug & 1) && (paramIndices + debugParamIndex == indices))
        // printf("[%i, %i] Start box %i %f %f:\n", BLOCKIDX, THREADIDX, i, ret, evt[8]);
        for(int j = 0; j < num_observables; ++j) {
            // TODO:Need to debug these
            int id        = RO_CACHE(pc.observables[pc.observableIdx + 1 + j]);
            fptype x      = evt[id];                                          // x, y, z...
            fptype offset = RO_CACHE(pc.parameters[pc.parameterIdx + 1 + j]); // x0, y0, z0...
            x -= offset;
            int currPower = currIndex % maxDegree;
            currIndex /= maxDegree;
            currTerm *= pow(x, currPower);
            sumOfIndices += currPower;
            // if ((gpuDebug & 1) && (THREADIDX == 50) && (BLOCKIDX == 3))
            // if ((BLOCKIDX == internalDebug1) && (THREADIDX == internalDebug2))
            // if ((1 > (int) floor(0.5 + evt[8])) && (gpuDebug & 1) && (paramIndices + debugParamIndex == indices))
            // printf("  [%f -> %f^%i = %f] (%i %i) \n", evt[indices[2 + indices[0] + j]], x, currPower, pow(x,
            // currPower), sumOfIndices, indices[2 + indices[0] + j]);
        }

        // if ((gpuDebug & 1) && (THREADIDX == 50) && (BLOCKIDX == 3))
        // if ((BLOCKIDX == internalDebug1) && (THREADIDX == internalDebug2))
        // printf(") End box %i\n", i);
        // All threads should hit this at the same time and with the same result. No branching.
        if(sumOfIndices >= maxDegree)
            continue;

        fptype coefficient = RO_CACHE(pc.parameters[pc.parameterIdx + 1 + coeffNumber]); // Coefficient from MINUIT
        coeffNumber++;
        // if ((gpuDebug & 1) && (THREADIDX == 50) && (BLOCKIDX == 3))
        // if ((BLOCKIDX == internalDebug1) && (THREADIDX == internalDebug2))
        // if ((1 > (int) floor(0.5 + evt[8])) && (gpuDebug & 1) && (paramIndices + debugParamIndex == indices))
        // printf("Box %i contributes %f * %f = %f -> %f\n", i, currTerm, p[indices[coeffNumber - 1]],
        // coefficient*currTerm, (ret + coefficient*currTerm));
        currTerm *= coefficient;
        ret += currTerm;
    }

    pc.incrementIndex(1, num_parameters, num_constants, num_observables, 1);

    return ret;
}

__device__ device_function_ptr ptr_to_Polynomial       = device_Polynomial;
__device__ device_function_ptr ptr_to_OffsetPolynomial = device_OffsetPolynomial;
__device__ device_function_ptr ptr_to_MultiPolynomial  = device_MultiPolynomial;

// Constructor for single-variate polynomial, with optional zero point.
__host__ PolynomialPdf::PolynomialPdf(
    std::string n, Variable *_x, std::vector<Variable *> weights, Variable *x0, unsigned int lowestDegree)
    : GooPdf(_x, n)
    , center(x0) {
    std::vector<unsigned int> pindices;
    pindices.push_back(lowestDegree);

    constantsList.push_back(lowestDegree);

    for(std::vector<Variable *>::iterator v = weights.begin(); v != weights.end(); ++v) {
        pindices.push_back(registerParameter(*v));
    }

    if(x0) {
        polyType = 1;
        pindices.push_back(registerParameter(x0));
        GET_FUNCTION_ADDR(ptr_to_OffsetPolynomial);
    } else {
        polyType = 0;
        GET_FUNCTION_ADDR(ptr_to_Polynomial);
    }

    initialize(pindices);
}

// Constructor for multivariate polynomial.
__host__ PolynomialPdf::PolynomialPdf(std::string n,
                                      std::vector<Variable *> obses,
                                      std::vector<Variable *> coeffs,
                                      std::vector<Variable *> offsets,
                                      unsigned int maxDegree)
    : GooPdf(nullptr, n) {
    unsigned int numParameters = 1;

    constantsList.push_back(maxDegree);

    // For 1 observable, equal to n = maxDegree + 1.
    // For two, n*(n+1)/2, ie triangular number. This generalises:
    // 3: Pyramidal number n*(n+1)*(n+2)/(3*2)
    // 4: Hyperpyramidal number n*(n+1)*(n+2)*(n+3)/(4*3*2)
    // ...
    for(unsigned int i = 0; i < obses.size(); ++i) {
        registerObservable(obses[i]);
        numParameters *= (maxDegree + 1 + i);

        // we are 'padding' the list.
        constantsList.push_back(maxDegree + 1 + i);
    }

    for(int i = observablesList.size(); i > 1; --i)
        numParameters /= i;

    while(numParameters > coeffs.size()) {
        char varName[100];
        sprintf(varName, "%s_extra_coeff_%i", getName().c_str(), static_cast<int>(coeffs.size()));

        Variable *newTerm = new Variable(varName, 0);
        coeffs.push_back(newTerm);

        std::cout << "Warning: " << getName() << " created dummy variable " << varName
                  << " (fixed at zero) to account for all terms.\n";
    }

    while(offsets.size() < obses.size()) {
        char varName[100];
        sprintf(varName, "%s_extra_offset_%i", getName().c_str(), static_cast<int>(offsets.size()));
        Variable *newOffset = new Variable(varName, 0);
        offsets.push_back(newOffset);
    }

    std::vector<unsigned int> pindices;
    pindices.push_back(maxDegree);

    for(auto &offset : offsets) {
        pindices.push_back(registerParameter(offset));
    }

    for(auto &coeff : coeffs) {
        pindices.push_back(registerParameter(coeff));
    }

    polyType = 2;
    GET_FUNCTION_ADDR(ptr_to_MultiPolynomial);
    initialize(pindices);
}

__host__ void PolynomialPdf::recursiveSetIndices() {
    if(polyType == 0) {
        GOOFIT_TRACE("host_function_table[{}] = {}({})", num_device_functions, getName(), "ptr_to_Polynomial");
        GET_FUNCTION_ADDR(ptr_to_Polynomial);
    } else if(polyType == 1) {
        GOOFIT_TRACE("host_function_table[{}] = {}({})", num_device_functions, getName(), "ptr_to_OffsetPolynomia");
        GET_FUNCTION_ADDR(ptr_to_OffsetPolynomial);
    } else if(polyType == 2) {
        GOOFIT_TRACE("host_function_table[{}] = {}({})", num_device_functions, getName(), "ptr_to_MultiPolynomial");
        GET_FUNCTION_ADDR(ptr_to_MultiPolynomial);
    }

    GOOFIT_TRACE("host_function_table[{}] = {}", num_device_functions, getName());
    host_function_table[num_device_functions] = host_fcn_ptr;
    functionIdx                               = num_device_functions++;

    populateArrays();
}

__host__ fptype PolynomialPdf::integrate(fptype lo, fptype hi) const {
    // This is *still* wrong. (13 Feb 2013.)
    fptype lowestDegree = host_constants[constantsIdx + 1];

    if(center) {
        hi -= host_observables[observablesIdx + 1];
        lo -= host_observables[observablesIdx + 2];
    }

    fptype ret = 0;

    for(int i = 2; i < host_parameters[parametersIdx] + (center ? 0 : 1); ++i) {
        fptype powerPlusOne = lowestDegree + i - 2;
        fptype curr         = pow(hi, powerPlusOne);
        curr -= pow(lo, powerPlusOne);
        curr /= powerPlusOne;
        ret += host_parameters[parametersIdx + i] * curr;
    }

    return ret;
}

__host__ fptype PolynomialPdf::getCoefficient(int coef) const {
    // NB! This function only works for single polynomials.
    if(1 != observablesList.size()) {
        std::cout << "Warning: getCoefficient method of PolynomialPdf not implemented for multi-dimensional "
                     "polynomials. Returning zero, which is very likely wrong.\n";
        return 0;
    }

    // True function is, say, ax^2 + bx + c.
    // We express this as (a'x^2 + b'x + c')*N.
    // So to get the true coefficient, multiply the internal
    // one by the normalisation. (In non-PDF cases the normalisation
    // equals one, which gives the same result.)

    // Structure is nP lowestdegree c1 c2 c3 nO o1
    if(coef < host_constants[constantsIdx + 1])
        return 0; // Less than least power.

    if(coef > host_constants[constantsIdx + 1] + (host_parameters[parametersIdx] - 1))
        return 0; // Greater than max power.

    fptype norm = normalize();
    norm        = (1.0 / norm);

    fptype param = host_parameters[parametersIdx + 2 + coef - int(host_constants[constantsIdx + 1])];
    return norm * param;
}
} // namespace GooFit
