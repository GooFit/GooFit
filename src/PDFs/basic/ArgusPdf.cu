#include "hip/hip_runtime.h"
#include <goofit/PDFs/basic/ArgusPdf.h>
#include <goofit/Variable.h>

namespace GooFit {

__device__ fptype device_Argus_Upper(fptype *evt, fptype *p, unsigned int *indices) {
    fptype x  = evt[indices[2 + indices[0]]];
    fptype m0 = p[indices[1]];

    double t = x / m0;

    if(t >= 1)
        return 0;

    fptype slope = p[indices[2]];
    fptype power = p[indices[3]];
    t            = 1 - t * t;
    // printf("device_Argus_Upper %f %f %f %f %f\n", x, m0, slope, t, x * pow(t, power) * exp(slope * t));

    return x * pow(t, power) * exp(slope * t);
}

__device__ fptype device_Argus_Lower(fptype *evt, fptype *p, unsigned int *indices) {
    fptype x  = evt[indices[2 + indices[0]]];
    fptype m0 = p[indices[1]];

    // printf("Argus: %i %i %f %f\n", indices[0], indices[2 + indices[0]], x, m0);
    // printf("Argus: %i %i\n", indices[0], indices[2 + indices[0]]);
    // return 1;

    fptype t = x / m0;

    if(t <= 1)
        return 0;

    t *= t;
    t -= 1;

    fptype slope = p[indices[2]];
    fptype power = p[indices[3]];
    fptype ret   = x * pow(t, power) * exp(slope * t);
    // if ((0 == THREADIDX) && (0 == BLOCKIDX) && (callnumber < 1)) cuPrintf("device_Argus_Lower %i %i %f %f %f %f
    // %f\n", indices[1], indices[2], x, m0, slope, t, ret);
    // if (isnan(ret)) printf("NaN Argus: %f %f %f %f %f %f %f\n", x, m0, t, slope, power, pow(t, power), exp(slope*t));
    // if ((0 == THREADIDX) && (0 == BLOCKIDX) && (gpuDebug & 1))
    // printf("(%i, %i) device_Argus_Lower %f %f %f %f %f\n", BLOCKIDX, THREADIDX, x, m0, slope, t, x * pow(t, power) *
    // exp(slope * t));

    return ret;
}

__device__ device_function_ptr ptr_to_Argus_Upper = device_Argus_Upper;
__device__ device_function_ptr ptr_to_Argus_Lower = device_Argus_Lower;

__host__ ArgusPdf::ArgusPdf(std::string n, Observable _x, Variable m0, Variable slope, bool upper)
    : ArgusPdf(n, _x, m0, slope, upper, Variable(n + "powervar", 0.5)) {}

__host__ ArgusPdf::ArgusPdf(std::string n, Observable _x, Variable m0, Variable slope, bool upper, Variable power)
    : GooPdf(n, _x) {
    registerParameter(m0);
    registerParameter(slope);
    registerParameter(power);

    std::vector<unsigned int> pindices;
    pindices.push_back(m0.getIndex());
    pindices.push_back(slope.getIndex());
    pindices.push_back(power.getIndex());

    if(upper) {
        GET_FUNCTION_ADDR(ptr_to_Argus_Upper);
    } else {
        GET_FUNCTION_ADDR(ptr_to_Argus_Lower);
    }

    initialize(pindices);
}

fptype argus_lower_helper(fptype x, fptype m0, fptype slope, fptype power) {
    fptype t = x / m0;

    if(t <= 1)
        return 0;

    t *= t;
    t -= 1;

    fptype ret = x * pow(t, power) * exp(slope * t);

    return ret;
}

__host__ double ArgusPdf::integrate(fptype lo, fptype hi) const {
    double norm           = 0;
    unsigned int *indices = host_indices + parameters;
    fptype m0             = host_params[indices[1]];
    fptype slope          = host_params[indices[2]];
    fptype power          = host_params[indices[3]];

    for(int j = 0; j < integrationBins; ++j) {
        double x = hi;
        x -= lo;
        x /= integrationBins;
        x *= j;
        x += lo;
        norm += argus_lower_helper(x, m0, slope, power);
    }

    norm *= ((hi - lo) / integrationBins);
    return norm;
}
} // namespace GooFit
