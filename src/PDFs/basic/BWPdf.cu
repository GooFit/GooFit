#include "hip/hip_runtime.h"
#include <goofit/PDFs/basic/BWPdf.h>

namespace GooFit {

__device__ fptype device_BW(fptype *evt, fptype *p, unsigned int *indices) {
    fptype x      = evt[indices[2 + indices[0]]];
    fptype mean   = p[indices[1]];
    fptype gamma  = p[indices[2]];
    fptype rootPi = -2. * atan2(-1.0, 0.0);
    fptype ret    = (gamma / ((x - mean) * (x - mean) + gamma * gamma / 4)) / (2 * rootPi);
    return ret;
}

__device__ device_function_ptr ptr_to_BW = device_BW;

__host__ BWPdf::BWPdf(std::string n, Observable _x, Variable mean, Variable width)
    : GooPdf(n, _x) {
    std::vector<unsigned int> pindices;
    pindices.push_back(registerParameter(mean));
    pindices.push_back(registerParameter(width));
    GET_FUNCTION_ADDR(ptr_to_BW);
    initialize(pindices);
}
} // namespace GooFit
