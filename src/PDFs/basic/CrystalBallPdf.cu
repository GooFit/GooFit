#include "hip/hip_runtime.h"
#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/basic/CrystalBallPdf.h>
#include <goofit/Variable.h>

namespace GooFit {

__device__ fptype device_CrystalBall(fptype *evt, ParameterContainer &pc) {
    // Left-hand tail if alpha is less than 0,
    // right-hand tail if greater, pure Gaussian if 0.
    // return 1;
    int id = pc.getObservable(0);

    fptype x     = RO_CACHE(evt[id]);
    fptype mean  = pc.getParameter(0);
    fptype sigma = pc.getParameter(1);
    fptype alpha = pc.getParameter(2);
    fptype power = pc.getParameter(3);
    fptype rx    = (sigma != 0) ? (x - mean) / sigma : 0;
    fptype ret   = 0;

    if((alpha > 0 && rx <= alpha) || // Right-hand tail, in Gaussian region
       (alpha < 0 && rx >= alpha) || // Left-hand tail, in Gaussian region
       (alpha == 0)) {               // Pure Gaussian
        ret = exp(-0.5 * rx * rx);
    } else { // Tail part
        fptype n_over_alpha = power / alpha;
        fptype a            = exp(-0.5 * alpha * alpha);
        fptype b            = n_over_alpha - alpha;
        fptype d            = b + rx;
        d                   = (d != 0) ? n_over_alpha / d : 0;
        ret                 = a * pow(d, power);
    }

    pc.incrementIndex(1, 4, 0, 1, 1);

    // if ((0 == THREADIDX) && (0 == BLOCKIDX)) printf("device_CB: %f %f %f %f %f %f\n", x, mean, sigma, alpha, power,
    // ret);
    return ret;
}

__device__ device_function_ptr ptr_to_CrystalBall = device_CrystalBall;

__host__ CrystalBallPdf::CrystalBallPdf(std::string n, Observable _x, Variable mean, Variable sigma, Variable alpha)
    : CrystalBallPdf(n, _x, mean, sigma, alpha, Variable(n + "_n", 2)) {}

__host__ CrystalBallPdf::CrystalBallPdf(
    std::string n, Observable _x, Variable mean, Variable sigma, Variable alpha, Variable power)
    : GooPdf("CrystalBallPdf", n, _x, mean, sigma, alpha, power) {
    registerFunction("ptr_to_CrystalBall", ptr_to_CrystalBall);

    initialize();
}

__host__ fptype CrystalBallPdf::integrate(fptype lo, fptype hi) const {
    static const fptype sqrtPiOver2 = 1.2533141373;
    static const fptype sqrt2       = 1.4142135624;

    fptype result = 0.0;
    bool useLog   = false;

    fptype mean  = parametersList[0].getValue();
    fptype sigma = parametersList[1].getValue();
    fptype alpha = parametersList[2].getValue();
    fptype power = parametersList[3].getValue();

    if(fabs(power - 1.0) < 1.0e-05)
        useLog = true;

    fptype tmin = (lo - mean) / sigma;
    fptype tmax = (hi - mean) / sigma;

    if(alpha < 0) {
        fptype tmp = tmin;
        tmin       = -tmax;
        tmax       = -tmp;
    }

    fptype absAlpha = fabs(alpha);

    if(tmin >= -absAlpha) {
        result += sigma * sqrtPiOver2 * (erf(tmax / sqrt2) - erf(tmin / sqrt2));
    } else if(tmax <= -absAlpha) {
        fptype a = pow(power / absAlpha, power) * exp(-0.5 * absAlpha * absAlpha);
        fptype b = power / absAlpha - absAlpha;

        if(useLog) {
            result += a * sigma * (log(b - tmin) - log(b - tmax));
        } else {
            result += a * sigma / (1.0 - power)
                      * (1.0 / (pow(b - tmin, power - 1.0)) - 1.0 / (pow(b - tmax, power - 1.0)));
        }
    } else {
        fptype a = pow(power / absAlpha, power) * exp(-0.5 * absAlpha * absAlpha);
        fptype b = power / absAlpha - absAlpha;

        fptype term1 = 0.0;

        if(useLog) {
            term1 = a * sigma * (log(b - tmin) - log(power / absAlpha));
        } else {
            term1 = a * sigma / (1.0 - power)
                    * (1.0 / (pow(b - tmin, power - 1.0)) - 1.0 / (pow(power / absAlpha, power - 1.0)));
        }

        fptype term2 = sigma * sqrtPiOver2 * (erf(tmax / sqrt2) - erf(-absAlpha / sqrt2));
        result += term1 + term2;
    }

    return result;
}

} // namespace GooFit
