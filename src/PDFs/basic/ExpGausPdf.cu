#include "hip/hip_runtime.h"
#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/basic/ExpGausPdf.h>

namespace GooFit {

__device__ fptype device_ExpGaus(fptype *evt, ParameterContainer &pc) {
    int id = pc.getObservable(0);

    fptype x     = evt[id];
    fptype mean  = pc.getParameter(0);
    fptype sigma = pc.getParameter(1);
    fptype alpha = pc.getParameter(2);

    fptype ret    = 0.5 * alpha;
    fptype exparg = ret * (2 * mean + alpha * sigma * sigma - 2 * x);
    fptype erfarg = (mean + alpha * sigma * sigma - x) / (sigma * 1.4142135623);

    ret *= exp(exparg);
    ret *= erfc(erfarg);

    pc.incrementIndex(1, 3, 0, 1, 1);

    return ret;
}

__device__ device_function_ptr ptr_to_ExpGaus = device_ExpGaus;

ExpGausPdf::ExpGausPdf(std::string n, Observable _x, Variable mean, Variable sigma, Variable tau)
    : GooPdf(n, _x) {
    registerParameter(mean);
    registerParameter(sigma);
    registerParameter(tau);

    initialize();
}

__host__ void ExpGausPdf::recursiveSetIndices() {
    GET_FUNCTION_ADDR(ptr_to_ExpGaus);

    GOOFIT_TRACE("host_function_table[{}] = {}({})", num_device_functions, getName(), "ptr_to_ExpGaus");
    host_function_table[num_device_functions] = host_fcn_ptr;
    functionIdx                               = num_device_functions++;

    populateArrays();
}

} // namespace GooFit
