#include "hip/hip_runtime.h"
#include <goofit/Log.h>
#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/basic/GaussianPdf.h>

namespace GooFit {

__device__ auto device_Gaussian(fptype *evt, ParameterContainer &pc) -> fptype {
    int id       = pc.getObservable(0);
    fptype x     = RO_CACHE(evt[id]);
    fptype mean  = pc.getParameter(0);
    fptype sigma = pc.getParameter(1);
    pc.incrementIndex(1, 2, 0, 1, 1);

    fptype ret = exp(-0.5 * (x - mean) * (x - mean) / (sigma * sigma));

    return ret;
}

__device__ device_function_ptr ptr_to_Gaussian = device_Gaussian;

__host__ GaussianPdf::GaussianPdf(std::string n, Observable _x, Variable mean, Variable sigma)
    : GooPdf("GaussianPdf", n, _x, mean, sigma) {
    registerFunction("ptr_to_Gaussian", ptr_to_Gaussian);

    initialize();
}

__host__ auto GaussianPdf::integrate(fptype lo, fptype hi) const -> fptype {
    static const fptype rootPi = sqrt(atan2(0.0, -1.0));

    // Integral over all R.
    fptype sigma = host_parameters[parametersIdx + 2];
    sigma *= root2 * rootPi;
    return sigma;
}

} // namespace GooFit
