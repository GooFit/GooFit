#include "hip/hip_runtime.h"
#include <algorithm>
#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/basic/InterHistPdf.h>
#include <goofit/Variable.h>

namespace GooFit {

__constant__ fptype *dev_base_interhists[100]; // Multiple histograms for the case of multiple PDFs
#define OBS_CODE 4242424242
// This number is presumably so high that it will never collide
// with an actual parameter index. It indicates that this dimension
// is an event observable.

// dev_powi is implemented in SmoothHistogramPdf.cu.

__device__ auto device_InterHistogram(fptype *evt, ParameterContainer &pc) -> fptype {
    // Structure is
    // nP totalHistograms (idx1 limit1 step1 bins1) (idx2 limit2 step2 bins2) nO o1 o2
    // where limit and step are indices into functorConstants.

    int numVars          = int(pc.getConstant(1) - 1) / 4; //(indices[0] - 1) / 4;
    int globalBin        = 0;
    int previous         = 1;
    int myHistogramIndex = pc.getConstant(0); // indices[1];
    fptype binDistances[10];                  // Ten dimensions should be more than enough!
    // Distance from bin center in units of bin width in each dimension.

    for(int i = 0; i < numVars; ++i) {
        fptype currVariable   = 0;
        unsigned int varIndex = pc.getConstant(2 + i * 4); // constantindices[2 + 4 * i];

        // check where we get our value
        if(varIndex == OBS_CODE) {
            // Interpret this number as observable index.
            // Notice that this if does not cause a fork
            // - all threads will hit the same index and
            // make the same decision.
            int id       = pc.getObservable(i);
            currVariable = RO_CACHE(evt[id]); // evt[indices[indices[0] + 2 + observablesSeen++]];
        } else {
            // Interpret as parameter index.
            currVariable = pc.getParameter(varIndex);
        }

        int lowerBoundIdx = 3 + 4 * i;
        fptype lowerBound = pc.getParameter(3 + i * 4);     // functorConstants[indices[lowerBoundIdx + 0]];
        fptype step       = pc.getParameter(4 + i * 4 + 2); // functorConstants[indices[lowerBoundIdx + 1]];

        currVariable -= lowerBound;
        currVariable /= step;

        auto localBin   = static_cast<int>(floor(currVariable));
        binDistances[i] = currVariable - localBin - fptype(0.5);
        globalBin += previous * localBin;
        previous *= pc.getConstant(lowerBoundIdx + 1); // indices[lowerBoundIdx + 2];

        if(0 == THREADIDX + BLOCKIDX)
            printf("Variable %i: %f %f %i\n", i, currVariable, currVariable * step + lowerBound, localBin);
    }

    fptype *myHistogram = dev_base_interhists[myHistogramIndex];
    fptype ret          = 0;

    //------------------
    //     |     |     |
    //  3  |  4  |  5  |
    //     |     |     |
    //------------------
    //    x|     |     |
    //  0  |  1  |  2  |
    //     |     |     |
    //------------------

    fptype totalWeight = 0;
    int totalBins      = dev_powi(3, numVars);

    for(int i = 0; i < totalBins; ++i) {
        int currBin          = globalBin;
        int localPrevious    = 1;
        int trackingBin      = globalBin;
        bool offSomeAxis     = false;
        fptype currentWeight = 0;

        // Loop over vars to get offset for each one.
        for(int v = 0; v < numVars; ++v) {
            int localNumBins = pc.parameters[pc.parameterIdx + 1 + v]; // indices[4 * (v + 1) + 1];
            int offset       = ((i / dev_powi(3, v)) % 3) - 1;

            currBin += offset * localPrevious;
            localPrevious *= localNumBins;

            int currVarBin = trackingBin % localNumBins;
            trackingBin /= localNumBins;

            if(currVarBin + offset < 0)
                offSomeAxis = true;

            if(currVarBin + offset >= localNumBins)
                offSomeAxis = true;

            fptype currDist = binDistances[v];
            currDist -= offset;
            currentWeight += currDist * currDist;

            if(0 == THREADIDX + BLOCKIDX)
                printf("%i, %i: %f %f %f %i %s\n",
                       i,
                       v,
                       currDist,
                       binDistances[v],
                       currentWeight,
                       offset,
                       offSomeAxis ? "off" : "on");
        }

        // Only interpolate the four closest boxes (in two dimensions; more in three dimensions).
        currentWeight       = currentWeight > 0
                                  ? (currentWeight <= sqrt(static_cast<fptype>(numVars)) ? 1 / sqrt(currentWeight) : 0)
                                  : 0;
        fptype currentEntry = offSomeAxis ? 0 : myHistogram[currBin];
        ret += currentWeight * currentEntry;
        totalWeight += currentWeight;

        if(0 == THREADIDX + BLOCKIDX)
            printf(
                "Adding bin content %i %f with weight %f for total %f.\n", currBin, currentEntry, currentWeight, ret);
    }

    ret /= totalWeight;
    return ret;
}

__device__ device_function_ptr ptr_to_InterHistogram = device_InterHistogram;

__host__
InterHistPdf::InterHistPdf(std::string n, BinnedDataSet *x, std::vector<Variable> params, std::vector<Observable> obses)
    : GooPdf("InterHistPdf", n)
    , numVars(x->numVariables()) {
    static unsigned int totalHistograms = 0;
    totalEvents                         = 0;

    // push on the histogram index and number of variables.
    registerConstant(totalHistograms);
    registerConstant(numVars);

    int varIndex = 0;

    for(Observable var : x->getObservables()) {
        registerObservable(var);
        registerConstant(OBS_CODE);

        registerConstant(var.getLowerLimit());
        registerConstant(var.getBinSize());
        registerConstant(var.getNumBins());

        // NB, do not put cIndex here, it is accounted for by the offset in MEMCPY_TO_SYMBOL below.
        varIndex++;
    }

    unsigned int numbins = x->getNumBins();
    thrust::host_vector<fptype> host_histogram;

    for(unsigned int i = 0; i < numbins; ++i) {
        fptype curr = x->getBinContent(i);
        host_histogram.push_back(curr);
        totalEvents += curr;
    }

    dev_base_histogram = new thrust::device_vector<fptype>(host_histogram);
    static fptype *dev_address[1];
    dev_address[0] = (&((*dev_base_histogram)[0])).get();
    MEMCPY_TO_SYMBOL(
        dev_base_interhists, dev_address, sizeof(fptype *), totalHistograms * sizeof(fptype *), hipMemcpyHostToDevice);

    registerFunction("ptr_to_InterHistogram", ptr_to_InterHistogram);

    initialize();

    totalHistograms++;
}

} // namespace GooFit
