#include "hip/hip_runtime.h"
#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/basic/JohnsonSUPdf.h>

namespace GooFit {

const fptype SQRT2PI = 2.506628;

__device__ fptype device_JohnsonSU(fptype *evt, ParameterContainer &pc) {
    int id     = pc.getObservable(0);
    fptype _Jm = pc.getParameter(0);
    fptype _Js = pc.getParameter(1);
    fptype _Jg = pc.getParameter(2);
    fptype _Jd = pc.getParameter(3);

    // we are using index 0.  If we need a different idx, we need to pass that information along.
    fptype x = evt[id];

    pc.incrementIndex(1, 4, 0, 1, 1);

    fptype px       = (x - _Jm) / _Js;
    fptype px2      = px * px;
    fptype sqrt_arg = sqrt(1 + px2);
    fptype inv_sinh = log(px + sqrt_arg);
    fptype gaus_arg = _Jg + _Jd * inv_sinh;
    // if ((gpuDebug & 1) && (0 == BLOCKIDX) && (0 == THREADIDX))
    // if (gpuDebug & 1)
    // printf("Johnson SU: %f %f %f %f | %f %f %i\n", _Jm, _Js, _Jg, _Jd, x, _Jd / (_Js * SQRT2PI * sqrt_arg) * exp(-0.5
    // * gaus_arg * gaus_arg), indices[2 + indices[0]]);
    // printf("Johnson SU: %f %f %f %f | %f %f %f %f\n", _Jm, _Js, _Jg, _Jd, x, _Jd / (_Js * SQRT2PI * sqrt_arg) *
    // exp(-0.5 * gaus_arg * gaus_arg), hipArray[indices[1]], hipArray[indices[2]]);
    return _Jd / (_Js * SQRT2PI * sqrt_arg) * exp(-0.5 * gaus_arg * gaus_arg);
}

__device__ device_function_ptr ptr_to_JohnsonSU = device_JohnsonSU;

__host__
JohnsonSUPdf::JohnsonSUPdf(std::string n, Observable _x, Variable mean, Variable sigma, Variable gamma, Variable delta)
    : GooPdf(n, _x) {
    registerParameter(mean);
    registerParameter(sigma);
    registerParameter(gamma);
    registerParameter(delta);

    initialize();
}

__host__ void JohnsonSUPdf::recursiveSetIndices() { GOOFIT_RECURSIVE_SET_INDICIES(ptr_to_JohnsonSU); }

__host__ fptype JohnsonSUPdf::integrate(fptype lo, fptype hi) const {
    return 1.0; // Analytic integral included in device function! (Correct for minus to plus inf.)
}
} // namespace GooFit
