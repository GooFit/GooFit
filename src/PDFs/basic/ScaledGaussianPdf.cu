#include "hip/hip_runtime.h"
#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/basic/ScaledGaussianPdf.h>
#include <goofit/Variable.h>

//#include <limits>

namespace GooFit {

__device__ auto device_ScaledGaussian(fptype *evt, ParameterContainer &pc) -> fptype {
    int id = pc.getObservable(0);

    fptype x     = RO_CACHE(evt[id]);
    fptype mean  = pc.getParameter(0) + pc.getParameter(2);
    fptype sigma = pc.getParameter(1) * (1 + pc.getParameter(3));
    fptype ret   = exp(-0.5 * (x - mean) * (x - mean) / (sigma * sigma));

    pc.incrementIndex(1, 4, 0, 1, 1);

    return ret;
}

__device__ device_function_ptr ptr_to_ScaledGaussian = device_ScaledGaussian;

__host__ ScaledGaussianPdf::ScaledGaussianPdf(
    std::string n, Observable _x, Variable mean, Variable sigma, Variable delta, Variable epsilon)
    : GooPdf("ScaledGaussianPdf", n, _x, mean, sigma, delta, epsilon) {
    registerFunction("ptr_to_ScaledGaussian", ptr_to_ScaledGaussian);

    initialize();
}

} // namespace GooFit
