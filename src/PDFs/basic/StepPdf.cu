#include <goofit/PDFs/basic/StepPdf.h>

namespace GooFit {

__device__ fptype device_Step(fptype *evt, fptype *p, unsigned int *indices) {
    fptype x  = evt[indices[2 + indices[0]]];
    fptype x0 = p[indices[1]];
    return (x > x0 ? 1 : 0);
}

__device__ device_function_ptr ptr_to_Step = device_Step;
device_function_ptr hptr_to_Step           = device_Step;

__host__ StepPdf::StepPdf(std::string n, Observable _x, Variable x0)
    : GooPdf(n, _x) {
    std::vector<unsigned int> pindices;
    pindices.push_back(registerParameter(x0));
    GET_FUNCTION_ADDR(ptr_to_Step);
    initialize(pindices);
}

__host__ fptype StepPdf::integrate(fptype lo, fptype hi) const {
    unsigned int *indices = host_indices + parameters;
    fptype x0             = host_params[indices[1]];
    return (hi - x0);
}

} // namespace GooFit
