#include "hip/hip_runtime.h"
#include <goofit/PDFs/basic/TrigThresholdPdf.h>

namespace GooFit {

__device__ fptype threshCalc(fptype distance, fptype linConst) {
    fptype ret
        = (distance > fptype(0.5) ? fptype(1) : (linConst + (1 - linConst) * sin(distance * fptype(3.14159265))));
    return ret;
}

__device__ fptype device_TrigThresholdUpper(fptype *evt, fptype *p, unsigned int *indices) {
    fptype x         = evt[indices[2 + indices[0]]];
    fptype thresh    = p[indices[1]];
    fptype trigConst = p[indices[2]];
    fptype linConst  = p[indices[3]];

    trigConst *= (thresh - x);
    return threshCalc(trigConst, linConst);
}

__device__ fptype device_TrigThresholdLower(fptype *evt, fptype *p, unsigned int *indices) {
    fptype x         = evt[indices[2 + indices[0]]];
    fptype thresh    = p[indices[1]];
    fptype trigConst = p[indices[2]];
    fptype linConst  = p[indices[3]];

    trigConst *= (x - thresh);
    return threshCalc(trigConst, linConst);
}

__device__ fptype device_VerySpecialEpisodeTrigThresholdUpper(fptype *evt, fptype *p, unsigned int *indices) {
    // Annoying special case for use with Mikhail's efficiency function across the Dalitz plot

    fptype x = evt[indices[2 + indices[0] + 0]];
    fptype y = evt[indices[2 + indices[0] + 1]];

    fptype thresh    = p[indices[1]];
    fptype trigConst = p[indices[2]];
    fptype linConst  = p[indices[3]];
    fptype z         = p[indices[4]] - x - y;

    trigConst *= (thresh - z);
    return threshCalc(trigConst, linConst);
}

__device__ fptype device_VerySpecialEpisodeTrigThresholdLower(fptype *evt, fptype *p, unsigned int *indices) {
    fptype x = evt[indices[2 + indices[0] + 0]];
    fptype y = evt[indices[2 + indices[0] + 1]];

    fptype thresh    = p[indices[1]];
    fptype trigConst = p[indices[2]];
    fptype linConst  = p[indices[3]];
    fptype z         = p[indices[4]] - x - y;

    trigConst *= (z - thresh);
    fptype ret = threshCalc(trigConst, linConst);

    // if ((1 > (int) floor(0.5 + evt[8])) && (gpuDebug & 1) && (paramIndices + debugParamIndex == indices))
    // printf("TrigThreshold: (%f - %f = %f) -> %f %f\n", z, thresh, trigConst, linConst, ret);

    return ret;
}

__device__ device_function_ptr ptr_to_TrigThresholdUpper = device_TrigThresholdUpper;
__device__ device_function_ptr ptr_to_TrigThresholdLower = device_TrigThresholdLower;
__device__ device_function_ptr ptr_to_VerySpecialEpisodeTrigThresholdUpper
    = device_VerySpecialEpisodeTrigThresholdUpper;
__device__ device_function_ptr ptr_to_VerySpecialEpisodeTrigThresholdLower
    = device_VerySpecialEpisodeTrigThresholdLower;

__host__ TrigThresholdPdf::TrigThresholdPdf(
    std::string n, Observable _x, Variable thresh, Variable trigConst, Variable linConst, bool upper)
    : GooPdf(n, _x) {
    std::vector<unsigned int> pindices;
    pindices.push_back(registerParameter(thresh));
    pindices.push_back(registerParameter(trigConst));
    pindices.push_back(registerParameter(linConst));

    if(upper) {
        GET_FUNCTION_ADDR(ptr_to_TrigThresholdUpper);
    } else {
        GET_FUNCTION_ADDR(ptr_to_TrigThresholdLower);
    }

    initialize(pindices);
}

__host__ TrigThresholdPdf::TrigThresholdPdf(std::string n,
                                            Observable _x,
                                            Observable _y,
                                            Variable thresh,
                                            Variable trigConst,
                                            Variable linConst,
                                            Variable massConstant,
                                            bool upper)
    : GooPdf(n, _x, _y) {
    std::vector<unsigned int> pindices;
    pindices.push_back(registerParameter(thresh));
    pindices.push_back(registerParameter(trigConst));
    pindices.push_back(registerParameter(linConst));
    pindices.push_back(registerParameter(massConstant));

    if(upper) {
        GET_FUNCTION_ADDR(ptr_to_VerySpecialEpisodeTrigThresholdUpper);
    } else {
        GET_FUNCTION_ADDR(ptr_to_VerySpecialEpisodeTrigThresholdLower);
    }

    initialize(pindices);
}
} // namespace GooFit
