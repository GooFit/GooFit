#include "hip/hip_runtime.h"
#include <goofit/Error.h>
#include <goofit/Log.h>
#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/combine/AddPdf.h>
#include <goofit/detail/ThrustOverride.h>

#include <thrust/iterator/constant_iterator.h>
#include <thrust/transform_reduce.h>

#ifdef GOOFIT_MPI
#include <mpi.h>
#endif

namespace GooFit {

__device__ fptype device_AddPdfs(fptype *evt, ParameterContainer &pc) {
    int numParameters  = pc.getNumParameters();
    fptype ret         = 0;
    fptype totalWeight = 0;

    // make a copy of our parameter container
    ParameterContainer pci = pc;

    // We only call increment once we read our weight/norm for the first iteration.
    pci.incrementIndex();

    for(int i = 0; i < numParameters; i++) {
        // fetch our values from AddPdf
        fptype weight = pc.getParameter(i);
        totalWeight += weight;

        // This is the normal value for the 'callFunction' PDF, so we read from pci
        fptype norm = pci.getNormalization(0);

        // call the first function to add in our PDF.
        fptype curr = callFunction(evt, pci);

        ret += weight * curr * norm;
    }

    // restore our new parameter container object
    pc = pci;

    // previous functions incremented the indices appropriately, so now we need to get the norm again
    // NOTE: this is the weight for the function about to be called.
    fptype normFactor = pc.getNormalization(0);

    fptype last = callFunction(evt, pc);
    ret += (1 - totalWeight) * last * normFactor;

    return ret;
}

__device__ fptype device_AddPdfsExt(fptype *evt, ParameterContainer &pc) {
    int numParameters  = pc.getNumParameters();
    fptype ret         = 0;
    fptype totalWeight = 0;

    // make a copy of our parameter container
    ParameterContainer pci = pc;

    // We only call increment once we read our weight/norm for the first iteration.
    pci.incrementIndex();

    for(int i = 0; i < numParameters; i++) {
        // grab the weight value
        fptype weight     = pci.getParameter(i);
        fptype normFactor = pci.getNormalization(0);

        fptype curr = callFunction(evt, pci);
        ret += weight * curr * normFactor;

        totalWeight += weight;
    }

    pc = pci;
    ret /= totalWeight;

    return ret;
}

__device__ device_function_ptr ptr_to_AddPdfs    = device_AddPdfs;
__device__ device_function_ptr ptr_to_AddPdfsExt = device_AddPdfsExt;

AddPdf::AddPdf(std::string n, std::vector<Variable> weights, std::vector<PdfBase *> comps)
    : GooPdf(n)
    , extended(true) {
    if(weights.size() != comps.size() && (weights.size() + 1) != comps.size())
        throw GooFit::GeneralError("Size of weights {} (+1) != comps {}", weights.size(), comps.size());

    // Indices stores (function index)(function parameter index)(weight index) triplet for each component.
    // Last component has no weight index unless function is extended.
    for(PdfBase *p : comps) {
        components.push_back(p);
        if(components.back() == nullptr)
            throw GooFit::GeneralError("Invalid component");
    }

    observablesList = getObservables();

    for(unsigned int w = 0; w < weights.size(); ++w) {
        if(components[w] == nullptr)
            throw GooFit::GeneralError("Invalid component");
        registerParameter(weights[w]);
    }

    if(components.back() == nullptr)
        throw GooFit::GeneralError("Invalid component");

    if(weights.size() < components.size()) {
        extended = false;
    }

    if(extended)
        registerFunction("ptr_to_AddPdfsExt", ptr_to_AddPdfsExt);
    else
        registerFunction("ptr_to_AddPdfs", ptr_to_AddPdfs);

    initialize();
}

AddPdf::AddPdf(std::string n, Variable frac1, PdfBase *func1, PdfBase *func2)
    : GooPdf(n)
    , extended(false) {
    // Special-case constructor for common case of adding two functions.
    components.push_back(func1);
    components.push_back(func2);

    observablesList = getObservables();

    registerParameter(frac1);

    registerFunction("ptr_to_AddPdfs", ptr_to_AddPdfs);

    initialize();
}

__host__ fptype AddPdf::normalize() {
    // if (cpuDebug & 1) std::cout << "Normalizing AddPdf " << getName() << std::endl;

    fptype ret         = 0;
    fptype totalWeight = 0;

    for(unsigned int i = 0; i < components.size() - 1; ++i) {
        // fptype weight = host_parameters[parametersIdx + 3*i + 1];
        fptype weight = parametersList[i].getValue();
        totalWeight += weight;
        fptype curr = components[i]->normalize();
        ret += curr * weight;
    }

    fptype last = components.back()->normalize();

    if(extended) {
        fptype lastWeight = host_parameters[parametersIdx + 2];
        totalWeight += lastWeight;
        ret += last * lastWeight;
        ret /= totalWeight;
    } else {
        ret += (1 - totalWeight) * last;
    }

    host_normalizations[normalIdx + 1] = 1.0;
    cachedNormalization                = 1.0;

    // TODO: Unsure of the exact location for this normalize...
    if(getSpecialMask() & PdfBase::ForceCommonNorm) {
        // Want to normalize this as
        // (f1 A + (1-f1) B) / int (f1 A + (1-f1) B)
        // instead of default
        // (f1 A / int A) + ((1-f1) B / int B).

        for(auto component : components) {
            host_normalizations[component->getParameterIndex()] = (1.0 / ret);
            // component->cachedNormalization = 1.0 / ret;
        }
    }

    // if (cpuDebug & 1) std::cout << getName() << " integral returning " << ret << std::endl;
    return ret;
}

__host__ double AddPdf::calculateNLL() {
    double ret = GooPdf::calculateNLL() / 2.0;

    if(extended) {
        fptype expEvents = 0;

        for(unsigned int i = 0; i < components.size(); ++i) {
            expEvents += host_parameters[parametersIdx + 3 * (i + 1)];
        }

        // Log-likelihood of numEvents with expectation of exp is (-exp + numEvents*ln(exp) - ln(numEvents!)).
        // The last is constant, so we drop it; and then multiply by minus one to get the negative log-likelihood.
        ret += (expEvents - numEvents * log(expEvents));
    }

    return ret * 2.0;
}
} // namespace GooFit
