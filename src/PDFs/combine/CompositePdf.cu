#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/combine/CompositePdf.h>

namespace GooFit {

__device__ fptype device_Composite(fptype *evt, ParameterContainer &pc) {
    // unsigned int coreFcnIndex  = RO_CACHE(indices[1]);
    // unsigned int coreParIndex  = RO_CACHE(indices[2]);
    // unsigned int shellFcnIndex = RO_CACHE(indices[3]);
    // unsigned int shellParIndex = RO_CACHE(indices[4]);
    pc.incrementIndex();

    // NB, not normalizing core function, it is not being used as a PDF.
    // fptype coreValue = (*(reinterpret_cast<device_function_ptr>(d_function_table[coreFcnIndex])))(evt,
    // hipArray, paramIndices+coreParIndex);
    fptype coreValue = callFunction(evt, pc);

    // unsigned int numShellPars  = pc.parameters[pc.parameterIdx];
    // unsigned int shellObsIndex = pc.parameters[pc.parameterIdx + 2];

    // int obs = pc.constants[pc.constantIdx + 1];
    int id = pc.getObservable(0);

    auto *fakeEvt = new fptype[10]; // Allow plenty of space in case events are large.
    fakeEvt[id]   = coreValue;

    // Don't normalize shell either, since we don't know what composite function is being used for.
    // It may not be a PDF. Normalizing at this stage would be presumptuous.
    // fptype ret = (*(reinterpret_cast<device_function_ptr>(d_function_table[shellFcnIndex])))(fakeEvt, hipArray,
    // shellParams);
    fptype ret = callFunction(fakeEvt, pc);
    delete[] fakeEvt;

    // if (0 == THREADIDX)
    // printf("Composite: %f %f %f %f %f %f\n", evt[4], evt[5], evt[6], evt[7], coreValue, ret);

    return ret;
}

__device__ device_function_ptr ptr_to_Composite = device_Composite;

__host__ CompositePdf::CompositePdf(std::string n, PdfBase *core, PdfBase *shell)
    : CombinePdf("CompositePdf", n) {
    // Add as components so that observables and parameters will be registered.
    components.push_back(core);
    components.push_back(shell);

    observablesList = getObservables();

    registerFunction("ptr_to_Composite", ptr_to_Composite);

    initialize();
}

__host__ fptype CompositePdf::normalize() {
    recursiveSetNormalization(1.0);

    // Note: Core is not normalized in composite calculation,
    // because it is not a PDF,
    // it is just a plain old function;
    // it can take any value.
    // Shell needn't be normalized either,
    // because we don't know that the composite
    // will be used as a PDF; and if it is, the
    // normalization should be applied at the level
    // of whatever calls the composite.
    // However: These functions may appear elsewhere
    // in the full function, and perhaps need to
    // be normalized there. Consequently, we
    // normalize them even though the information
    // may not be used.

    for(auto component : components) {
        component->normalize();
    }

    // Normalize composite in the usual binned-integral way.
    return GooPdf::normalize();
}
} // namespace GooFit
