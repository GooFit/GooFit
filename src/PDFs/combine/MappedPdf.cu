#include "hip/hip_runtime.h"
#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/combine/MappedPdf.h>

namespace GooFit {

__device__ auto device_Mapped(fptype *evt, ParameterContainer &pc) -> fptype {
    // Structure : nP mapFunctionIndex mapParamIndex functionIndex1 parameterIndex1 functionIndex2 parameterIndex2 ...
    // Find mapping between event variables and function to evaluate
    auto numConstants = pc.getNumConstants();

    auto pc_mapped = pc;

    // Mapping PDF happens directly after, so just increment.
    pc.incrementIndex();
    auto targetFunction = static_cast<int>(floor(0.5 + callFunction(evt, pc)));
    // increment until target function
    int indicestoskip = 0;
    for(int i = 0; i < targetFunction; i++) {
        indicestoskip += (int)pc_mapped.getConstant(1 + i);
    }

    auto cur_funcIdx    = pc.funcIdx;
    auto target_funcIdx = cur_funcIdx + indicestoskip;
    while(pc.funcIdx < target_funcIdx)
        pc.incrementIndex();
    fptype norm = pc.getNormalization(0);
    fptype ret  = callFunction(evt, pc);
    ret *= norm;
    int finalIndex = cur_funcIdx;
    // now need to increase index until end
    for(int i = 1; i < numConstants; i++) {
        finalIndex += (int)pc_mapped.getConstant(i);
    }

    while(pc.funcIdx < finalIndex)
        pc.incrementIndex();
    return ret;
}

__device__ device_function_ptr ptr_to_Mapped = device_Mapped;

__host__ auto countComponents(PdfBase *func) -> int {
    auto subcomponents = func->getComponents();
    int n_components   = 0;
    if(subcomponents.size() > 0) {
        for(auto subcomponent : subcomponents) {
            n_components++;
            n_components += countComponents(subcomponent);
        }
    }

    return n_components;
}

__host__ MappedPdf::MappedPdf(std::string n, GooPdf *m, std::vector<GooPdf *> &t)
    : CombinePdf("MappedPdf", n) {
    components.push_back(m);

    std::vector<int> nComponents;

    for(GooPdf *f : t) {
        components.push_back(f);

        // count number of subfunctions
        int n_components = countComponents(f);
        // also count total function
        n_components++;
        nComponents.push_back(n_components);
    }

    // This makes sure we have the appropriate amount of obs in our structure
    observablesList = getObservables();

    // add a constant value for the number of 't' functions, skipping 'm'.
    registerConstant(components.size() - 1);
    for(auto nComponent : nComponents)
        registerConstant(nComponent);

    registerFunction("ptr_to_Mapped", ptr_to_Mapped);

    initialize();
}

__host__ auto MappedPdf::normalize() -> fptype {
    fptype ret = 0;

    for(unsigned int i = 1; i < components.size(); ++i) { // No need to normalize mapping function.
        fptype curr = components[i]->normalize();
        ret += curr;
    }

    host_normalizations[normalIdx + 1] = 1.0;
    cachedNormalization                = 1.0;

    return ret;
}
} // namespace GooFit
