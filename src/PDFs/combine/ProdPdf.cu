#include "hip/hip_runtime.h"
#include <goofit/Log.h>
#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/combine/ProdPdf.h>

#include <algorithm>

namespace GooFit {

__device__ fptype device_ProdPdfs(fptype *evt, ParameterContainer &pc) {
    int numCons  = pc.getNumConstants();
    int numComps = pc.getConstant(0);
    int numObs   = pc.getNumObservables();
    fptype ret   = 1;

    pc.incrementIndex(1, 0, numCons, numObs, 1);
    // pc.incrementIndex();
    for(int i = 0; i < numComps; i++) {
        fptype norm = pc.getNormalization(0);
        fptype curr = callFunction(evt, pc);

        curr *= norm;
        ret *= curr;
    }

    return ret;
}

__device__ device_function_ptr ptr_to_ProdPdfs = device_ProdPdfs;

ProdPdf::ProdPdf(std::string n, std::vector<PdfBase *> comps)
    : CombinePdf("ProdPdf", n)
    , varOverlaps(false) {
    for(PdfBase *p : comps) {
        components.push_back(p);
        // we push a placeholder that is used to indicate
        // constantsList.push_back (0);
    }

    observablesList = getObservables(); // Gathers from components

    // Add that we have a components size
    registerConstant(components.size());

    std::vector<Observable> observableCheck; // Use to check for overlap in observables

    for(PdfBase *p : comps) {
        if(varOverlaps)
            continue; // Only need to establish this once.

        std::vector<Observable> currObses = p->getObservables();

        for(Observable &o : currObses) {
            if(find(observableCheck.begin(), observableCheck.end(), o) == observableCheck.end())
                continue;

            varOverlaps = true;
            break;
        }

        observableCheck = p->getObservables();
    }

    if(varOverlaps) { // Check for components forcing separate normalization
        for(PdfBase *p : comps) {
            if(p->getSeparateNorm())
                varOverlaps = false;
        }
    }

    registerFunction("ptr_to_ProdPdfs", ptr_to_ProdPdfs);

    initialize();
}

__host__ fptype ProdPdf::normalize() {
    if(varOverlaps) {
        // Two or more components share an observable and cannot be separately
        // normalized, since \int A*B dx does not equal int A dx * int B dx.
        recursiveSetNormalization(1.0);
        host_normalizations.sync(d_normalizations);

        // Normalize numerically.
        // std::cout << "Numerical normalization of " << getName() << " due to varOverlaps.\n";
        fptype ret = GooPdf::normalize();
        // if (cpuDebug & 1)
        // std::cout << "ProdPdf " << getName() << " has normalization " << ret << " " << host_callnumber << std::endl;
        return ret;
    }

    // Normalize components individually
    for(PdfBase *c : components) {
        c->normalize();
    }

    host_normalizations.at(normalIdx + 1) = 1.0;
    cachedNormalization                   = 1.0;

    return 1.0;
}
} // namespace GooFit
