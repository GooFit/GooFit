#include "hip/hip_runtime.h"
#include <mcbooster/Evaluate.h>
#include <mcbooster/EvaluateArray.h>
#include <mcbooster/GContainers.h>
#include <mcbooster/GFunctional.h>
#include <mcbooster/GTypes.h>
#include <mcbooster/Generate.h>
#include <mcbooster/Vector4R.h>

#include <goofit/Error.h>
#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/physics/Amp3BodySqDP.h>
#include <goofit/PDFs/physics/Amp3BodyBase.h>
#include <goofit/PDFs/physics/detail/Dim2.h>
#include <goofit/PDFs/physics/detail/SpecialSqDpResonanceCalculator.h>
#include <goofit/PDFs/physics/detail/SpecialSqDpResonanceIntegrator.h>
#include <goofit/PDFs/physics/resonances/Resonance.h>
#include <goofit/detail/Complex.h>

#include <thrust/copy.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/system/omp/execution_policy.h>
#include <thrust/system/cpp/execution_policy.h>
#include <thrust/system/cuda/execution_policy.h>
#include <array>
#include <vector>



namespace GooFit {


__host__ __device__   auto inSqDalitz(const fptype &mprime,const fptype &thetaprime) -> bool{
    return (mprime>0.0 && mprime<1.0)&&(thetaprime>0.0 && thetaprime<1.0);
}

__host__ __device__  auto calc_mprime(const fptype &m12, const fptype &m_mother, const fptype &m1, const fptype &m2, const fptype &m3)->fptype{
    fptype min = m1+m2;
    fptype max = m_mother-m3;
    fptype mprime = (2*(m12 - min)/(max-min)) - 1.0;

    if(mprime<-1.)
        mprime=-1.;
    
    if(mprime>1.)
        mprime=1.;

    return acos(mprime)/M_PI;
}

__host__ __device__  auto calc_thetaprime(const fptype &m12,const fptype &m13, const fptype &m_mother, const fptype &m1, const fptype &m2, const fptype &m3)->fptype{
    fptype m13Sq = m13*m13;
    fptype m12Sq = m12*m12;
    fptype m_motherSq = m_mother*m_mother;
    fptype m1Sq = m1*m1;
    fptype m2Sq = m2*m2;
    fptype m3Sq = m3*m3;

    fptype EiCmsij = (m12Sq - m2Sq + m1Sq)/(2.0*m12);
    fptype EkCmsij = (m_motherSq - m12Sq - m3Sq)/(2.0*m12);

    //printf("EiCmsij=%f \t EkCmsij=%f \n",m12, m13);

    fptype qi = EiCmsij*EiCmsij - m1Sq;
    qi = qi>0. ? sqrt(qi) : 0.;

    fptype qk = EkCmsij*EkCmsij - m3Sq;
    qk = qk>0. ?  sqrt(qk) : 0.;
    
    fptype coshel = (m13Sq - m1Sq - m3Sq - 2.0*EiCmsij*EkCmsij)/(2.0*qi*qk);

    if(coshel<-1.)
        coshel=-1.;
    
    if(coshel>1.)
        coshel=1.;

    fptype thetaprime = acos(coshel)/M_PI;

    // if(thetaprime>0.5)
    //     thetaprime=1.0-thetaprime;

    return thetaprime;
}

__host__ __device__   auto calc_m12(const fptype &mprime, const fptype &m_mother, const fptype &m1, const fptype &m2, const fptype &m3)->fptype{

    fptype m12 = 0.5*( (m_mother-m3) - (m1+m2) )*(1.0 + cos(M_PI*mprime)) + (m1+m2);

    return m12;
}

__host__ __device__   auto calc_m13(const fptype &m12, const fptype &cos_12, const fptype &m_mother, const fptype &m1, const fptype &m2, const fptype &m3)->fptype{
    
    fptype m12Sq = m12*m12;
    fptype m_motherSq = m_mother*m_mother;
    fptype m1Sq = m1*m1;
    fptype m2Sq = m2*m2;
    fptype m3Sq = m3*m3;

    fptype EiCmsij = (m12Sq - m2Sq + m1Sq)/(2.0*m12);
    fptype EkCmsij = (m_motherSq - m12Sq - m3Sq)/(2.0*m12);

    fptype qi = EiCmsij*EiCmsij - m1Sq;
    qi = qi>0. ? sqrt(qi) : 0.;

    fptype qk = EkCmsij*EkCmsij - m3Sq;
    qk = qk>0. ? sqrt(qk)  : 0.;

    //printf("coshel = %.2f \n",cos_12);
    
    fptype m13Sq = m1Sq + m3Sq + 2.0*EiCmsij*EkCmsij - 2.0*qi*qk*cos_12;

    return sqrt(m13Sq);
}


__host__ __device__  auto calc_SqDp_Jacobian(const fptype &mprime ,const fptype &thetaprime, const fptype &m_mother, const fptype &m1, const fptype &m2, const fptype &m3)->fptype{

    fptype m12 = calc_m12(mprime,m_mother,m1,m2,m3);
    fptype m12Sq = m12*m12;

    fptype m_motherSq = m_mother*m_mother;
    fptype m1Sq = m1*m1;
    fptype m2Sq = m2*m2;
    fptype m3Sq = m3*m3;

    fptype EiCmsij = (m12Sq - m2Sq + m1Sq)/(2.0*m12);
    fptype EkCmsij = (m_motherSq - m12Sq - m3Sq)/(2.0*m12);
    
    fptype qi = EiCmsij*EiCmsij - m1Sq;
    qi = qi>0. ? sqrt(qi) : 0.;

    fptype qk = EkCmsij*EkCmsij - m3Sq;
     qk = qk>0. ? sqrt(qk)  : 0.;
    
    fptype deriv1 = 0.5*M_PI*((m_mother-m3) - (m1+m2))*sin(M_PI*mprime);
    fptype deriv2 = M_PI*sin(M_PI*thetaprime);

    fptype jacobian = 4.0*qi*qk*m12*deriv1*deriv2;

    return jacobian;
}

struct prg
{
    fptype a, b;

    __host__ __device__
    prg(fptype _a=0., fptype _b=1.) : a(_a), b(_b) {};

    __host__ __device__
        fptype operator()(const unsigned int n) const
        {
            thrust::default_random_engine rng(n);
            thrust::uniform_real_distribution<fptype> dist(a, b);
            rng.discard(n);

            return dist(rng);
        }
};



__host__  void genNormFakeEvents(size_t n){
   
    thrust::device_vector<fptype> mprime(n);
    thrust::device_vector<fptype> thprime(n);

    thrust::counting_iterator<unsigned int> index_sequence(0);    

    thrust::transform(index_sequence,
            index_sequence + n,
            mprime.begin(),
            prg(0.,1.));

    thrust::counting_iterator<unsigned int> index_sequence2(324234234);
    thrust::transform(index_sequence2,
            index_sequence2 + n,
            thprime.begin(),
            prg(0.,1.));

    mcbooster::VariableSet_d GooVarSet_d(2);
    GooVarSet_d[0] = &mprime;
    GooVarSet_d[1] = &thprime;

    auto DS = new mcbooster::RealVector_d(3 * n);

    thrust::counting_iterator<unsigned int> eventNumber(0);

#pragma unroll
    for(int i = 0; i < 2; ++i) {
        mcbooster::strided_range<mcbooster::RealVector_d::iterator> sr(DS->begin() + i, DS->end(), 3);
        thrust::copy(GooVarSet_d[i]->begin(), GooVarSet_d[i]->end(), sr.begin());
    }

    mcbooster::strided_range<mcbooster::RealVector_d::iterator> sr(DS->begin() + 2, DS->end(), 3);
    thrust::copy(eventNumber, eventNumber + n, sr.begin());

    dev_fake_event_array = thrust::raw_pointer_cast(DS->data());
   

    
}



// Functor used for fit fraction sum
struct CoefSumFunctor {
    fpcomplex coef_i;
    fpcomplex coef_j;

    CoefSumFunctor(fpcomplex coef_i, fpcomplex coef_j)
        : coef_i(coef_i)
        , coef_j(coef_j) {}

    __device__ auto operator()(thrust::tuple<fpcomplex, fpcomplex> val) -> fptype {
        return (coef_i * thrust::conj<fptype>(coef_j) * thrust::get<0>(val) * thrust::conj<fptype>(thrust::get<1>(val)))
            .real();
    }
};

constexpr int resonanceOffset_DP = 4; // Offset of the first resonance into the parameter index array
// Offset is number of parameters, constant index, number of resonances (not calculable
// from nP because we don't know what the efficiency might need), and cache index. Efficiency
// parameters are after the resonance information.

// The function of this array is to hold all the cached waves; specific
// waves are recalculated when the corresponding resonance mass or width
// changes. Note that in a multithread environment each thread needs its
// own cache, hence the '10'. Ten threads should be enough for anyone!

// NOTE: This is does not support ten instances (ten threads) of resoncances now, only one set of resonances.
// this needs to be large enough to hold all samples
__device__ fpcomplex *cSqDpResonances[16 * 20];
fptype cResonancesIntegrals[20][16];

__device__ inline auto parIndexFromResIndex_DP(int resIndex) -> int {
    return resonanceOffset_DP + resIndex * resonanceSize;
}


__device__ auto device_SqDalitzPlot(fptype *evt, ParameterContainer &pc) -> fptype {
    int num_obs = pc.getNumObservables();
    int id_mprime  = pc.getObservable(0);
    int id_thetaprime  = pc.getObservable(1);
    int id_num  = pc.getObservable(2);

    fptype mprime = RO_CACHE(evt[id_mprime]);
    fptype thetaprime = RO_CACHE(evt[id_thetaprime]);

    unsigned int numResonances = pc.getConstant(0);
    unsigned int cacheToUse    = pc.getConstant(1);

    if(!inSqDalitz(mprime, thetaprime)) {
        pc.incrementIndex(1, numResonances * 2, 2, num_obs, 1);

        // loop over resonances and efficiency functions
        for(int i = 0; i < numResonances; i++)
            pc.incrementIndex();

        // increment the efficiency function
        pc.incrementIndex();
        return 0;
    }

    fptype evtIndex = RO_CACHE(evt[id_num]);

    auto evtNum = static_cast<int>(floor(0.5 + evtIndex));

    fpcomplex totalAmp(0, 0);

    for(int i = 0; i < numResonances; ++i) {
        for(int j = 0; j < numResonances; ++j) {
            fpcomplex amp_i = fpcomplex(pc.getParameter(i * 2), pc.getParameter(i * 2 + 1));
            fpcomplex me_i = RO_CACHE(cSqDpResonances[i + (16 * cacheToUse)][evtNum]);
            fpcomplex amp_j = fpcomplex(pc.getParameter(j * 2), -pc.getParameter(j * 2 + 1));
            fpcomplex me_j = RO_CACHE(cSqDpResonances[j + (16 * cacheToUse)][evtNum]);
            totalAmp += amp_i*amp_j*me_i*conj(me_j);
        }
    }

    fptype ret = thrust::abs(totalAmp);


    // for(int i = 0; i < numResonances; ++i) {
    //         fpcomplex amp_i = fpcomplex(pc.getParameter(i * 2), pc.getParameter(i * 2 + 1));
    //         fpcomplex me_i = RO_CACHE(cSqDpResonances[i + (16 * cacheToUse)][evtNum]);
    //         totalAmp += amp_i*me_i;
    // }

    // fptype ret = thrust::norm(totalAmp);
   
    pc.incrementIndex(1, numResonances * 2, 2, num_obs, 1);

    // loop to efficiency idx
    for(int i = 0; i < numResonances; i++)
        pc.incrementIndex();

    fptype eff = callFunction(evt, pc);
  
    //fptype jacobian = calc_SqDp_Jacobian(mprime, thetaprime, c_motherMass, c_daug1Mass, c_daug2Mass, c_daug3Mass);
    ret *= eff;

    // printf("likelihood=%f eff=%f \n",ret, eff);

    return ret;
}

int Amp3BodySqDP::cacheCount                         = 0;
__device__ device_function_ptr ptr_to_SqDalitzPlot = device_SqDalitzPlot;

__host__ Amp3BodySqDP::Amp3BodySqDP(
    std::string n, Observable mprime, Observable thetaprime, EventNumber eventNumber, DecayInfo3 decay, GooPdf *efficiency)
    : Amp3BodyBase("Amp3BodySqDP", n, mprime, thetaprime, eventNumber)
    , decayInfo(decay)
    , _mprime(mprime)
    , _thetaprime(thetaprime)
    , _eventNumber(eventNumber)
    , dalitzNormRange(nullptr)
    //, cachedWaves(0)
    , integrals(nullptr)
    , integrals_ff(nullptr)
    , NumNormEvents(decay.SetSizeNormSample)
    , forceRedoIntegrals(true)
    , totalEventSize(3) // Default 3 = mprime, thetaprime, evtNum
    , cacheToUse(0)
    , integrators(nullptr)
    , integrators_ff(nullptr)
    , calculators(nullptr) {
    for(auto &cachedWave : cachedWaves)
        cachedWave = nullptr;

    // Passing values to the defined constants.  Rather than push into list, which means each resonance
    MEMCPY_TO_SYMBOL(c_motherMass, &decay.motherMass, sizeof(fptype), 0, hipMemcpyHostToDevice);
    MEMCPY_TO_SYMBOL(c_daug1Mass, &decay.daug1Mass, sizeof(fptype), 0, hipMemcpyHostToDevice);
    MEMCPY_TO_SYMBOL(c_daug2Mass, &decay.daug2Mass, sizeof(fptype), 0, hipMemcpyHostToDevice);
    MEMCPY_TO_SYMBOL(c_daug3Mass, &decay.daug3Mass, sizeof(fptype), 0, hipMemcpyHostToDevice);
    MEMCPY_TO_SYMBOL(c_meson_radius, &decay.meson_radius, sizeof(fptype), 0, hipMemcpyHostToDevice);
    MEMCPY_TO_SYMBOL(c_mother_meson_radius, &decay.mother_meson_radius, sizeof(fptype), 0, hipMemcpyHostToDevice);

    

    // registered to 0 position
    registerConstant(decayInfo.resonances.size());

    // TODO increase after registerConstant?
    cacheToUse = cacheCount++;
    // registered to 1 position
    registerConstant(cacheToUse);

    for(auto &resonance : decayInfo.resonances) {
        // registering 2 parameters
        registerParameter(resonance->amp_real);
        registerParameter(resonance->amp_imag);
        components.push_back(resonance);
    }

    components.push_back(efficiency);

    registerFunction("ptr_to_SqDalitzPlot", ptr_to_SqDalitzPlot);

    initialize();

    redoIntegral = new bool[decayInfo.resonances.size()];
    cachedMasses = new fptype[decayInfo.resonances.size()];
    cachedWidths = new fptype[decayInfo.resonances.size()];
    integrals    = new fpcomplex **[decayInfo.resonances.size()];
    integrators  = new SpecialSqDpResonanceIntegrator **[decayInfo.resonances.size()];
    integrals_ff    = new fpcomplex **[decayInfo.resonances.size()];
    integrators_ff  = new SpecialSqDpResonanceIntegrator **[decayInfo.resonances.size()];
    calculators  = new SpecialSqDpResonanceCalculator *[decayInfo.resonances.size()];

    for(int i = 0; i < decayInfo.resonances.size(); ++i) {
        redoIntegral[i] = true;
        cachedMasses[i] = -1;
        cachedWidths[i] = -1;
        integrators[i]  = new SpecialSqDpResonanceIntegrator *[decayInfo.resonances.size()];
        calculators[i]  = new SpecialSqDpResonanceCalculator(parameters, i);
        integrals[i]    = new fpcomplex *[decayInfo.resonances.size()];

        integrals_ff[i]    = new fpcomplex *[decayInfo.resonances.size()];
        integrators_ff[i]  = new SpecialSqDpResonanceIntegrator *[decayInfo.resonances.size()];

        for(int j = 0; j < decayInfo.resonances.size(); ++j) {
            integrals[i][j]   = new fpcomplex(0, 0);
            integrators[i][j] = new SpecialSqDpResonanceIntegrator(parameters, i, j);
            integrals_ff[i][j]   = new fpcomplex(0, 0);
            integrators_ff[i][j] = new SpecialSqDpResonanceIntegrator(parameters, i, j);
        }
    }

 

    
    setSeparateNorm();
    
}

void Amp3BodySqDP::populateArrays() {
    PdfBase::populateArrays();

    // save our efficiency function.  Resonance's are saved first, then the efficiency function.  Take -1 as efficiency!
    efficiencyFunction = host_function_table.size() - 1;
}
__host__ void Amp3BodySqDP::setDataSize(unsigned int dataSize, unsigned int evtSize, unsigned int offset) {
    // Default 3 is mprime, thetaprime, evtNum
    totalEventSize = evtSize;
    if(totalEventSize < 3)
        throw GooFit::GeneralError("totalEventSize {} must be 3 or more", totalEventSize);

    // if (cachedWaves) delete cachedWaves;
    if(cachedWaves[0]) {
        for(auto &cachedWave : cachedWaves) {
            delete cachedWave;
            cachedWave = nullptr;
        }
    }

    numEntries  = dataSize;
    eventOffset = offset;

    for(int i = 0; i < 16; i++) {
#ifdef GOOFIT_MPI
        cachedWaves[i] = new thrust::device_vector<fpcomplex>(m_iEventsPerTask);
#else
        cachedWaves[i] = new thrust::device_vector<fpcomplex>(dataSize);
#endif
        void *dummy = thrust::raw_pointer_cast(cachedWaves[i]->data());
        MEMCPY_TO_SYMBOL(cSqDpResonances,
                         &dummy,
                         sizeof(fpcomplex *),
                         ((16 * cacheToUse) + i) * sizeof(fpcomplex *),
                         hipMemcpyHostToDevice);
    }
  

    setForceIntegrals();

    if(dev_fake_event_array==nullptr){
        std::cout << "START: Fill normalization data array with " << NumNormEvents << " events. \n";
        genNormFakeEvents(NumNormEvents);
        hipDeviceSynchronize();
        std::cout << "END: Done! \n";
    }

}

__host__ auto Amp3BodySqDP::normalize() -> fptype {
    recursiveSetNormalization(1.0); // Not going to normalize efficiency,
    // so set normalization factor to 1 so it doesn't get multiplied by zero.
    // Copy at this time to ensure that the SpecialResonanceCalculators, which need the efficiency,
    // don't get zeroes through multiplying by the normFactor.
    // we need to update the normal here, as values are used at this point.
    host_normalizations.sync(d_normalizations);
    
    int totalBins = _mprime.getNumBins() * _thetaprime.getNumBins();
    double binSizeFactor = 1;
    binSizeFactor *= _mprime.getBinSize();
    binSizeFactor *= _thetaprime.getBinSize();

    if(!dalitzNormRange) {
        gooMalloc((void **)&dalitzNormRange, 6 * sizeof(fptype));
    }

    // This line runs once
    static std::array<fptype, 6> host_norms{{0, 0, 0, 0, 0, 0}};

    std::array<fptype, 6> current_host_norms{{_mprime.getLowerLimit(),
                                              _mprime.getUpperLimit(),
                                              static_cast<fptype>(_mprime.getNumBins()),
                                              _thetaprime.getLowerLimit(),
                                              _thetaprime.getUpperLimit(),
                                              static_cast<fptype>(_thetaprime.getNumBins())}};

    if(host_norms != current_host_norms) {
        host_norms = current_host_norms;
    }
    MEMCPY(dalitzNormRange, host_norms.data(), 6 * sizeof(fptype), hipMemcpyHostToDevice);
    for(unsigned int i = 0; i < decayInfo.resonances.size(); ++i) {
        redoIntegral[i] = forceRedoIntegrals;

        if(!(decayInfo.resonances[i]->parametersChanged()))
            continue;

        redoIntegral[i] = true;
    }

    forceRedoIntegrals = false;

    thrust::constant_iterator<fptype *> arrayAddress(dalitzNormRange);
    thrust::counting_iterator<int> binIndex(0);
    thrust::counting_iterator<int> fakeEvtIndex(0);
   
    thrust::constant_iterator<fptype *> dataArray(dev_event_array);
    thrust::constant_iterator<fptype *> fakedataArray(dev_fake_event_array);

    thrust::constant_iterator<int> fakeeventSize(totalEventSize);
    thrust::constant_iterator<int> eventSize(totalEventSize);
    thrust::counting_iterator<int> eventIndex(eventOffset);

    //Calculate Resonances integrals without multiply by eff and calculate cache
    //These intergrals are used to normalize the RBWs
    size_t n_res=decayInfo.resonances.size();
    thrust::device_vector<fptype> device_integrals(n_res);
    for(int i = 0; i < n_res; ++i) {
                if((!redoIntegral[i]))
                    continue;

                integrators[i][i]->setDalitzIndex(getFunctionIndex());
                integrators[i][i]->setResonanceIndex(decayInfo.resonances[i]->getFunctionIndex());
                integrators[i][i]->setEfficiencyIndex(decayInfo.resonances[i]->getFunctionIndex());
                integrators[i][i]->setNoEff();
                thrust::constant_iterator<int> effFunc(efficiencyFunction);
                fpcomplex dummy(0, 0);
                thrust::plus<fpcomplex> complexSum;
                (*(integrals[i][i])) = thrust::transform_reduce(
                    thrust::make_zip_iterator(thrust::make_tuple(fakeEvtIndex, fakedataArray, fakeeventSize, effFunc)),
                    thrust::make_zip_iterator(thrust::make_tuple(fakeEvtIndex + NumNormEvents, arrayAddress, fakeeventSize, effFunc)),
                    *(integrators[i][i]),
                    dummy,
                    complexSum); 

                //Check if integral is 0.
                fptype integral = ((*(integrals[i][i])).real()/NumNormEvents);
                
                if(integral<=0.)
                    GOOFIT_ERROR("ERROR: Integral %d = %f.2f", i, integral); 
              
                fptype norm = 1./integral;

                cResonancesIntegrals[cacheToUse][i] = norm;
                //printf("Res  %d -  norm = %f \n",i,1./norm);

                calculators[i]->setResonanceIndex(decayInfo.resonances[i]->getFunctionIndex());
                calculators[i]->setDalitzIndex(getFunctionIndex());
            
                thrust::transform(
                    thrust::make_zip_iterator(thrust::make_tuple(eventIndex, dataArray, eventSize)),
                    thrust::make_zip_iterator(thrust::make_tuple(eventIndex + numEntries, arrayAddress, eventSize)),
                    strided_range<thrust::device_vector<fpcomplex>::iterator>(cachedWaves[i]->begin(), cachedWaves[i]->end(), 1).begin(),
                    *(calculators[i])
                );

                
                thrust::transform(
                    cachedWaves[i]->begin(),
                    cachedWaves[i]->end(), 
                    thrust::make_constant_iterator(sqrt(norm)),
                    cachedWaves[i]->begin(),
                    thrust::multiplies<fpcomplex>());
               
                  
    }
    
    //Calculate All Integrals with Eff
    for(int i = 0; i < n_res; ++i) {
        for(int j = 0; j < n_res; ++j) {
            if((!redoIntegral[i]) && (!redoIntegral[j]))
                continue;
        
            integrators[i][j]->setDalitzIndex(getFunctionIndex());
            integrators[i][j]->setResonanceIndex(decayInfo.resonances[i]->getFunctionIndex());
            integrators[i][j]->setEfficiencyIndex(decayInfo.resonances[j]->getFunctionIndex());
            thrust::constant_iterator<int> effFunc(efficiencyFunction);
            fpcomplex dummy(0, 0);
            thrust::plus<fpcomplex> complexSum;
            (*(integrals[i][j])) = thrust::transform_reduce(
                thrust::system::cuda::par,
                thrust::make_zip_iterator(thrust::make_tuple(fakeEvtIndex, fakedataArray, fakeeventSize, effFunc)),
                thrust::make_zip_iterator(thrust::make_tuple(fakeEvtIndex + NumNormEvents, arrayAddress, fakeeventSize, effFunc)),
                *(integrators[i][j]),
                dummy,
                complexSum);
        }
    }

    fpcomplex sumIntegral(0, 0);
    for(unsigned int i = 0; i < n_res; ++i) {
        fpcomplex amplitude_i(host_parameters[parametersIdx + i * 2 + 1], host_parameters[parametersIdx + i * 2 + 2]);
        for(unsigned int j = 0; j < n_res ; ++j) {
            fpcomplex amplitude_j(host_parameters[parametersIdx + j * 2 + 1],
                                  -host_parameters[parametersIdx + j * 2 + 2]);

            fptype fNorm_i = cResonancesIntegrals[cacheToUse][i];
            fptype fNorm_j = cResonancesIntegrals[cacheToUse][j];

            sumIntegral += amplitude_i * amplitude_j * ((*(integrals[i][j]))/NumNormEvents)*sqrt(fNorm_i)*sqrt(fNorm_j);
            
       
        }
    }

    fptype ret           = sumIntegral.real(); // That complex number is a square, so it's fully real
  
    //ret /= totalFakeEvent;
    host_normalizations[normalIdx + 1] = 1.0 / ret;
    cachedNormalization                = 1.0 / ret;
    return ret;
}

__host__ auto Amp3BodySqDP::sumCachedWave(size_t i) const -> fpcomplex {
    const thrust::device_vector<fpcomplex> &vec = getCachedWaveNoCopy(i);

    fpcomplex ret = thrust::reduce(vec.begin(), vec.end(), fpcomplex(0, 0), thrust::plus<fpcomplex>());

    return ret;
}

__host__ auto Amp3BodySqDP::getCachedWave(size_t i) const -> const std::vector<std::complex<fptype>> {
    // TODO: This calls itself immediately ?
    auto ret_thrust = getCachedWave(i);
    std::vector<std::complex<fptype>> ret(ret_thrust.size());
    thrust::copy(ret_thrust.begin(), ret_thrust.end(), ret.begin());
    return ret;
}

__host__ auto Amp3BodySqDP::fit_fractions(bool print) -> std::vector<std::vector<fptype>> {
     recursiveSetNormalization(1.0);

    host_normalizations.sync(d_normalizations);
    
    size_t n_res     = getDecayInfo().resonances.size();
    size_t totalBins = _mprime.getNumBins() * _thetaprime.getNumBins();
    double binSizeFactor = 1;
    binSizeFactor *= _mprime.getBinSize();
    binSizeFactor *= _thetaprime.getBinSize();

    if(!dalitzNormRange) {
        gooMalloc((void **)&dalitzNormRange, 6 * sizeof(fptype));
    }

    // This line runs once
    static std::array<fptype, 6> host_norms{{0, 0, 0, 0, 0, 0}};

    std::array<fptype, 6> current_host_norms{{_mprime.getLowerLimit(),
                                              _mprime.getUpperLimit(),
                                              static_cast<fptype>(_mprime.getNumBins()),
                                              _thetaprime.getLowerLimit(),
                                              _thetaprime.getUpperLimit(),
                                              static_cast<fptype>(_thetaprime.getNumBins())}};

    if(host_norms != current_host_norms) {
        host_norms = current_host_norms;
    }

    MEMCPY(dalitzNormRange, host_norms.data(), 6 * sizeof(fptype), hipMemcpyHostToDevice);

    for(unsigned int i = 0; i < decayInfo.resonances.size(); ++i) {
        redoIntegral[i] = forceRedoIntegrals;

        if(!(decayInfo.resonances[i]->parametersChanged()))
            continue;

        redoIntegral[i] = true;
    }

    forceRedoIntegrals = false;

    // Only do this bit if masses or widths have changed.
    thrust::constant_iterator<fptype *> arrayAddress(dalitzNormRange);
    thrust::counting_iterator<int> fakeEvtIndex(0);
    thrust::constant_iterator<fptype *> fakedataArray(dev_fake_event_array);
    thrust::constant_iterator<int> fakeeventSize(totalEventSize);

    for(int i = 0; i < n_res; ++i) {
        for(int j = 0; j < n_res ; ++j) {

            integrators_ff[i][j]->setDalitzIndex(getFunctionIndex());
            integrators_ff[i][j]->setResonanceIndex(decayInfo.resonances[i]->getFunctionIndex());
            integrators_ff[i][j]->setEfficiencyIndex(decayInfo.resonances[j]->getFunctionIndex());
            integrators_ff[i][j]->setNoEff();
            thrust::constant_iterator<int> effFunc(efficiencyFunction);
            fpcomplex dummy_ff(0, 0);
            thrust::plus<fpcomplex> complexSum_ff;
            (*(integrals_ff[i][j])) = thrust::transform_reduce(
                thrust::make_zip_iterator(thrust::make_tuple(fakeEvtIndex, fakedataArray, fakeeventSize, effFunc)),
                thrust::make_zip_iterator(thrust::make_tuple(fakeEvtIndex + NumNormEvents, arrayAddress, fakeeventSize, effFunc)),
                *(integrators_ff[i][j]),
                dummy_ff,
                complexSum_ff);
            
        }

    }

    // End of time-consuming integrals.
    fpcomplex sumIntegral(0, 0);
    std::vector<std::vector<fptype>> AmpIntegral(n_res, std::vector<fptype>(n_res));

    for(unsigned int i = 0; i < n_res; ++i) {
        fpcomplex amplitude_i(host_parameters[parametersIdx + i * 2 + 1], host_parameters[parametersIdx + i * 2 + 2]);
        fpcomplex buffer(0., 0.);

        for(unsigned int j = 0; j < n_res; ++j) {
            fpcomplex amplitude_j(host_parameters[parametersIdx + j * 2 + 1],
                                  -host_parameters[parametersIdx + j * 2 + 2]);

            fptype norm_i = 1./((*(integrals[i][i])).real()/NumNormEvents);
            fptype norm_j = 1./((*(integrals[j][j])).real()/NumNormEvents);
            
            buffer = amplitude_i * amplitude_j * ((*(integrals[i][j]))/NumNormEvents)*sqrt(norm_i)*sqrt(norm_j);
    
            AmpIntegral[i][j] = buffer.real();
            sumIntegral += buffer;
        }
    }

    totalFF_integral = sumIntegral.real();

    for(int i = 0; i < n_res; i++) {
        for(int j = 0; j < n_res; j++) {
            AmpIntegral[i][j] /= totalFF_integral;
            AmpIntegral[i][j] *= 100;
        }
    }

    if(print) {
        std::cout << "Fit Fractions Matrix (%): \n";
        std::cout << "*Note: the order of diag FFs is equal to the order that which resonances are pushed into the "
                     "resonance vector. \n";
        Eigen::MatrixXd m(n_res, n_res);
        for(int i = 0; i < n_res; i++)
            m.row(i) = Eigen::Map<Eigen::VectorXd>(&AmpIntegral[i][0], n_res);

        std::cout << std::fixed << m << std::endl;
        fptype sumdiagffs = 0.;

        std::cout << "\n ";
        std::cout << "Diagonal Fit Fractions (%): \n";


        for(int i = 0; i < n_res; i++){
            auto name = decayInfo.resonances[i]->getName();
            std::cout  << name << "\t" << std::fixed << m(i, i) << '\n';
            sumdiagffs += m(i, i);
        }
        std::cout << "Sum of Diag FFs: " << sumdiagffs << "\n";
        std::cout << "\n";
    }

    return AmpIntegral;
}

__host__ auto Amp3BodySqDP::GenerateSig(unsigned int numEvents, int seed) -> std::
    tuple<mcbooster::ParticlesSet_h, mcbooster::VariableSet_h, mcbooster::RealVector_h, mcbooster::RealVector_h> {
    // Must configure our functions before any calculations!
    // setupObservables();
    // setIndices();

    initialize();

    // Defining phase space
    std::vector<mcbooster::GReal_t> masses{decayInfo.daug1Mass, decayInfo.daug2Mass, decayInfo.daug3Mass};
    mcbooster::PhaseSpace phsp(decayInfo.motherMass, masses, numEvents, generation_offset);

    if(seed != 0) {
        phsp.SetSeed(seed);
    } else {
        GOOFIT_INFO("Current generator seed {}, offset {}", phsp.GetSeed(), generation_offset);
    }

    // Generating numEvents events. Events are all generated inside the phase space with uniform distribution in
    // momentum space. Events must be weighted to have phase space distribution
    phsp.Generate(mcbooster::Vector4R(decayInfo.motherMass, 0.0, 0.0, 0.0));

    auto d1 = phsp.GetDaughters(0);
    auto d2 = phsp.GetDaughters(1);
    auto d3 = phsp.GetDaughters(2);

    mcbooster::ParticlesSet_d pset(3);
    pset[0] = &d1;
    pset[1] = &d2;
    pset[2] = &d3;

    auto SigGen_M12_d = mcbooster::RealVector_d(numEvents);
    auto SigGen_M13_d = mcbooster::RealVector_d(numEvents);
    auto SigGen_M23_d = mcbooster::RealVector_d(numEvents);

    mcbooster::VariableSet_d VarSet_d(3);
    VarSet_d[0] = &SigGen_M12_d;
    VarSet_d[1] = &SigGen_M23_d;
    VarSet_d[2] = &SigGen_M13_d;

    // Evaluating invariant masses for each event
    
    Dim2 eval = Dim2();
    mcbooster::EvaluateArray<Dim2>(eval, pset, VarSet_d);

    mcbooster::VariableSet_d GooVarSet_d(3);
    GooVarSet_d[0] = VarSet_d[0];
    GooVarSet_d[1] = VarSet_d[2];
    GooVarSet_d[2] = VarSet_d[1];

    auto h1 = new mcbooster::Particles_h(d1);
    auto h2 = new mcbooster::Particles_h(d2);
    auto h3 = new mcbooster::Particles_h(d3);

    mcbooster::ParticlesSet_h ParSet(3);
    ParSet[0] = h1;
    ParSet[1] = h2;
    ParSet[2] = h3;

    auto SigGen_M12_h = new mcbooster::RealVector_h(SigGen_M12_d);
    auto SigGen_M23_h = new mcbooster::RealVector_h(SigGen_M23_d);
    auto SigGen_M13_h = new mcbooster::RealVector_h(SigGen_M13_d);

    mcbooster::VariableSet_h VarSet(3);
    VarSet[0] = SigGen_M12_h;
    VarSet[1] = SigGen_M23_h;
    VarSet[2] = SigGen_M13_h;

    mcbooster::RealVector_d weights(phsp.GetWeights());
    phsp.FreeResources();

    auto DS = new mcbooster::RealVector_d(3 * numEvents);
    thrust::counting_iterator<int> eventNumber(0);

#pragma unroll

    for(int i = 0; i < 2; ++i) {
        mcbooster::strided_range<mcbooster::RealVector_d::iterator> sr(DS->begin() + i, DS->end(), 3);
        thrust::copy(GooVarSet_d[i]->begin(), GooVarSet_d[i]->end(), sr.begin());
    }

    mcbooster::strided_range<mcbooster::RealVector_d::iterator> sr(DS->begin() + 2, DS->end(), 3);
    thrust::copy(eventNumber, eventNumber + numEvents, sr.begin());

    // Giving events to GooFit. Format of dev_evt_array must be (s12, s13, eventNumber). s23 is calculated automatically
    // in src/PDFs/physics/detail/SpecialResonanceCalculator.cu
    dev_event_array = thrust::raw_pointer_cast(DS->data());
    setDataSize(numEvents, 3);

    generation_no_norm = true; // we need no normalization for generation, but we do need to make sure that norm = 1;
    SigGenSetIndices();
    copyParams();
    normalize();
    setForceIntegrals();
    host_normalizations.sync(d_normalizations);

    auto fc = fitControl;
    setFitControl(std::make_shared<ProbFit>());

    thrust::device_vector<fptype> results;
    GooPdf::evaluate_with_metric(results);

    // evaluating amplitudes for generated events, amplitudes are incorporated in weights
    thrust::transform(
        results.begin(), results.end(), weights.begin(), weights.begin(), thrust::multiplies<mcbooster::GReal_t>());

    // Filing accept/reject flags for resonant distribution for each generated event
    mcbooster::BoolVector_d flags(numEvents);
    fillMCFlags(flags, weights, numEvents);

    auto weights_h = mcbooster::RealVector_h(weights);
    auto results_h = mcbooster::RealVector_h(results);
    auto flags_h   = mcbooster::BoolVector_h(flags);
    hipDeviceSynchronize();

    setFitControl(fc);

    return std::make_tuple(ParSet, VarSet, weights_h, flags_h);
}

} // namespace GooFit
