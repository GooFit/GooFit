#include "hip/hip_runtime.h"
#include <mcbooster/Evaluate.h>
#include <mcbooster/GContainers.h>
#include <mcbooster/GFunctional.h>
#include <mcbooster/GTypes.h>
#include <mcbooster/functors/FlagAcceptReject.h>

#include <thrust/extrema.h>

#include <goofit/PDFs/physics/Amp4BodyBase.h>
#include <goofit/PDFs/physics/Amp4BodyGlobals.h>

namespace GooFit {
/// Start with mcbooster::BoolVector_d flags(numEvents);
void Amp4BodyBase::fillMCFlags(mcbooster::BoolVector_d &flags,
                               const mcbooster::RealVector_d &weights,
                               unsigned int numEvents) {
    thrust::counting_iterator<mcbooster::GLong_t> first(0);
    thrust::counting_iterator<mcbooster::GLong_t> last = first + numEvents;

    auto max = thrust::max_element(weights.begin(), weights.end());
    thrust::transform(first, last, weights.begin(), flags.begin(), mcbooster::FlagAcceptReject((fptype)*max));
}

} // namespace GooFit
