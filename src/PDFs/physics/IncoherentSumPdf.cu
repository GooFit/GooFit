#include "hip/hip_runtime.h"
#include <goofit/Error.h>
#include <goofit/PDFs/physics/IncoherentSumPdf.h>
#include <goofit/PDFs/physics/ResonancePdf.h>

#include <thrust/transform_reduce.h>

namespace GooFit {

const int resonanceOffset_incoherent = 4; // Offset of the first resonance into the parameter index array.
// Notice that this is different from the TddpPdf case because there's no time information.
// In particular the offset consists of nP, constant index, number of resonances, and cache index.

__device__ fpcomplex *cResonanceValues[10];

__device__ inline int parIndexFromResIndex_incoherent(int resIndex) {
    return resonanceOffset_incoherent + resIndex * resonanceSize;
}

__device__ fptype device_incoherent(fptype *evt, fptype *p, unsigned int *indices) {
    // Calculates the incoherent sum over the resonances.
    auto evtNum = static_cast<int>(floor(0.5 + evt[indices[4 + indices[0]]]));

    fptype ret                 = 0;
    unsigned int numResonances = indices[2];
    unsigned int cacheToUse    = indices[3];

    for(int i = 0; i < numResonances; ++i) {
        int paramIndex   = parIndexFromResIndex_incoherent(i);
        fptype amplitude = p[indices[paramIndex + 0]];

        fpcomplex matrixelement = cResonanceValues[cacheToUse][evtNum * numResonances + i];
        ret += amplitude * thrust::norm(matrixelement);
    }

    // Multiply by efficiency
    int effFunctionIdx = parIndexFromResIndex_incoherent(numResonances);
    fptype eff         = callFunction(evt, indices[effFunctionIdx], indices[effFunctionIdx + 1]);

    ret *= eff;

    return ret;
}

__device__ device_function_ptr ptr_to_incoherent = device_incoherent;

__host__ IncoherentSumPdf::IncoherentSumPdf(
    std::string n, Observable m12, Observable m13, EventNumber eventNumber, DecayInfo3 decay, GooPdf *eff)
    : GooPdf(n, m12, m13, eventNumber)
    , decayInfo(decay)
    , _m12(m12)
    , _m13(m13)
    , dalitzNormRange(nullptr)
    , cachedResonances(nullptr)
    , integrals(nullptr)
    , forceRedoIntegrals(true)
    , totalEventSize(3) // Default 3 = m12, m13, evtNum. Will likely be overridden.
    , cacheToUse(0)
    , efficiency(eff)
    , integrators(nullptr)
    , calculators(nullptr) {
    std::vector<unsigned int> pindices;
    pindices.push_back(registerConstants(5));
    fptype decayConstants[5];
    decayConstants[0] = decayInfo.motherMass;
    decayConstants[1] = decayInfo.daug1Mass;
    decayConstants[2] = decayInfo.daug2Mass;
    decayConstants[3] = decayInfo.daug3Mass;
    decayConstants[4] = decayInfo.meson_radius;
    MEMCPY_TO_SYMBOL(
        functorConstants, decayConstants, 5 * sizeof(fptype), cIndex * sizeof(fptype), hipMemcpyHostToDevice);

    pindices.push_back(decayInfo.resonances.size());
    static int cacheCount = 0;
    cacheToUse            = cacheCount++;
    pindices.push_back(cacheToUse);

    for(auto &resonance : decayInfo.resonances) {
        pindices.push_back(registerParameter(resonance->amp_real));
        pindices.push_back(registerParameter(resonance->amp_real));
        // Not going to use amp_imag, but need a dummy index so the resonance size will be consistent.
        pindices.push_back(resonance->getFunctionIndex());
        pindices.push_back(resonance->getParameterIndex());
        resonance->setConstantIndex(cIndex);
        components.push_back(resonance);
    }

    pindices.push_back(efficiency->getFunctionIndex());
    pindices.push_back(efficiency->getParameterIndex());
    components.push_back(efficiency);

    GET_FUNCTION_ADDR(ptr_to_incoherent);
    initialize(pindices);

    redoIntegral = new bool[decayInfo.resonances.size()];
    cachedMasses = new fptype[decayInfo.resonances.size()];
    cachedWidths = new fptype[decayInfo.resonances.size()];
    integrals    = new double[decayInfo.resonances.size()];

    for(int i = 0; i < decayInfo.resonances.size(); ++i) {
        redoIntegral[i] = true;
        cachedMasses[i] = -1;
        cachedWidths[i] = -1;
        integrals[i]    = 0;
    }

    integrators = new SpecialIncoherentIntegrator *[decayInfo.resonances.size()];
    calculators = new SpecialIncoherentResonanceCalculator *[decayInfo.resonances.size()];

    for(int i = 0; i < decayInfo.resonances.size(); ++i) {
        integrators[i] = new SpecialIncoherentIntegrator(parameters, i);
        calculators[i] = new SpecialIncoherentResonanceCalculator(parameters, i);
    }

    addSpecialMask(PdfBase::ForceSeparateNorm);
}

__host__ void IncoherentSumPdf::setDataSize(unsigned int dataSize, unsigned int evtSize) {
    // Default 3 is m12, m13, evtNum
    totalEventSize = evtSize;
    if(totalEventSize < 3)
        throw GooFit::GeneralError("totalEventSize {} must be 3 or more", totalEventSize);

    if(cachedResonances) {
        delete cachedResonances;
    }

    numEntries       = dataSize;
    cachedResonances = new thrust::device_vector<fpcomplex>(dataSize * decayInfo.resonances.size());
    void *dummy      = thrust::raw_pointer_cast(cachedResonances->data());
    MEMCPY_TO_SYMBOL(
        cResonanceValues, &dummy, sizeof(fpcomplex *), cacheToUse * sizeof(fpcomplex *), hipMemcpyHostToDevice);
    setForceIntegrals();
}

__host__ fptype IncoherentSumPdf::normalize() const {
    recursiveSetNormalisation(1); // Not going to normalize efficiency,
    // so set normalisation factor to 1 so it doesn't get multiplied by zero.
    // Copy at this time to ensure that the SpecialCalculators, which need the efficiency,
    // don't get zeroes through multiplying by the normFactor.
    MEMCPY_TO_SYMBOL(normalisationFactors, host_normalisation, totalParams * sizeof(fptype), 0, hipMemcpyHostToDevice);

    int totalBins = _m12.getNumBins() * _m13.getNumBins();

    if(!dalitzNormRange) {
        gooMalloc((void **)&dalitzNormRange, 6 * sizeof(fptype));

        auto *host_norms = new fptype[6];
        host_norms[0]    = _m12.getLowerLimit();
        host_norms[1]    = _m12.getUpperLimit();
        host_norms[2]    = _m12.getNumBins();
        host_norms[3]    = _m13.getLowerLimit();
        host_norms[4]    = _m13.getUpperLimit();
        host_norms[5]    = _m13.getNumBins();
        MEMCPY(dalitzNormRange, host_norms, 6 * sizeof(fptype), hipMemcpyHostToDevice);
        delete[] host_norms;
    }

    // Check if efficiency changes force redoing the integrals.
    if(efficiency->parametersChanged()) {
        forceRedoIntegrals = true;
    }

    // Check for changed masses or forced integral redo.
    for(unsigned int i = 0; i < decayInfo.resonances.size(); ++i) {
        redoIntegral[i] = forceRedoIntegrals;

        if(!(decayInfo.resonances[i]->parametersChanged()))
            continue;

        redoIntegral[i] = true;
    }

    forceRedoIntegrals = false;

    thrust::constant_iterator<fptype *> arrayAddress(dalitzNormRange);
    thrust::counting_iterator<int> binIndex(0);

    // NB, SpecialIncoherentResonanceCalculator assumes that fit is unbinned!
    // And it needs to know the total event size, not just observables
    // for this particular PDF component.
    thrust::constant_iterator<fptype *> dataArray(dev_event_array);
    thrust::constant_iterator<int> eventSize(totalEventSize);
    thrust::counting_iterator<int> eventIndex(0);

    for(int i = 0; i < decayInfo.resonances.size(); ++i) {
        if(redoIntegral[i]) {
            thrust::transform(
                thrust::make_zip_iterator(thrust::make_tuple(eventIndex, dataArray, eventSize)),
                thrust::make_zip_iterator(thrust::make_tuple(eventIndex + numEntries, arrayAddress, eventSize)),
                strided_range<thrust::device_vector<fpcomplex>::iterator>(
                    cachedResonances->begin() + i, cachedResonances->end(), decayInfo.resonances.size())
                    .begin(),
                *(calculators[i]));

            fptype dummy = 0;
            static thrust::plus<fptype> cudaPlus;
            integrals[i] = thrust::transform_reduce(
                thrust::make_zip_iterator(thrust::make_tuple(binIndex, arrayAddress)),
                thrust::make_zip_iterator(thrust::make_tuple(binIndex + totalBins, arrayAddress)),
                *(integrators[i]),
                dummy,
                cudaPlus);
        }
    }

    // End of time-consuming integrals and caching of BWs over Dalitz plot.

    fptype ret = 0;

    for(unsigned int i = 0; i < decayInfo.resonances.size(); ++i) {
        int param_i      = parameters + resonanceOffset_incoherent + resonanceSize * i;
        fptype amplitude = host_params[host_indices[param_i]];
        ret += amplitude * integrals[i];
    }

    double binSizeFactor = 1;
    binSizeFactor *= _m12.getBinSize();
    binSizeFactor *= _m13.getBinSize();
    ret *= binSizeFactor;

    host_normalisation[parameters] = 1.0 / ret;
    return ret;
}

SpecialIncoherentIntegrator::SpecialIncoherentIntegrator(int pIdx, unsigned int ri)
    : resonance_i(ri)
    , parameters(pIdx) {}

__device__ fptype SpecialIncoherentIntegrator::operator()(thrust::tuple<int, fptype *> t) const {
    // Returns integral of specific BW over Dalitz plot, to be cached and
    // multiplied by rapidly-changing amplitude.

    // Bin index, base address [lower, upper,getNumBins]
    // Notice that this is basically MetricTaker::operator (binned) with the special-case knowledge
    // that event size is two, and that the function to call is getResonanceAmplitude.

    int globalBinNumber  = thrust::get<0>(t);
    fptype lowerBoundM12 = thrust::get<1>(t)[0];
    fptype upperBoundM12 = thrust::get<1>(t)[1];
    auto numBinsM12      = static_cast<int>(floor(thrust::get<1>(t)[2] + 0.5));
    int binNumberM12     = globalBinNumber % numBinsM12;
    fptype binCenterM12  = upperBoundM12 - lowerBoundM12;
    binCenterM12 /= numBinsM12;
    binCenterM12 *= (binNumberM12 + 0.5);
    binCenterM12 += lowerBoundM12;

    globalBinNumber /= numBinsM12;
    fptype lowerBoundM13 = thrust::get<1>(t)[3];
    fptype upperBoundM13 = thrust::get<1>(t)[4];
    auto numBinsM13      = static_cast<int>(floor(thrust::get<1>(t)[5] + 0.5));
    fptype binCenterM13  = upperBoundM13 - lowerBoundM13;
    binCenterM13 /= numBinsM13;
    binCenterM13 *= (globalBinNumber + 0.5);
    binCenterM13 += lowerBoundM13;

    unsigned int *indices = paramIndices + parameters;
    fptype motherMass     = functorConstants[indices[1] + 0];
    fptype daug1Mass      = functorConstants[indices[1] + 1];
    fptype daug2Mass      = functorConstants[indices[1] + 2];
    fptype daug3Mass      = functorConstants[indices[1] + 3];

    if(!inDalitz(binCenterM12, binCenterM13, motherMass, daug1Mass, daug2Mass, daug3Mass))
        return 0;

    int parameter_i
        = parIndexFromResIndex_incoherent(resonance_i); // Find position of this resonance relative to TDDP start
    unsigned int functn_i = indices[parameter_i + 2];
    unsigned int params_i = indices[parameter_i + 3];
    fptype m23 = motherMass * motherMass + daug1Mass * daug1Mass + daug2Mass * daug2Mass + daug3Mass * daug3Mass
                 - binCenterM12 - binCenterM13;
    fpcomplex ret = getResonanceAmplitude(binCenterM12, binCenterM13, m23, functn_i, params_i);

    unsigned int numResonances = indices[2];
    fptype fakeEvt[10]; // Need room for many observables in case m12 or m13 were assigned a high index in an
                        // event-weighted fit.
    fakeEvt[indices[indices[0] + 2 + 0]] = binCenterM12;
    fakeEvt[indices[indices[0] + 2 + 1]] = binCenterM13;
    int effFunctionIdx                   = parIndexFromResIndex_incoherent(numResonances);
    fptype eff                           = callFunction(fakeEvt, indices[effFunctionIdx], indices[effFunctionIdx + 1]);

    return thrust::norm(ret) * eff;
}

SpecialIncoherentResonanceCalculator::SpecialIncoherentResonanceCalculator(int pIdx, unsigned int res_idx)
    : resonance_i(res_idx)
    , parameters(pIdx) {}

__device__ fpcomplex SpecialIncoherentResonanceCalculator::operator()(thrust::tuple<int, fptype *, int> t) const {
    // Returns the BW, or other resonance function, for a specific resonance.
    // Is special because the value is expected to change slowly, so it's
    // useful to cache the result.
    int evtNum  = thrust::get<0>(t);
    fptype *evt = thrust::get<1>(t) + (evtNum * thrust::get<2>(t));

    unsigned int *indices = paramIndices + parameters; // Jump to TDDP position within parameters array
    fptype m12            = evt[indices[2 + indices[0]]];
    fptype m13            = evt[indices[3 + indices[0]]];
    fptype motherMass     = functorConstants[indices[1] + 0];
    fptype daug1Mass      = functorConstants[indices[1] + 1];
    fptype daug2Mass      = functorConstants[indices[1] + 2];
    fptype daug3Mass      = functorConstants[indices[1] + 3];

    if(!inDalitz(m12, m13, motherMass, daug1Mass, daug2Mass, daug3Mass))
        return {0., 0.};

    fptype m23
        = motherMass * motherMass + daug1Mass * daug1Mass + daug2Mass * daug2Mass + daug3Mass * daug3Mass - m12 - m13;

    int parameter_i
        = parIndexFromResIndex_incoherent(resonance_i); // Find position of this resonance relative to TDDP start
    unsigned int functn_i = indices[parameter_i + 2];
    unsigned int params_i = indices[parameter_i + 3];
    fpcomplex ret         = getResonanceAmplitude(m12, m13, m23, functn_i, params_i);

    return ret;
}

} // namespace GooFit
