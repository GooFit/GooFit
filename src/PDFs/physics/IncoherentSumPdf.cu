#include "hip/hip_runtime.h"
#include <goofit/Error.h>
#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/physics/IncoherentSumPdf.h>
#include <goofit/PDFs/physics/ResonancePdf.h>
#include <thrust/complex.h>

#include <thrust/transform_reduce.h>

namespace GooFit {

const int resonanceOffset_incoherent = 4; // Offset of the first resonance into the parameter index array.
// Notice that this is different from the TddpPdf case because there's no time information.
// In particular the offset consists of nP, constant index, number of resonances, and cache index.

__device__ fpcomplex *cResonanceValues[10];

__device__ inline int parIndexFromResIndex_incoherent(int resIndex) {
    return resonanceOffset_incoherent + resIndex * resonanceSize;
}

__device__ fptype device_incoherent(fptype *evt, ParameterContainer &pc) {
    // Calculates the incoherent sum over the resonances.
    int numObs  = pc.getNumObservables();
    int evtId   = pc.getObservable(2);
    auto evtNum = static_cast<int>(floor(0.5 + evt[evtId]));

    fptype ret                 = 0;
    unsigned int numResonances = pc.getConstant(4);
    unsigned int cacheToUse    = pc.getConstant(5);

    for(int i = 0; i < numResonances; ++i) {
        // int paramIndex   = parIndexFromResIndex_incoherent(i);
        // fptype amplitude = p[indices[paramIndex + 0]];
        fptype amplitude = pc.getParameter(i);

        fpcomplex matrixelement = cResonanceValues[cacheToUse][evtNum * numResonances + i];
        ret += amplitude * thrust::norm(matrixelement);
    }

    // pc.incrementIndex(1, numResonances, 2, numObs, 1);
    pc.incrementIndex();

    // increment through resonances
    for(int i = 0; i < numResonances; i++)
        pc.incrementIndex();
    // Multiply by efficiency
    // int effFunctionIdx = parIndexFromResIndex_incoherent(numResonances);
    fptype eff = callFunction(evt, pc);

    ret *= eff;

    return ret;
}

__device__ device_function_ptr ptr_to_incoherent = device_incoherent;

__host__ IncoherentSumPdf::IncoherentSumPdf(
    std::string n, Observable m12, Observable m13, EventNumber eventNumber, DecayInfo3 decay, GooPdf *eff)
    : GooPdf(n, m12, m13, eventNumber)
    , decayInfo(decay)
    , _m12(m12)
    , _m13(m13)
    , dalitzNormRange(nullptr)
    , cachedResonances(nullptr)
    , integrals(nullptr)
    , forceRedoIntegrals(true)
    , totalEventSize(3) // Default 3 = m12, m13, evtNum. Will likely be overridden.
    , cacheToUse(0)
    , efficiency(eff)
    , integrators(nullptr)
    , calculators(nullptr) {
    registerConstant(observablesList.size());
    registerConstant(0);
    registerConstant(0);
    registerConstant(0);

    MEMCPY_TO_SYMBOL(c_motherMass, &decayInfo.motherMass, sizeof(fptype), 0, hipMemcpyHostToDevice);
    MEMCPY_TO_SYMBOL(c_daug1Mass, &decayInfo.daug1Mass, sizeof(fptype), 0, hipMemcpyHostToDevice);
    MEMCPY_TO_SYMBOL(c_daug2Mass, &decayInfo.daug2Mass, sizeof(fptype), 0, hipMemcpyHostToDevice);
    MEMCPY_TO_SYMBOL(c_daug3Mass, &decayInfo.daug3Mass, sizeof(fptype), 0, hipMemcpyHostToDevice);
    MEMCPY_TO_SYMBOL(c_meson_radius, &decayInfo.meson_radius, sizeof(fptype), 0, hipMemcpyHostToDevice);
    static int cacheCount = 0;
    cacheToUse            = cacheCount++;
    registerConstant(decayInfo.resonances.size());
    registerConstant(cacheToUse);

    for(auto &resonance : decayInfo.resonances) {
        // Not going to use amp_imag, but need a dummy index so the resonance size will be consistent.
        // pindices.push_back(resonance->getFunctionIndex());
        // pindices.push_back(resonance->getParameterIndex());
        // resonance->setConstantIndex(cIndex);
        components.push_back(resonance);
    }

    components.push_back(efficiency);

    initialize();

    redoIntegral = new bool[decayInfo.resonances.size()];
    cachedMasses = new fptype[decayInfo.resonances.size()];
    cachedWidths = new fptype[decayInfo.resonances.size()];
    integrals    = new double[decayInfo.resonances.size()];

    for(int i = 0; i < decayInfo.resonances.size(); ++i) {
        redoIntegral[i] = true;
        cachedMasses[i] = -1;
        cachedWidths[i] = -1;
        integrals[i]    = 0;
    }

    integrators = new SpecialIncoherentIntegrator *[decayInfo.resonances.size()];
    calculators = new SpecialIncoherentResonanceCalculator *[decayInfo.resonances.size()];

    for(int i = 0; i < decayInfo.resonances.size(); ++i) {
        integrators[i] = new SpecialIncoherentIntegrator(parameters, i);
        calculators[i] = new SpecialIncoherentResonanceCalculator(parameters, i);
    }

    addSpecialMask(PdfBase::ForceSeparateNorm);
}

__host__ void IncoherentSumPdf::recursiveSetIndices() {
    GOOFIT_RECURSIVE_SET_INDICIES(ptr_to_incoherent);

    // save our efficiency function.  Resonance's are saved first, then the efficiency function.  Take -1 as efficiency!
    efficiencyFunction = num_device_functions - 1;
}
__host__ void IncoherentSumPdf::setDataSize(unsigned int dataSize, unsigned int evtSize) {
    // Default 3 is m12, m13, evtNum
    totalEventSize = evtSize;
    if(totalEventSize < 3)
        throw GooFit::GeneralError("totalEventSize {} must be 3 or more", totalEventSize);

    if(cachedResonances) {
        delete cachedResonances;
    }

    numEntries       = dataSize;
    cachedResonances = new thrust::device_vector<fpcomplex>(dataSize * decayInfo.resonances.size());
    void *dummy      = thrust::raw_pointer_cast(cachedResonances->data());
    MEMCPY_TO_SYMBOL(
        cResonanceValues, &dummy, sizeof(fpcomplex *), cacheToUse * sizeof(fpcomplex *), hipMemcpyHostToDevice);
    setForceIntegrals();
}

__host__ fptype IncoherentSumPdf::normalize() const {
    recursiveSetNormalisation(1); // Not going to normalize efficiency,
    // so set normalisation factor to 1 so it doesn't get multiplied by zero.
    // Copy at this time to ensure that the SpecialCalculators, which need the efficiency,
    // don't get zeroes through multiplying by the normFactor.
    MEMCPY_TO_SYMBOL(
        d_normalisations, host_normalisations, totalNormalisations * sizeof(fptype), 0, hipMemcpyHostToDevice);

    int totalBins = _m12.getNumBins() * _m13.getNumBins();

    if(!dalitzNormRange) {
        gooMalloc((void **)&dalitzNormRange, 6 * sizeof(fptype));

        auto *host_norms = new fptype[6];
        host_norms[0]    = _m12.getLowerLimit();
        host_norms[1]    = _m12.getUpperLimit();
        host_norms[2]    = _m12.getNumBins();
        host_norms[3]    = _m13.getLowerLimit();
        host_norms[4]    = _m13.getUpperLimit();
        host_norms[5]    = _m13.getNumBins();
        MEMCPY(dalitzNormRange, host_norms, 6 * sizeof(fptype), hipMemcpyHostToDevice);
        delete[] host_norms;
    }

    // Check if efficiency changes force redoing the integrals.
    if(efficiency->parametersChanged()) {
        forceRedoIntegrals = true;
    }

    // Check for changed masses or forced integral redo.
    for(unsigned int i = 0; i < decayInfo.resonances.size(); ++i) {
        redoIntegral[i] = forceRedoIntegrals;

        if(!(decayInfo.resonances[i]->parametersChanged()))
            continue;

        redoIntegral[i] = true;
    }

    forceRedoIntegrals = false;

    thrust::constant_iterator<fptype *> arrayAddress(dalitzNormRange);
    thrust::counting_iterator<int> binIndex(0);

    // NB, SpecialIncoherentResonanceCalculator assumes that fit is unbinned!
    // And it needs to know the total event size, not just observables
    // for this particular PDF component.
    thrust::constant_iterator<fptype *> dataArray(dev_event_array);
    thrust::constant_iterator<int> eventSize(totalEventSize);
    thrust::counting_iterator<int> eventIndex(0);

    for(int i = 0; i < decayInfo.resonances.size(); ++i) {
        if(redoIntegral[i]) {
            calculators[i]->setIncoherentIndex(getFunctionIndex());
            calculators[i]->setResonanceIndex(decayInfo.resonances[i]->getFunctionIndex());
            thrust::transform(
                thrust::make_zip_iterator(thrust::make_tuple(eventIndex, dataArray, eventSize)),
                thrust::make_zip_iterator(thrust::make_tuple(eventIndex + numEntries, arrayAddress, eventSize)),
                strided_range<thrust::device_vector<fpcomplex>::iterator>(
                    cachedResonances->begin() + i, cachedResonances->end(), decayInfo.resonances.size())
                    .begin(),
                *(calculators[i]));

            integrators[i]->setIncoherentIndex(getFunctionIndex());
            integrators[i]->setEfficiencyIndex(efficiencyFunction);
            integrators[i]->setResonanceIndex(decayInfo.resonances[i]->getFunctionIndex());
            fptype dummy = 0;
            static thrust::plus<fptype> cudaPlus;
            integrals[i] = thrust::transform_reduce(
                thrust::make_zip_iterator(thrust::make_tuple(binIndex, arrayAddress)),
                thrust::make_zip_iterator(thrust::make_tuple(binIndex + totalBins, arrayAddress)),
                *(integrators[i]),
                dummy,
                cudaPlus);
        }
    }

    // End of time-consuming integrals and caching of BWs over Dalitz plot.

    fptype ret = 0;

    for(unsigned int i = 0; i < decayInfo.resonances.size(); ++i) {
        // int param_i      = parameters + resonanceOffset_incoherent + resonanceSize * i;
        fptype amplitude = host_parameters[parametersIdx + i + 1];
        ret += amplitude * integrals[i];
    }

    double binSizeFactor = 1;
    binSizeFactor *= _m12.getBinSize();
    binSizeFactor *= _m13.getBinSize();
    ret *= binSizeFactor;

    host_normalisations[normalIdx + 1] = 1.0 / ret;
    return ret;
}

SpecialIncoherentIntegrator::SpecialIncoherentIntegrator(int pIdx, unsigned int ri)
    : resonance_i(ri)
    , parameters(pIdx) {}

__device__ fptype SpecialIncoherentIntegrator::operator()(thrust::tuple<int, fptype *> t) const {
    // Returns integral of specific BW over Dalitz plot, to be cached and
    // multiplied by rapidly-changing amplitude.

    // Bin index, base address [lower, upper,getNumBins]
    // Notice that this is basically MetricTaker::operator (binned) with the special-case knowledge
    // that event size is two, and that the function to call is getResonanceAmplitude.

    int globalBinNumber  = thrust::get<0>(t);
    fptype lowerBoundM12 = thrust::get<1>(t)[0];
    fptype upperBoundM12 = thrust::get<1>(t)[1];
    auto numBinsM12      = static_cast<int>(floor(thrust::get<1>(t)[2] + 0.5));
    int binNumberM12     = globalBinNumber % numBinsM12;
    fptype binCenterM12  = upperBoundM12 - lowerBoundM12;
    binCenterM12 /= numBinsM12;
    binCenterM12 *= (binNumberM12 + 0.5);
    binCenterM12 += lowerBoundM12;

    globalBinNumber /= numBinsM12;
    fptype lowerBoundM13 = thrust::get<1>(t)[3];
    fptype upperBoundM13 = thrust::get<1>(t)[4];
    auto numBinsM13      = static_cast<int>(floor(thrust::get<1>(t)[5] + 0.5));
    fptype binCenterM13  = upperBoundM13 - lowerBoundM13;
    binCenterM13 /= numBinsM13;
    binCenterM13 *= (globalBinNumber + 0.5);
    binCenterM13 += lowerBoundM13;

    ParameterContainer pc;

    while(pc.funcIdx < incoherentSum)
        pc.incrementIndex();

    if(!inDalitz(binCenterM12, binCenterM13, c_motherMass, c_daug1Mass, c_daug2Mass, c_daug3Mass))
        return 0;

    int id_m12 = RO_CACHE(pc.observables[pc.observableIdx + 1]);
    int id_m13 = RO_CACHE(pc.observables[pc.observableIdx + 2]);

    int num_res = RO_CACHE(pc.constants[pc.constantIdx + 5]);

    // int parameter_i
    //    = parIndexFromResIndex_incoherent(resonance_i); // Find position of this resonance relative to TDDP start
    // unsigned int functn_i = indices[parameter_i + 2];
    // unsigned int params_i = indices[parameter_i + 3];
    fptype m23 = c_motherMass * c_motherMass + c_daug1Mass * c_daug1Mass + c_daug2Mass * c_daug2Mass
                 + c_daug3Mass * c_daug3Mass - binCenterM12 - binCenterM13;

    while(pc.funcIdx < num_res)
        pc.incrementIndex();

    fpcomplex ret = getResonanceAmplitude(binCenterM12, binCenterM13, m23, pc);

    while(pc.funcIdx < efficiency)
        pc.incrementIndex();

    // unsigned int numResonances = indices[2];
    fptype fakeEvt[10]; // Need room for many observables in case m12 or m13 were assigned a high index in an
                        // event-weighted fit.
    fakeEvt[0]      = 2;
    fakeEvt[id_m12] = binCenterM12;
    fakeEvt[id_m13] = binCenterM13;
    // int effFunctionIdx                   = parIndexFromResIndex_incoherent(numResonances);
    fptype eff = callFunction(fakeEvt, pc);

    return thrust::norm(ret) * eff;
}

SpecialIncoherentResonanceCalculator::SpecialIncoherentResonanceCalculator(int pIdx, unsigned int res_idx)
    : resonance_i(res_idx)
    , parameters(pIdx) {}

__device__ fpcomplex SpecialIncoherentResonanceCalculator::operator()(thrust::tuple<int, fptype *, int> t) const {
    // Returns the BW, or other resonance function, for a specific resonance.
    // Is special because the value is expected to change slowly, so it's
    // useful to cache the result.
    int evtNum  = thrust::get<0>(t);
    fptype *evt = thrust::get<1>(t) + (evtNum * thrust::get<2>(t));

    // unsigned int *indices = paramIndices + parameters; // Jump to TDDP position within parameters array
    ParameterContainer pc;

    while(pc.funcIdx < incoherentSum)
        pc.incrementIndex();

    int id_m12 = RO_CACHE(pc.observables[pc.observableIdx + 1]);
    int id_m13 = RO_CACHE(pc.observables[pc.observableIdx + 2]);

    fptype m12 = evt[id_m12];
    fptype m13 = evt[id_m13];

    if(!inDalitz(m12, m13, c_motherMass, c_daug1Mass, c_daug2Mass, c_daug3Mass))
        return {0.0, 0.0};

    fptype m23 = c_motherMass * c_motherMass + c_daug1Mass * c_daug1Mass + c_daug2Mass * c_daug2Mass
                 + c_daug3Mass * c_daug3Mass - m12 - m13;

    while(pc.funcIdx < resonance_i)
        pc.incrementIndex();

    // int parameter_i
    //    = parIndexFromResIndex_incoherent(resonance_i); // Find position of this resonance relative to TDDP start
    // unsigned int functn_i       = indices[parameter_i + 2];
    // unsigned int params_i       = indices[parameter_i + 3];
    fpcomplex ret = getResonanceAmplitude(m12, m13, m23, pc);

    return ret;
}

} // namespace GooFit
