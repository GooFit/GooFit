#include "hip/hip_runtime.h"
/*
04/05/2016 Christoph Hasse
DISCLAIMER:

This code is not sufficently tested yet and still under heavy development!

This file includes some lineshapes and spinfactors.
Also right now it is the home to some helper functions needed and an implementation of a simple 4-vec class that works
on the GPU
*/

#include "goofit/PDFs/physics/SpinFactors.h"
#include "goofit/PDFs/physics/LineshapesPdf.h"

#include <utility>

#include <goofit/detail/Macros.h>

namespace GooFit {

// Form factors as in pdg http://pdg.lbl.gov/2012/reviews/rpp2012-rev-dalitz-analysis-formalism.pdf
__device__ fptype BL_PRIME(fptype z2, fptype z02, int L) {
    if(0 == L)
        return 1.0;
    else if(1 == L)
        return (1 + z02) / (1 + z2);
    else if(2 == L)
        return (z02 * z02 + 3 * z02 + 9) / (z2 * z2 + 3 * z2 + 9);
    else {
        printf("ERROR! Oribtal > 2 not supported!\n");
        return 0;
    }

    // Spin 3 and up not accounted for.
}

__device__ fptype BL(fptype z2, int L) {
    if(0 == L)
        return 1.0;
    else if(1 == L)
        return 2 * z2 / (1 + z2);
    else if(2 == L)
        return (13 * z2 * z2) / (z2 * z2 + 3 * z2 + 9);
    else {
        printf("ERROR! Oribtal > 2 not supported!\n");
        return 0;
    }

    // Spin 3 and up not accounted for.
}

__device__ fptype BL2(fptype z2, int L) {
    if(0 == L)
        return 1.0;
    else if(1 == L)
        return 1.0 / (1 + z2);
    else if(2 == L)
        return 1.0 / (z2 * z2 + 3 * z2 + 9);
    else {
        printf("ERROR! Oribtal > 2 not supported!\n");
        return 0;
    }

    // Spin 3 and up not accounted for.
}

__device__ fpcomplex LS_ONE(fptype Mpair, fptype m1, fptype m2, unsigned int *indices) { return fpcomplex(1, 0); }

// This function is modeled after BW_BW::getVal() in BW_BW.cpp from the MINT package written by Jonas Rademacker.
__device__ fpcomplex BW(fptype Mpair, fptype m1, fptype m2, unsigned int *indices) {
    fptype meson_radius  = functorConstants[indices[7]];
    fptype resmass       = hipArray[indices[2]];
    fptype reswidth      = hipArray[indices[3]];
    unsigned int orbital = indices[4];
    unsigned int FF      = indices[6];

    const unsigned int to2Lplus1 = 2 * orbital + 1;

    fptype mass          = resmass;
    fptype width         = reswidth;
    fptype mumsRecoMass2 = Mpair * Mpair;

    fptype mpsq        = (m1 + m2) * (m1 + m2);
    fptype mmsq        = (m1 - m2) * (m1 - m2);
    fptype num         = (mumsRecoMass2 - mpsq) * (mumsRecoMass2 - mmsq);
    fptype num2        = (mass * mass - mpsq) * (mass * mass - mmsq);
    fptype pABSq       = num / (4 * mumsRecoMass2);
    fptype prSqForGofM = num2 / (4 * mass * mass);
    fptype prSq2       = prSqForGofM < 0 ? 0 : prSqForGofM;
    prSqForGofM        = fabs(prSqForGofM);

    fptype pratio = sqrt(pABSq / prSqForGofM);

    fptype pratio_to_2Jplus1 = 1;

    for(int i = 0; i < to2Lplus1; i++) {
        pratio_to_2Jplus1 *= pratio;
    }

    fptype mratio   = mass / Mpair;
    fptype r2       = meson_radius * meson_radius;
    fptype thisFR   = BL_PRIME(pABSq * r2, prSqForGofM * r2, orbital);
    fptype frFactor = 1;

    if(0 != orbital and 0 != FF) {
        frFactor = (FF == 1 ? BL(pABSq * r2, orbital) : BL_PRIME(pABSq * r2, prSq2 * r2, orbital));
        frFactor = (FF == 3 ? BL2(pABSq * r2, orbital) : frFactor);
    }

    fptype GofM = width * pratio_to_2Jplus1 * mratio * thisFR;

    fptype gamma = mass * sqrt((mass * mass + width * width));
    fptype k     = (2.0 * sqrt(2.0) / M_PI) * mass * width * gamma
               / sqrt(mass * mass + gamma); // Note added additional factor of 2*sqrt(2)/PI here so results are
                                            // comparable to MINT3. MINT2 doesn't have include this.

    fpcomplex BW(mass * mass - mumsRecoMass2, mass * GofM);
    fptype den = (mass * mass - mumsRecoMass2) * (mass * mass - mumsRecoMass2) + mass * GofM * mass * GofM;

    fpcomplex ret = (sqrt(k * frFactor)) / den * BW;
    // printf("m1, m2, Mpair, to2Lplus1, GofM, thisFR, pratio, mratio, pABSq , prSqForGofM, FF, ret.real, ret.imag\n");
    // printf("BW %.7g, %.7g, %.7g, %i, %i, %i, %i\n",meson_radius, resmass, reswidth, orbital, FF, indices[2],
    // indices[3]);
    // printf("BW %.7g, %.7g, %.7g, %i, %.7g, %.7g, %.7g, %.7g, %.7g, %.7g, %.7g, %.7g, %.7g\n", m1, m2, Mpair,
    // to2Lplus1, GofM, thisFR, pratio, mratio, pABSq, prSqForGofM, frFactor, ret.real, ret.imag );
    return ret;
}

// This function is modeled after SBW from the MINT package written by Jonas Rademacker.
__device__ fpcomplex SBW(fptype Mpair, fptype m1, fptype m2, unsigned int *indices) {
    fptype resmass       = GOOFIT_GET_PARAM(2);
    fptype reswidth      = GOOFIT_GET_PARAM(3);
    unsigned int orbital = GOOFIT_GET_INT(4);
    // GOOFIT_GET_INT(5, Mpair, "Mpair");
    unsigned int FF     = GOOFIT_GET_INT(6);
    fptype meson_radius = GOOFIT_GET_CONST(7);

    // fptype meson_radius  = functorConstants[indices[7]];
    // fptype resmass       = hipArray[indices[2]];
    // fptype reswidth      = hipArray[indices[3]];
    // unsigned int orbital = indices[4];
    // unsigned int FF      = indices[6];

    fptype mass          = resmass;
    fptype width         = reswidth;
    fptype mumsRecoMass2 = Mpair * Mpair;

    fptype mpsq  = (m1 + m2) * (m1 + m2);
    fptype mmsq  = (m1 - m2) * (m1 - m2);
    fptype num   = (mumsRecoMass2 - mpsq) * (mumsRecoMass2 - mmsq);
    fptype num2  = (mass * mass - mpsq) * (mass * mass - mmsq);
    fptype pABSq = num / (4 * mumsRecoMass2);
    fptype prSq  = num2 / (4 * mass * mass);
    fptype prSq2 = prSq < 0 ? 0 : prSq;
    prSq         = fabs(prSq);

    fptype r2       = meson_radius * meson_radius;
    fptype frFactor = 1;

    if(0 != orbital and 0 != FF) {
        frFactor = (FF == 1 ? BL(pABSq * r2, orbital) : BL_PRIME(pABSq * r2, prSq2 * r2, orbital));
        frFactor = (FF == 3 ? BL2(pABSq * r2, orbital) : frFactor);
    }

    fptype GofM = width;

    fptype gamma = sqrt(mass * mass * (mass * mass + width * width));
    fptype k     = mass * width * gamma / sqrt(mass * mass + gamma);

    fpcomplex BW(mass * mass - mumsRecoMass2, mass * GofM);
    fptype den = (mass * mass - mumsRecoMass2) * (mass * mass - mumsRecoMass2) + mass * GofM * mass * GofM;

    fpcomplex ret = (sqrt(k * frFactor)) / den * BW;

    // printf("m1, m2, Mpair, GofM, pABSq , prSq, FF, ret.real, ret.imag\n");
    // printf("SBW %.7g, %.7g, %.7g, %.7g, %.7g, %.7g, %.7g, %.7g, %.7g\n", m1, m2, Mpair, GofM, pABSq, prSq, frFactor,
    // ret.real, ret.imag );
    return ret;
}

__device__ fpcomplex bugg_rho2(const fptype &s, const fptype m) {
    fptype rho_squared  = 1. - 4. * m * m / s;
    fpcomplex returnVal = (rho_squared >= 0) ? fpcomplex(1, 0) : fpcomplex(0, 1);
    rho_squared         = (rho_squared >= 0) ? sqrt(rho_squared) : sqrt(-rho_squared);
    return rho_squared * returnVal;
}

__device__ fptype bugg_j1(const fptype &s, const fptype m) {
    fptype rho_pipi  = bugg_rho2(s, m).real();
    fptype returnVal = 2.;
    returnVal += (rho_pipi > 0.) ? rho_pipi * log((1. - rho_pipi) / (1. + rho_pipi)) : 0;
    return returnVal / M_PI;
}

__device__ fptype bugg_Gamma_4pi(const fptype &s,
                                 const fptype mpi,
                                 const fptype &g_4pi,
                                 const fptype &M,
                                 const fptype &lambda_4pi,
                                 const fptype &s0_4pi) {
    fptype returnVal = (s < (16. * mpi * mpi)) ? 0
                                               : g_4pi * (1. / (1 + exp(lambda_4pi * (s0_4pi - s))))
                                                     / (1. / (1 + exp(lambda_4pi * (s0_4pi - M * M))));
    return returnVal;
}

// This function is an adaptation from the bugg lineshape implemented in the MINT package written by Jonas Rademacker.
// this lineshape is not tested yet!
__device__ fpcomplex bugg_MINT(fptype Mpair, fptype m1, fptype m2, unsigned int *indices) {
    fptype s = Mpair * Mpair;

    fptype M          = 0.953;
    fptype b1         = 1.302;
    fptype b2         = 0.340;
    fptype A          = 2.426;
    fptype g_4pi      = 0.011;
    fptype g_2K       = 0.6;
    fptype g_2eta     = 0.2;
    fptype alpha      = 1.3;
    fptype sA         = 0.41;
    fptype s0_4pi     = 7.082 / 2.845;
    fptype lambda_4pi = 2.845;
    fptype mPiPlus    = .13957018;
    fptype mKPlus     = .493677;
    fptype mEta       = .54751;

    fptype g1sq = (b1 + b2 * s) * exp(-(s - M * M) / A);
    fptype z    = bugg_j1(s, mPiPlus) - bugg_j1(M * M, mPiPlus);

    fpcomplex gamma_2pi = fpcomplex(
        g1sq * (s - sA * mPiPlus * mPiPlus) / (M * M - sA * mPiPlus * mPiPlus) * bugg_rho2(s, mPiPlus).real(), 0);
    fpcomplex gamma_2K = g_2K * g1sq * s / (M * M)
                         * exp((-1) * alpha * sqrt((s - 4. * mKPlus * mKPlus) * (s - 4. * mKPlus * mKPlus)))
                         * bugg_rho2(s, mKPlus);
    fpcomplex gamma_2eta = g_2eta * g1sq * s / (M * M)
                           * exp((-1) * alpha * sqrt((s - 4. * mEta * mEta) * (s - 4. * mEta * mEta)))
                           * bugg_rho2(s, mEta);
    fpcomplex gamma_4pi = fpcomplex(bugg_Gamma_4pi(s, mPiPlus, g_4pi, M, lambda_4pi, s0_4pi), 0);

    fpcomplex Gamma_tot = gamma_2pi + gamma_2K + gamma_2eta + gamma_4pi;

    // fpcomplex num = M * gamma_2pi; //only for elastic scattering, not production
    fpcomplex den
        = fpcomplex(M * M - s - M * g1sq * (s - sA * mPiPlus * mPiPlus) / (M * M - sA * mPiPlus * mPiPlus) * z, 0)
          - fpcomplex(0, 1) * M * Gamma_tot;
    fpcomplex returnVal = 1.0 / den;
    // printf("Bugg %.5g %.5g %.5g %.5g %.5g %.5g %.5g %.5g \n",gamma_2pi.real, gamma_2pi.imag, gamma_2K.real,
    // gamma_2K.imag, gamma_2eta.real, gamma_2eta.imag, gamma_4pi.real, gamma_4pi.imag);
    // printf("Bugg %.5g %.5g %.5g %.5g %.5g %.5g %.5g %.5g %.5g \n",Mpair, Gamma_tot.real, Gamma_tot.imag, g1sq, z,
    // den.real, den.imag, returnVal.real, returnVal.imag);

    // the factor sqrt(1000) gives the correct result in comparison with mint2, I think its because BW/SBW
    // have a factor of sqrt(k) which these lineshapes dont have. For now this stays because it works. further
    // investigation needed.
    return returnVal * sqrt(1000.0);
}

__device__ fpcomplex bugg_MINT3(fptype Mpair, fptype m1, fptype m2, unsigned int *indices) {
    fptype s          = Mpair * Mpair;
    fptype M          = 0.953;
    fptype b1         = 1.302;
    fptype b2         = 0.340;
    fptype A          = 2.426;
    fptype g_4pi      = 0.011;
    fptype g_2K       = 0.6;
    fptype g_2eta     = 0.2;
    fptype alpha      = 1.3;
    fptype s0_4pi     = 7.082 / 2.845;
    fptype lambda_4pi = 2.845;
    fptype mPiPlus    = .13957018;
    fptype mKPlus     = .493677;
    fptype mEta       = .547862;
    fptype sA         = 0.41 * mPiPlus * mPiPlus;

    fptype g1sq      = M * (b1 + b2 * s) * exp(-(s - M * M) / A);
    fptype z         = bugg_j1(s, mPiPlus) - bugg_j1(M * M, mPiPlus);
    fptype adlerZero = (s - sA) / (M * M - sA);

    fptype mk4  = 4. * mKPlus * mKPlus;
    fptype me4  = 4. * mEta * mEta;
    fptype tmp1 = s > mk4 ? s - mk4 : mk4 - s;
    fptype tmp2 = s > me4 ? s - me4 : me4 - s;

    fpcomplex gamma_2pi  = fpcomplex(g1sq * adlerZero * bugg_rho2(s, mPiPlus).real(), 0);
    fpcomplex gamma_2K   = g_2K * g1sq * s / (M * M) * exp((-1) * alpha * tmp1) * bugg_rho2(s, mKPlus);
    fpcomplex gamma_2eta = g_2eta * g1sq * s / (M * M) * exp((-1) * alpha * tmp2) * bugg_rho2(s, mEta);
    fpcomplex gamma_4pi  = fpcomplex(bugg_Gamma_4pi(s, mPiPlus, g_4pi, M, lambda_4pi, s0_4pi), 0);

    fpcomplex Gamma_tot = gamma_2pi + gamma_2K + gamma_2eta + gamma_4pi;

    // fpcomplex num = M * gamma_2pi; //only for elastic scattering, not production
    fpcomplex den       = fpcomplex(M * M - s - adlerZero * g1sq * z, 0) - fpcomplex(0, 1) * Gamma_tot;
    fpcomplex returnVal = 1.0 / den;
    // printf("Bugg %.5g %.5g %.5g %.5g %.5g %.5g %.5g %.5g \n",gamma_2pi.real, gamma_2pi.imag, gamma_2K.real,
    // gamma_2K.imag, gamma_2eta.real, gamma_2eta.imag, gamma_4pi.real, gamma_4pi.imag);
    // printf("Bugg %.5g %.5g %.5g %.5g %.5g %.5g %.5g %.5g %.5g \n",Mpair, Gamma_tot.real, Gamma_tot.imag, g1sq, z,
    // den.real, den.imag, returnVal.real, returnVal.imag);

    return returnVal;
}

__device__ fpcomplex lass_MINT(fptype Mpair, fptype m1, fptype m2, unsigned int *indices) {
    fptype resmass  = hipArray[indices[2]];
    fptype reswidth = hipArray[indices[3]];
    fptype rMass2   = Mpair * Mpair;

    fptype a = 2.07;
    fptype r = 3.32;

    fptype mpsq  = (m1 + m2) * (m1 + m2);
    fptype mmsq  = (m1 - m2) * (m1 - m2);
    fptype num   = (rMass2 - mpsq) * (rMass2 - mmsq);
    fptype num2  = (resmass * resmass - mpsq) * (resmass * resmass - mmsq);
    fptype pABSq = num / (4 * rMass2);
    fptype prSq  = fabs(num2 / (4 * resmass * resmass));

    fptype y          = 2.0 * a * sqrt(pABSq);
    fptype x          = 2.0 + a * r * pABSq;
    fptype cotDeltaBg = x / y;
    fpcomplex phaseshift((cotDeltaBg * cotDeltaBg - 1) / (1 + cotDeltaBg * cotDeltaBg),
                         2 * cotDeltaBg / (1 + cotDeltaBg * cotDeltaBg));
    // (cotDeltaBg*cotDeltaBg-1)/(1+cotDeltaBg*cotDeltaBg) = cos(2*delta)     2*cotDeltaBg / ( 1 +
    // cotDeltaBg*cotDeltaBg) = sin(2*delta)
    fpcomplex den(sqrt(pABSq) * cotDeltaBg, (-1.) * sqrt(pABSq));
    fptype SF           = Mpair * sqrt(prSq) / (resmass * resmass * reswidth);
    fpcomplex BG        = SF / den;
    fpcomplex returnVal = BG + phaseshift * BW(Mpair, m1, m2, indices);
    // printf("Lass: %.5g %.5g %.5g %.5g %.5g %.5g\n",BG.real, BG.imag, phaseshift.real, phaseshift.imag,
    // returnVal.real, returnVal.imag);

    return returnVal;
}

// generalized lass lineshape as implemented in MINT3 by Tim Evans. if F=R=1 and phiF=phiR=0 this is equal to normal
// lass as implemented in Mint3.
// The difference between this and lass mint is not quite clear to me. need to get back to this later.
__device__ fpcomplex glass_MINT3(fptype Mpair, fptype m1, fptype m2, unsigned int *indices) {
    fptype meson_radius  = functorConstants[indices[7]];
    fptype resmass       = hipArray[indices[2]];
    fptype reswidth      = hipArray[indices[3]];
    unsigned int orbital = indices[4];
    fptype rMass2        = Mpair * Mpair;

    // fptype a = 2.07;
    // fptype r = 3.32;
    // fptype phiF = 0.0;
    // fptype phiR = 0.0;
    // fptype F = 1.0;
    fptype a    = hipArray[indices[8]];
    fptype r    = hipArray[indices[9]];
    fptype phiF = hipArray[indices[10]];
    fptype phiR = hipArray[indices[11]];
    fptype F    = hipArray[indices[12]];

    fptype R = 1.0;
    // printf("GLass: %.5g %.5g %.5g %.5g %.5g %.5g\n",a, r, phiF, phiR, F, R);
    // printf("GLass2: %.5g %.5g %.5g %u \n",meson_radius, resmass, reswidth, orbital);

    fptype mpsq  = (m1 + m2) * (m1 + m2);
    fptype mmsq  = (m1 - m2) * (m1 - m2);
    fptype num   = (rMass2 - mpsq) * (rMass2 - mmsq);
    fptype num2  = (resmass * resmass - mpsq) * (resmass * resmass - mmsq);
    fptype pABSq = num / (4 * rMass2);
    fptype prSq  = fabs(num2 / (4 * resmass * resmass));

    fptype pratio = sqrt(pABSq / prSq);

    fptype pratio_to_2Jplus1 = 1;

    for(int i = 0; i < 2 * orbital + 1; i++) {
        pratio_to_2Jplus1 *= pratio;
    }

    fptype mratio = resmass / Mpair;
    fptype r2     = meson_radius * meson_radius;
    fptype thisFR = BL_PRIME(pABSq * r2, prSq * r2, orbital);
    fptype GofM   = reswidth * pratio_to_2Jplus1 * mratio * thisFR;

    fptype y          = 2.0 * a * sqrt(pABSq);
    fptype x          = 2.0 + a * r * pABSq;
    fptype scattphase = phiF + atan(y / x);
    fptype resphase   = phiR + atan(resmass * GofM / (resmass * resmass - rMass2));
    fptype rho        = 1.0 / sqrt(pABSq / rMass2);
    fpcomplex returnVal
        = (F * sin(scattphase) * fpcomplex(cos(scattphase), sin(scattphase))
           + R * sin(resphase) * fpcomplex(cos(resphase + 2 * scattphase), sin(resphase + 2 * scattphase)))
          * rho;
    // printf("GLass3: %.5g %.5g %.5g %.5g %.5g %.5g\n",rMass2, pABSq, rho, GofM, scattphase, resphase);

    // printf("GLass4: %.5g %.5g\n",returnVal.real, returnVal.imag);
    return returnVal;
}

__device__ fpcomplex aSqrtTerm(const fptype &m0, const fptype &m) {
    fptype a2           = 1 - (2 * m0 / m) * (2 * m0 / m);
    fpcomplex returnVal = a2 > 0 ? fpcomplex(sqrt(a2), 0) : fpcomplex(0, sqrt(-a2));
    return returnVal;
}

__device__ fpcomplex Flatte_MINT(fptype Mpair, fptype m1, fptype m2, unsigned int *indices) {
    fptype meson_radius  = functorConstants[indices[7]];
    fptype resmass       = hipArray[indices[2]];
    unsigned int orbital = indices[4];
    fptype frFactor      = 1;
    fptype rMass2        = Mpair * Mpair;

    // As far as I understand, this is only valid for the f980
    fptype gPi       = .165;
    fptype gK_by_gPi = 4.21;
    fptype gK        = gPi * gK_by_gPi;
    fptype mPi0      = .1349766;
    fptype mPiPlus   = .13957018;
    fptype mKPlus    = .493677;
    fptype mK0       = .497648;

    fptype mpsq = (m1 + m2) * (m1 + m2);
    fptype mmsq = (m1 - m2) * (m1 - m2);
    fptype num  = (rMass2 - mpsq) * (rMass2 - mmsq);
    // fptype num2  = (resmass*resmass - mpsq)*(resmass*resmass - mmsq);
    fptype pABSq = num / (4 * rMass2);
    // fptype prSq = fabs(num2/(4*resmass*resmass));

    fpcomplex Gpipi       = (1. / 3.) * aSqrtTerm(mPi0, Mpair) + (2. / 3.) * aSqrtTerm(mPiPlus, Mpair);
    fpcomplex GKK         = (1. / 2.) * aSqrtTerm(mK0, Mpair) + (1. / 2.) * aSqrtTerm(mKPlus, Mpair);
    fpcomplex FlatteWidth = gPi * Gpipi + gK * GKK;
    // printf("%.5g %.5g %.5g %.5g %.5g %.5g %.5g %.5g \n",Gpipi.real, Gpipi.imag, GKK.real, GKK.imag, FlatteWidth.real,
    // FlatteWidth.imag, Mpair, pABSq);

    frFactor     = BL2(pABSq * meson_radius * meson_radius, orbital);
    fpcomplex BW = sqrt(frFactor) / fpcomplex(resmass * resmass - rMass2, 0) - fpcomplex(0, 1) * resmass * FlatteWidth;
    return BW;
}

__device__ fpcomplex Spline_TDP(fptype Mpair, fptype m1, fptype m2, unsigned int *indices) {
    fpcomplex ret(0, 0);

    const unsigned int nKnobs = indices[4]; // orbital

    unsigned int idx                 = 5; // Next index
    unsigned int i                   = 0;
    const unsigned int pwa_coefs_idx = idx;
    idx += 2 * nKnobs;
    const fptype *mKKlimits = &(functorConstants[indices[idx]]);
    fptype mAB = Mpair, mAC = m1, mBC = m2;

    int khiAB = 0, khiAC = 0;
    fptype dmKK, aa, bb, aa3, bb3;
    unsigned int timestorun = 1; //+doSwap;
    while(khiAB < nKnobs) {
        if(mAB < mKKlimits[khiAB])
            break;
        khiAB++;
    }

    if(khiAB <= 0 || khiAB == nKnobs)
        timestorun = 0;
    while(khiAC < nKnobs) {
        if(mAC < mKKlimits[khiAC])
            break;
        khiAC++;
    }

    if(khiAC <= 0 || khiAC == nKnobs)
        timestorun = 0;

    for(i = 0; i < timestorun; i++) {
        unsigned int kloAB                = khiAB - 1; //, kloAC = khiAC -1;
        unsigned int twokloAB             = kloAB + kloAB;
        unsigned int twokhiAB             = khiAB + khiAB;
        fptype pwa_coefs_real_kloAB       = hipArray[indices[pwa_coefs_idx + twokloAB]];
        fptype pwa_coefs_real_khiAB       = hipArray[indices[pwa_coefs_idx + twokhiAB]];
        fptype pwa_coefs_imag_kloAB       = hipArray[indices[pwa_coefs_idx + twokloAB + 1]];
        fptype pwa_coefs_imag_khiAB       = hipArray[indices[pwa_coefs_idx + twokhiAB + 1]];
        fptype pwa_coefs_prime_real_kloAB = 0; // cDeriatives[twokloAB];
        fptype pwa_coefs_prime_real_khiAB = 0; // cDeriatives[twokhiAB];
        fptype pwa_coefs_prime_imag_kloAB = 0; // cDeriatives[twokloAB+1];
        fptype pwa_coefs_prime_imag_khiAB = 0; // cDeriatives[twokhiAB+1];
        //  printf("m12: %f: %f %f %f %f %f %f %d %d %d\n", mAB, mKKlimits[0], mKKlimits[nKnobs-1],
        //  pwa_coefs_real_khiAB, pwa_coefs_imag_khiAB, pwa_coefs_prime_real_khiAB, pwa_coefs_prime_imag_khiAB, khiAB,
        //  khiAC, timestorun );

        dmKK = mKKlimits[khiAB] - mKKlimits[kloAB];
        aa   = (mKKlimits[khiAB] - mAB) / dmKK;
        bb   = 1 - aa;
        aa3  = aa * aa * aa;
        bb3  = bb * bb * bb;
        //  ret += aa * pwa_coefs[kloAB] + bb * pwa_coefs[khiAB] + ((aa3 - aa)*pwa_coefs_prime[kloAB] + (bb3 - bb) *
        //  pwa_coefs_prime[khiAB]) * (dmKK*dmKK)/6.0;
        ret.real(ret.real() + aa * pwa_coefs_real_kloAB + bb * pwa_coefs_real_khiAB
                 + ((aa3 - aa) * pwa_coefs_prime_real_kloAB + (bb3 - bb) * pwa_coefs_prime_real_khiAB) * (dmKK * dmKK)
                       / 6.0);
        ret.imag(ret.imag() + aa * pwa_coefs_imag_kloAB + bb * pwa_coefs_imag_khiAB
                 + ((aa3 - aa) * pwa_coefs_prime_imag_kloAB + (bb3 - bb) * pwa_coefs_prime_imag_khiAB) * (dmKK * dmKK)
                       / 6.0);
        khiAB = khiAC;
        mAB   = mAC;
    }
    return ret;
}

__device__ fpcomplex nonres_DP(fptype Mpair, fptype m1, fptype m2, unsigned int *indices) {
    fptype meson_radius  = functorConstants[indices[7]];
    unsigned int orbital = indices[4];

    fptype mumsRecoMass2 = Mpair * Mpair;

    fptype mpsq       = (m1 + m2) * (m1 + m2);
    fptype mmsq       = (m1 - m2) * (m1 - m2);
    fptype num        = (mumsRecoMass2 - mpsq) * (mumsRecoMass2 - mmsq);
    fptype pABSq      = num / (4 * mumsRecoMass2);
    fptype formfactor = sqrt(BL2(pABSq * meson_radius * meson_radius, orbital));
    // printf("NonRes q2:%.7g FF:%.7g, s %.7g m1 %.7g m2 %.7g r %.7g L %u \n",pABSq, formfactor, mumsRecoMass2,
    // m1,m2,meson_radius, orbital );
    return fpcomplex(1, 0) * formfactor;
}

__device__ resonance_function_ptr ptr_to_LS_ONE     = LS_ONE;
__device__ resonance_function_ptr ptr_to_BW_DP4     = BW;
__device__ resonance_function_ptr ptr_to_lass       = lass_MINT;
__device__ resonance_function_ptr ptr_to_glass3     = glass_MINT3;
__device__ resonance_function_ptr ptr_to_bugg_MINT  = bugg_MINT;
__device__ resonance_function_ptr ptr_to_bugg_MINT3 = bugg_MINT3;
__device__ resonance_function_ptr ptr_to_SBW        = SBW;
__device__ resonance_function_ptr ptr_to_NONRES_DP  = nonres_DP;
__device__ resonance_function_ptr ptr_to_Flatte     = Flatte_MINT;
__device__ resonance_function_ptr ptr_to_Spline     = Spline_TDP;

Lineshape::Lineshape(std::string name,
                     Variable *mass,
                     Variable *width,
                     unsigned int L,
                     unsigned int Mpair,
                     LS kind,
                     FF FormFac,
                     fptype radius,
                     std::vector<Variable *> AdditionalVars,
                     spline_t SplineInfo)
    : GooPdf(nullptr, name)
    , _mass(mass)
    , _width(width)
    , _L(L)
    , _Mpair(Mpair)
    , _kind(kind)
    , _FormFac(FormFac)
    , _SplineInfo(SplineInfo) {
    GOOFIT_START_PDF;

    // Making room for index of decay-related constants. Assumption:
    // These are mother mass and three daughter masses in that order.
    // They will be registered by the object that uses this resonance,
    // which will tell this object where to find them by calling setConstantIndex.

    GOOFIT_ADD_PARAM(2, mass, "mass");
    GOOFIT_ADD_PARAM(3, width, "width");

    GOOFIT_ADD_INT(4, L, "L");
    GOOFIT_ADD_INT(5, Mpair, "Mpair");

    GOOFIT_ADD_INT(6, enum_to_underlying(FormFac), "FormFac");

    GOOFIT_ADD_CONST(7, radius, "radius");

    // pindices.push_back(registerParameter(mass));
    // pindices.push_back(registerParameter(width));
    // pindices.push_back(L);
    // pindices.push_back(Mpair);
    // pindices.push_back(enum_to_underlying(FormFac));

    // pindices.push_back(registerConstants(1));
    // MEMCPY_TO_SYMBOL(functorConstants, &radius, sizeof(fptype), cIndex * sizeof(fptype), hipMemcpyHostToDevice);

    switch(kind) {
    case LS::ONE:
        GET_FUNCTION_ADDR(ptr_to_LS_ONE);
        break;

    case LS::BW:
        GET_FUNCTION_ADDR(ptr_to_BW_DP4);
        break;

    case LS::Lass:
        GET_FUNCTION_ADDR(ptr_to_lass);
        break;

    case LS::Lass_M3:
        if(5 != AdditionalVars.size()) {
            fprintf(stderr,
                    "It seems you forgot to provide the vector with the five necessary variables for GLASS, a, "
                    "r, phiF, phiR and F (in that order)");
            exit(0);
        }

        for(int i = 0; i < 5; i++) {
            GOOFIT_ADD_PARAM(8 + i, AdditionalVars[i], "LassVars");
            // pindices.push_back(registerParameter(AdditionalVars[i]));
        }

        GET_FUNCTION_ADDR(ptr_to_glass3);
        break;

    case LS::nonRes:
        GET_FUNCTION_ADDR(ptr_to_NONRES_DP);
        break;

    case LS::Bugg:
        GET_FUNCTION_ADDR(ptr_to_bugg_MINT);
        break;

    case LS::Bugg3:
        GET_FUNCTION_ADDR(ptr_to_bugg_MINT3);
        break;

    case LS::SBW:
        GET_FUNCTION_ADDR(ptr_to_SBW);
        break;

    case LS::Flatte:
        GET_FUNCTION_ADDR(ptr_to_Flatte);
        break;

    case LS::Spline:
        if(std::get<2>(_SplineInfo) != AdditionalVars.size())
            throw GeneralError("bins {} != vars {}", std::get<2>(_SplineInfo), AdditionalVars.size());
        GOOFIT_ADD_CONST(8, std::get<0>(_SplineInfo), "MinSpline");
        GOOFIT_ADD_CONST(9, std::get<1>(_SplineInfo), "MaxSpline");
        GOOFIT_ADD_CONST(10, std::get<2>(_SplineInfo), "NSpline");
        {
            int i = 11;
            for(auto &par : AdditionalVars) {
                GOOFIT_ADD_PARAM(i++, par, "Knot");
            }
        }

        GET_FUNCTION_ADDR(ptr_to_Spline);
        break;

    default:
        throw GeneralError("It seems that the requested lineshape is not implemented yet. Check LineshapesPdf.cu");
    }

    GOOFIT_FINALIZE_PDF;
}

Lineshape::Lineshape(std::string name)
    : GooPdf(nullptr, name) {
    GOOFIT_START_PDF;
    GET_FUNCTION_ADDR(ptr_to_NONRES_DP);
    GOOFIT_FINALIZE_PDF;
}

Amplitude::Amplitude(std::string uniqueDecayStr,
                     Variable *ar,
                     Variable *ai,
                     std::vector<Lineshape *> LS,
                     std::vector<SpinFactor *> SF,
                     unsigned int nPerm)
    : _uniqueDecayStr(std::move(uniqueDecayStr))
    , _ar(ar)
    , _ai(ai)
    , _SF(std::move(SF))
    , _LS(std::move(LS))
    , _nPerm(nPerm) {}

bool Amplitude::operator==(const Amplitude &A) const {
    return _uniqueDecayStr == A._uniqueDecayStr and _ar == A._ar and _ai == A._ai and _LS == A._LS and _SF == A._SF
           and _nPerm == A._nPerm;
}

} // namespace GooFit
