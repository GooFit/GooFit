#include "hip/hip_runtime.h"
#include <goofit/Error.h>
#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/physics/TddpPdf.h>

#include <thrust/transform_reduce.h>

#ifdef GOOFIT_MPI
#include <mpi.h>
#endif

namespace GooFit {

const int resonanceOffset = 8; // Offset of the first resonance into the parameter index array
// Offset is number of parameters, constant index, indices for tau, xmix, and ymix, index
// of resolution function, and finally number of resonances (not calculable from nP
// because we don't know what the efficiency and time resolution might need). Efficiency
// and time-resolution parameters are after the resonance information.
const unsigned int SPECIAL_RESOLUTION_FLAG = 999999999;

// The function of this array is to hold all the cached waves; specific
// waves are recalculated when the corresponding resonance mass or width
// changes. Note that in a multithread environment each thread needs its
// own cache, hence the '10'. Ten threads should be enough for anyone!

// NOTE: only one set of wave holders is supported currently.
__device__ WaveHolder_s *cWaves[16];

__device__ inline int parIndexFromResIndex(int resIndex) { return resonanceOffset + resIndex * resonanceSize; }

__device__ fpcomplex getResonanceAmplitude(fptype m12, fptype m13, fptype m23, ParameterContainer &pc) {
    auto func = reinterpret_cast<resonance_function_ptr>(device_function_table[pc.funcIdx]);
    return (*func)(m12, m13, m23, pc);
}

__device__ ThreeComplex
device_Tddp_calcIntegrals(fptype m12, fptype m13, int res_i, int res_j, ParameterContainer &pc) {
    // For calculating Dalitz-plot integrals. What's needed is the products
    // AiAj*, AiBj*, and BiBj*, where
    // Ai = BW_i(x, y) + BW_i(y, x)
    // and Bi reverses the sign of the second BW.
    // This function returns the above values at a single point.
    // NB: Multiplication by efficiency is done by the calling function.
    // Note that this function expects
    // to be called on a normalization grid, not on
    // observed points, that's why it doesn't use
    // cWaves. No need to cache the values at individual
    // grid points - we only care about totals.

    ThreeComplex ret;

    if(!inDalitz(m12, m13, c_motherMass, c_daug1Mass, c_daug2Mass, c_daug3Mass))
        return ret;

    fptype m23 = c_motherMass * c_motherMass + c_daug1Mass * c_daug1Mass + c_daug2Mass * c_daug2Mass
                 + c_daug3Mass * c_daug3Mass - m12 - m13;

    ParameterContainer ipc = pc;
    while(ipc.funcIdx < res_i)
        ipc.incrementIndex();

    ParameterContainer t = ipc;
    fpcomplex ai         = getResonanceAmplitude(m12, m13, m23, t);
    t                    = ipc;
    fpcomplex bi         = getResonanceAmplitude(m13, m12, m23, t);

    ParameterContainer jpc = pc;
    while(jpc.funcIdx < res_j)
        jpc.incrementIndex();

    t            = jpc;
    fpcomplex aj = conj(getResonanceAmplitude(m12, m13, m23, t));
    t            = jpc;
    fpcomplex bj = conj(getResonanceAmplitude(m13, m12, m23, t));

    ret = ThreeComplex(
        (ai * aj).real(), (ai * aj).imag(), (ai * bj).real(), (ai * bj).imag(), (bi * bj).real(), (bi * bj).imag());
    return ret;
}

__device__ fptype device_Tddp(fptype *evt, ParameterContainer &pc) {
    int num_parameters  = pc.getNumParameters();
    int num_constants   = pc.getNumConstants();
    int num_observables = pc.getNumObservables();

    int id_m12 = pc.getObservable(2);
    int id_m13 = pc.getObservable(3);
    int id_num = pc.getObservable(4);
    int id_mis = 0;
    if(num_observables > 5)
        id_mis = pc.getObservable(5);

    fptype m12 = evt[id_m12];
    fptype m13 = evt[id_m13];

    unsigned int numResonances = pc.getConstant(0);

    if(!inDalitz(m12, m13, c_motherMass, c_daug1Mass, c_daug2Mass, c_daug3Mass)) {
        unsigned int endEfficiencyFunc = RO_CACHE(pc.constants[pc.constantIdx + 4]);
        pc.incrementIndex(1, num_parameters, num_constants, num_observables, 1);

        // increment the resonances
        for(int i = 0; i < numResonances; i++)
            pc.incrementIndex();

        // increment the resolution function
        pc.incrementIndex();

        // increment our efficiency function
        // pc.incrementIndex();
        while(pc.funcIdx < endEfficiencyFunc)
            pc.incrementIndex();
        return 0;
    }

    auto evtNum = static_cast<int>(floor(0.5 + evt[id_num]));

    fpcomplex sumWavesA(0, 0);
    fpcomplex sumWavesB(0, 0);
    fpcomplex sumRateAA(0, 0);
    fpcomplex sumRateAB(0, 0);
    fpcomplex sumRateBB(0, 0);

    // unsigned int cacheToUse = pc.getConstant(1);
    fptype mistag = pc.getConstant(2);

    for(int i = 0; i < numResonances; ++i) {
        // int paramIndex = parIndexFromResIndex(i);
        fpcomplex amp{pc.getParameter(i * 2 + 3), pc.getParameter(i * 2 + 4)};

        // fpcomplex matrixelement(thrust::get<0>(cWaves[cacheToUse][evtNum*numResonances + i]),
        //				     thrust::get<1>(cWaves[cacheToUse][evtNum*numResonances + i]));
        // Note, to make this more efficient we should change it to only an array of fptype's, and read double2 at a
        // time.
        fpcomplex ai{RO_CACHE(cWaves[i][evtNum].ai_real), RO_CACHE(cWaves[i][evtNum].ai_imag)};
        fpcomplex bi{RO_CACHE(cWaves[i][evtNum].bi_real), RO_CACHE(cWaves[i][evtNum].bi_imag)};

        fpcomplex matrixelement = ai * amp;
        sumWavesA += matrixelement;

        // matrixelement = fpcomplex(thrust::get<2>(cWaves[cacheToUse][evtNum*numResonances + i]),
        //				       thrust::get<3>(cWaves[cacheToUse][evtNum*numResonances + i]));
        matrixelement = bi * amp;
        sumWavesB += matrixelement;
    }

    fptype _tau     = pc.getParameter(0);
    fptype _xmixing = pc.getParameter(1);
    fptype _ymixing = pc.getParameter(2);

    int id_time  = pc.getObservable(0);
    int id_sigma = pc.getObservable(1);

    fptype _time  = evt[id_time];
    fptype _sigma = evt[id_sigma];

    // TODO: Test that we have a special flag by comparing size of numconstants?
    // fptype special_flag = pc.getConstant(3);

    // if ((gpuDebug & 1) && (0 == BLOCKIDX) && (0 == THREADIDX))
    // if (0 == evtNum) printf("TDDP: (%f, %f) (%f, %f)\n", sumWavesA.real, sumWavesA.imag, sumWavesB.real,
    // sumWavesB.imag);
    // printf("TDDP: %f %f %f %f | %f %f %i\n", m12, m13, _time, _sigma, _xmixing, _tau, evtNum);

    /*
    fptype ret = 0;
    ret += (norm2(sumWavesA) + norm2(sumWavesB))*cosh(_ymixing * _time);
    ret += (norm2(sumWavesA) - norm2(sumWavesB))*cos (_xmixing * _time);
    sumWavesA *= conj(sumWavesB);
    ret -= 2*sumWavesA.real * sinh(_ymixing * _time);
    ret -= 2*sumWavesA.imag * sin (_xmixing * _time); // Notice sign difference wrt to Mikhail's code, because I have
    AB* and he has A*B.
    ret *= exp(-_time);
    */

    fptype term1 = thrust::norm(sumWavesA) + thrust::norm(sumWavesB);
    fptype term2 = thrust::norm(sumWavesA) - thrust::norm(sumWavesB);
    sumWavesA *= conj(sumWavesB);
    // printf("(%i, %i) TDDP: %f %f %f %f %f %f %f\n", BLOCKIDX, THREADIDX, term1, term2, sumWavesA.real,
    // sumWavesA.imag, m12, m13, _tau);

    // Cannot use callFunction on resolution function.
    // int effFunctionIdx = parIndexFromResIndex(numResonances);
    // int resFunctionIdx = RO_CACHE(indices[5]);
    // int resFunctionPar = 2 + effFunctionIdx;
    fptype ret = 0;
    // int md0_offset     = 0;

    // if(resFunctionIdx == SPECIAL_RESOLUTION_FLAG) {
    // In this case there are multiple resolution functions, they are stored after the efficiency function,
    // and which one we use depends on the measured mother-particle mass.
    //    md0_offset     = 1;
    // int id_massd0 = pc.constants[pc.constantIdx + 6];
    // fptype massd0  = RO_CACHE(evt[id_massd0]);
    // fptype minMass = RO_CACHE(pc.constants[pc.constantIdx + 7]);
    // fptype md0Step = RO_CACHE(pc.constants[pc.constantIdx + 8]);
    // int res_to_use = (massd0 <= minMass) ? 0 : static_cast<int>(floor((massd0 - minMass) / md0Step));
    // int maxFcn     = RO_CACHE(indices[2 + effFunctionIdx]);

    //    if(res_to_use > maxFcn)
    //        res_to_use = maxFcn;

    // Now calculate index of resolution function.
    // At the end of the array are indices efficiency_function, efficiency_parameters, maxFcn, res_function_1,
    // res_function_1_nP, par1, par2 ... res_function_2, res_function_2_nP, ...
    // res_to_use = 3 + effFunctionIdx + res_to_use * (2 + RO_CACHE(indices[effFunctionIdx + 4]));
    // NB this assumes all resolution functions have the same number of parameters. The number
    // of parameters in the first resolution function is stored in effFunctionIdx+3; add one to
    // account for the index of the resolution function itself in the device function table, one
    // to account for the number-of-parameters index, and this is the space taken up by each
    // resolution function. Multiply by res_to_use to get the number of spaces to skip to get to
    // the one we want.

    // resFunctionIdx = RO_CACHE(indices[res_to_use]);
    // resFunctionPar = res_to_use + 1;
    //}

    pc.incrementIndex(1, num_parameters, num_constants, num_observables, 1);

    // increment over resonance functions here?
    for(int i = 0; i < numResonances; i++)
        pc.incrementIndex();

    ret = (*(reinterpret_cast<device_resfunction_ptr>(device_function_table[pc.funcIdx])))(
        term1, term2, sumWavesA.real(), sumWavesA.imag(), _tau, _time, _xmixing, _ymixing, _sigma, pc);

    // For the reversed (mistagged) fraction, we make the
    // interchange A <-> B. So term1 stays the same,
    // term2 changes sign, and AB* becomes BA*.
    // Efficiency remains the same for the mistagged part,
    // because it depends on the momenta of the pi+ and pi-,
    // which don't change even though we tagged a D0 as D0bar.

    // fptype mistag = RO_CACHE(functorConstants[RO_CACHE(indices[1]) + 5]);

    if(mistag > 0) { // This should be either true or false for all events, so no branch is caused.
        // See header file for explanation of 'mistag' variable - it is actually the probability
        // of having the correct sign, given that we have a correctly reconstructed D meson.
        mistag = evt[id_mis];
        ret *= mistag;
        ret += (1 - mistag)
               * (*(reinterpret_cast<device_resfunction_ptr>(device_function_table[pc.funcIdx])))(
                     term1, -term2, sumWavesA.real(), -sumWavesA.imag(), _tau, _time, _xmixing, _ymixing, _sigma, pc);
    }

    // increment our resolution function
    pc.incrementIndex();

    fptype eff = callFunction(evt, pc);
    // internalDebug = 0;
    ret *= eff;

    return ret;
}

__device__ device_function_ptr ptr_to_Tddp = device_Tddp;

__host__ TddpPdf::TddpPdf(std::string n,
                          Observable _dtime,
                          Observable _sigmat,
                          Observable m12,
                          Observable m13,
                          EventNumber eventNumber,
                          DecayInfo3t decay,
                          MixingTimeResolution *r,
                          GooPdf *efficiency,
                          Observable *mistag)
    : GooPdf(n, _dtime, _sigmat, m12, m13, eventNumber)
    , decayInfo(decay)
    , _m12(m12)
    , _m13(m13)
    , resolution(r)
    , totalEventSize(6) // Default 5 = m12, m13, time, sigma_t, evtNum
{
    for(auto &cachedWave : cachedWaves)
        cachedWave = nullptr;

    if(mistag) {
        registerObservable(*mistag);
        totalEventSize = 6;
    }

    MEMCPY_TO_SYMBOL(c_motherMass, &decay.motherMass, sizeof(fptype), 0, hipMemcpyHostToDevice);
    MEMCPY_TO_SYMBOL(c_daug1Mass, &decay.daug1Mass, sizeof(fptype), 0, hipMemcpyHostToDevice);
    MEMCPY_TO_SYMBOL(c_daug2Mass, &decay.daug2Mass, sizeof(fptype), 0, hipMemcpyHostToDevice);
    MEMCPY_TO_SYMBOL(c_daug3Mass, &decay.daug3Mass, sizeof(fptype), 0, hipMemcpyHostToDevice);
    MEMCPY_TO_SYMBOL(c_meson_radius, &decay.meson_radius, sizeof(fptype), 0, hipMemcpyHostToDevice);

    registerParameter(decay._tau);
    registerParameter(decay._xmixing);
    registerParameter(decay._ymixing);

    if(resolution->getDeviceFunction() < 0)
        throw GooFit::GeneralError("The resolution device function index {} must be more than 0",
                                   resolution->getDeviceFunction());

    registerConstant(decay.resonances.size());

    static int cacheCount = 0;
    cacheToUse            = cacheCount++;
    registerConstant(cacheToUse);

    if(mistag == nullptr)
        registerConstant(1);
    else
        registerConstant(0);

    for(auto &resonance : decay.resonances) {
        registerParameter(resonance->amp_real);
        registerParameter(resonance->amp_imag);
        components.push_back(resonance);
    }

    components.push_back(resolution);
    components.push_back(efficiency);

    // this is the funcID after the efficiency routine
    registerConstant(0);

    // TODO: Figure out what this needs?
    resolution->createParameters(this);

    registerFunction("ptr_to_Tddp", ptr_to_Tddp);

    initialize();

    redoIntegral = new bool[decay.resonances.size()];
    cachedMasses = new fptype[decay.resonances.size()];
    cachedWidths = new fptype[decay.resonances.size()];
    integrals    = new ThreeComplex **[decay.resonances.size()];
    integrators  = new SpecialDalitzIntegrator **[decay.resonances.size()];
    calculators  = new SpecialWaveCalculator *[decay.resonances.size()];

    for(int i = 0; i < decay.resonances.size(); ++i) {
        redoIntegral[i] = true;
        cachedMasses[i] = -1;
        cachedWidths[i] = -1;
        integrators[i]  = new SpecialDalitzIntegrator *[decay.resonances.size()];
        calculators[i]  = new SpecialWaveCalculator(parameters, i);
        integrals[i]    = new ThreeComplex *[decay.resonances.size()];

        for(int j = 0; j < decay.resonances.size(); ++j) {
            integrals[i][j]   = new ThreeComplex(0, 0, 0, 0, 0, 0);
            integrators[i][j] = new SpecialDalitzIntegrator(parameters, i, j);
        }
    }

    addSpecialMask(PdfBase::ForceSeparateNorm);
}

__host__ TddpPdf::TddpPdf(std::string n,
                          Observable _dtime,
                          Observable _sigmat,
                          Observable m12,
                          Observable m13,
                          EventNumber eventNumber,
                          DecayInfo3t decay,
                          std::vector<MixingTimeResolution *> &r,
                          GooPdf *efficiency,
                          Observable md0,
                          Observable *mistag)
    : GooPdf(n, _dtime, _sigmat, m12, m13, eventNumber, md0)
    , decayInfo(decay)
    , _m12(m12)
    , _m13(m13)
    , resolution(
          r[0]) // Only used for normalization, which only depends on x and y - it doesn't matter which one we use.
    , totalEventSize(6) // This case adds the D0 mass by default.
{
    for(auto &cachedWave : cachedWaves)
        cachedWave = nullptr;

    if(mistag) {
        registerObservable(*mistag);
        totalEventSize++;
    }

    MEMCPY_TO_SYMBOL(c_motherMass, &decay.motherMass, sizeof(fptype), 0, hipMemcpyHostToDevice);
    MEMCPY_TO_SYMBOL(c_daug1Mass, &decay.daug1Mass, sizeof(fptype), 0, hipMemcpyHostToDevice);
    MEMCPY_TO_SYMBOL(c_daug2Mass, &decay.daug2Mass, sizeof(fptype), 0, hipMemcpyHostToDevice);
    MEMCPY_TO_SYMBOL(c_daug3Mass, &decay.daug3Mass, sizeof(fptype), 0, hipMemcpyHostToDevice);
    MEMCPY_TO_SYMBOL(c_meson_radius, &decay.meson_radius, sizeof(fptype), 0, hipMemcpyHostToDevice);

    registerParameter(decay._tau);
    registerParameter(decay._xmixing);
    registerParameter(decay._ymixing);
    printf("Multiple resolution functions not supported yet!\n");

    registerConstant(decayInfo.resonances.size());

    static int cacheCount = 0;
    cacheToUse            = cacheCount++;
    registerConstant(cacheToUse);

    if(mistag)
        registerConstant(1);
    else
        registerConstant(0);

    registerConstant(SPECIAL_RESOLUTION_FLAG);

    // TODO: Do these need to be set as constants?
    registerConstant(md0.getLowerLimit());
    registerConstant((md0.getUpperLimit() - md0.getLowerLimit()) / r.size());

    for(auto &resonance : decayInfo.resonances) {
        registerParameter(resonance->amp_real);
        registerParameter(resonance->amp_imag);

        components.push_back(resonance);
    }

    // components.push_back(resolution);

    for(auto &i : r) {
        if(i->getDeviceFunction() < 0)
            throw GooFit::GeneralError("Device function index {} must be more than 0", i->getDeviceFunction());

        // TODO:
        i->createParameters(this);

        components.push_back(i);
    }

    components.push_back(efficiency);

    registerFunction("ptr_to_Tddp", ptr_to_Tddp);

    initialize();

    // this is the funcID after the efficiency routine
    registerConstant(0);

    redoIntegral = new bool[decay.resonances.size()];
    cachedMasses = new fptype[decay.resonances.size()];
    cachedWidths = new fptype[decay.resonances.size()];
    integrals    = new ThreeComplex **[decay.resonances.size()];
    integrators  = new SpecialDalitzIntegrator **[decay.resonances.size()];
    calculators  = new SpecialWaveCalculator *[decay.resonances.size()];

    for(int i = 0; i < decay.resonances.size(); ++i) {
        redoIntegral[i] = true;
        cachedMasses[i] = -1;
        cachedWidths[i] = -1;
        integrators[i]  = new SpecialDalitzIntegrator *[decay.resonances.size()];
        calculators[i]  = new SpecialWaveCalculator(parameters, i);
        integrals[i]    = new ThreeComplex *[decay.resonances.size()];

        for(int j = 0; j < decay.resonances.size(); ++j) {
            integrals[i][j]   = new ThreeComplex(0, 0, 0, 0, 0, 0);
            integrators[i][j] = new SpecialDalitzIntegrator(parameters, i, j);
        }
    }

    addSpecialMask(PdfBase::ForceSeparateNorm);
}

// Note: We need to manually populate the arrays so we can track the efficiency function!
__host__ void TddpPdf::populateArrays() {
    // populate all the arrays
    GOOFIT_TRACE("TddpPdf: Populating Arrays for {}", getName());

    // reconfigure the host_parameters array with the new indexing scheme.
    GOOFIT_TRACE("host_parameters[{}] = {}", totalParameters, parametersList.size());
    host_parameters[totalParameters] = parametersList.size();
    parametersIdx                    = totalParameters;
    totalParameters++;
    for(auto &i : parametersList) {
        GOOFIT_TRACE("host_parameters[{}] = {}", totalParameters, i.getValue());
        host_parameters[totalParameters] = i.getValue();
        totalParameters++;
    }

    GOOFIT_TRACE("host_constants[{}] = {}", totalConstants, constantsList.size());
    host_constants[totalConstants] = constantsList.size();
    constantsIdx                   = totalConstants;
    totalConstants++;
    for(double i : constantsList) {
        GOOFIT_TRACE("host_constants[{}] = {}", totalConstants, i);
        host_constants[totalConstants] = i;
        totalConstants++;
    }

    GOOFIT_TRACE("host_observables[{}] = {}", totalObservables, observablesList.size());
    host_observables[totalObservables] = observablesList.size();
    observablesIdx                     = totalObservables;
    totalObservables++;
    for(auto &i : observablesList) {
        GOOFIT_TRACE("host_observables[{}] = {}", totalObservables, i.getIndex());
        host_observables[totalObservables] = i.getIndex();
        totalObservables++;
    }

    GOOFIT_TRACE("host_normalizations[{}] = {}", totalNormalizations, 1);
    host_normalizations[totalNormalizations] = 1;
    normalIdx                                = totalNormalizations++;
    GOOFIT_TRACE("host_normalizations[{}] = {}", totalNormalizations, 0);
    host_normalizations[totalNormalizations] = cachedNormalization;
    totalNormalizations++;

    int numResonances = decayInfo.resonances.size();

    // add our resonance functions
    for(unsigned int i = 0; i < numResonances; i++)
        components[i]->recursiveSetIndices();

    // TODO: Add resolution function here
    resolutionFunction = num_device_functions;
    components[numResonances]->recursiveSetIndices();

    // Next index starts our efficiency function
    efficiencyFunction = num_device_functions;
    for(unsigned int i = numResonances + 1; i < components.size(); i++)
        components[i]->recursiveSetIndices();

    // update constants
    constantsList[constantsList.size() - 1] = num_device_functions;
    GOOFIT_TRACE("Rewriting constants!");
    for(int i = 0; i < constantsList.size(); i++) {
        GOOFIT_TRACE("host_constants[{}] = {}", constantsIdx, constantsList[i]);
        host_constants[constantsIdx + 1 + i] = constantsList[i];
    }
}
__host__ void TddpPdf::setDataSize(unsigned int dataSize, unsigned int evtSize) {
    // Default 5 is m12, m13, time, sigma_t, evtNum
    totalEventSize = evtSize;
    if(totalEventSize < 5)
        throw GooFit::GeneralError("totalEventSize {} must be 5 or more", totalEventSize);

    if(cachedWaves[0]) {
        for(auto &cachedWave : cachedWaves)
            delete cachedWave;
    }

    numEntries = dataSize;

// Ideally this would not be required, this would be called AFTER setData which will set m_iEventsPerTask
#ifdef GOOFIT_MPI
    int myId, numProcs;
    MPI_Comm_size(MPI_COMM_WORLD, &numProcs);
    MPI_Comm_rank(MPI_COMM_WORLD, &myId);

    int perTask = numEntries / numProcs;

    int *counts = new int[numProcs];

    for(int i = 0; i < numProcs - 1; i++)
        counts[i] = perTask;

    counts[numProcs - 1] = numEntries - perTask * (numProcs - 1);

    setNumPerTask(this, counts[myId]);

    delete[] counts;
#endif

    for(int i = 0; i < 16; i++) {
#ifdef GOOFIT_MPI
        cachedWaves[i] = new thrust::device_vector<WaveHolder_s>(m_iEventsPerTask);
#else
        cachedWaves[i] = new thrust::device_vector<WaveHolder_s>(dataSize);
#endif
        void *dummy = thrust::raw_pointer_cast(cachedWaves[i]->data());
        MEMCPY_TO_SYMBOL(cWaves, &dummy, sizeof(WaveHolder_s *), i * sizeof(WaveHolder_s *), hipMemcpyHostToDevice);
    }

    setForceIntegrals();
}

__host__ fptype TddpPdf::normalize() {
    recursiveSetNormalization(1.0); // Not going to normalize efficiency,
    // so set normalization factor to 1 so it doesn't get multiplied by zero.
    // Copy at this time to ensure that the SpecialWaveCalculators, which need the efficiency,
    // don't get zeroes through multiplying by the normFactor.
    MEMCPY_TO_SYMBOL(
        d_normalizations, host_normalizations, totalNormalizations * sizeof(fptype), 0, hipMemcpyHostToDevice);

    int totalBins = _m12.getNumBins() * _m13.getNumBins();

    if(!dalitzNormRange) {
        gooMalloc((void **)&dalitzNormRange, 6 * sizeof(fptype));

        auto *host_norms = new fptype[6];
        host_norms[0]    = _m12.getLowerLimit();
        host_norms[1]    = _m12.getUpperLimit();
        host_norms[2]    = _m12.getNumBins();
        host_norms[3]    = _m13.getLowerLimit();
        host_norms[4]    = _m13.getUpperLimit();
        host_norms[5]    = _m13.getNumBins();
        MEMCPY(dalitzNormRange, host_norms, 6 * sizeof(fptype), hipMemcpyHostToDevice);
        delete[] host_norms;
    }

    for(unsigned int i = 0; i < decayInfo.resonances.size(); ++i) {
        redoIntegral[i] = forceRedoIntegrals;

        if(!(decayInfo.resonances[i]->parametersChanged()))
            continue;

        redoIntegral[i] = true;
    }

    forceRedoIntegrals = false;

    // Only do this bit if masses or widths have changed.
    thrust::constant_iterator<fptype *> arrayAddress(dalitzNormRange);
    thrust::counting_iterator<int> binIndex(0);

    // NB, SpecialWaveCalculator assumes that fit is unbinned!
    // And it needs to know the total event size, not just observables
    // for this particular PDF component.
    thrust::constant_iterator<fptype *> dataArray(dev_event_array);
    thrust::constant_iterator<int> eventSize(totalEventSize);
    thrust::counting_iterator<int> eventIndex(0);

    static int normCall = 0;
    normCall++;

    for(int i = 0; i < decayInfo.resonances.size(); ++i) {
        // printf("calculate i=%i, res_i=%i\n", i, decayInfo->resonances[i]->getFunctionIndex());
        calculators[i]->setTddpIndex(getFunctionIndex());
        calculators[i]->setResonanceIndex(decayInfo.resonances[i]->getFunctionIndex());
        if(redoIntegral[i]) {
#ifdef GOOFIT_MPI
            thrust::transform(
                thrust::make_zip_iterator(thrust::make_tuple(eventIndex, dataArray, eventSize)),
                thrust::make_zip_iterator(thrust::make_tuple(eventIndex + m_iEventsPerTask, arrayAddress, eventSize)),
                strided_range<thrust::device_vector<WaveHolder_s>::iterator>(
                    cachedWaves[i]->begin(), cachedWaves[i]->end(), 1)
                    .begin(),
                *(calculators[i]));
#else
            thrust::transform(
                thrust::make_zip_iterator(thrust::make_tuple(eventIndex, dataArray, eventSize)),
                thrust::make_zip_iterator(thrust::make_tuple(eventIndex + numEntries, arrayAddress, eventSize)),
                strided_range<thrust::device_vector<WaveHolder_s>::iterator>(
                    cachedWaves[i]->begin(), cachedWaves[i]->end(), 1)
                    .begin(),
                *(calculators[i]));
#endif
            // std::cout << "Integral for resonance " << i << " " << numEntries << " " << totalEventSize << std::endl;
        }

        // Possibly this can be done more efficiently by exploiting symmetry?
        for(int j = 0; j < decayInfo.resonances.size(); ++j) {
            if((!redoIntegral[i]) && (!redoIntegral[j]))
                continue;

            integrators[i][j]->setTddpIndex(getFunctionIndex());
            integrators[i][j]->setResonanceIndex(decayInfo.resonances[i]->getFunctionIndex());
            integrators[i][j]->setEfficiencyIndex(decayInfo.resonances[j]->getFunctionIndex());

            // printf("integrate i=%i j=%i, res_i=%i res_j=%i\n", i, j, decayInfo->resonances[i]->getFunctionIndex (),
            //    decayInfo->resonances[j]->getFunctionIndex ());
            ThreeComplex dummy(0, 0, 0, 0, 0, 0);
            SpecialComplexSum complexSum;
            thrust::constant_iterator<int> effFunc(efficiencyFunction);
            (*(integrals[i][j])) = thrust::transform_reduce(
                thrust::make_zip_iterator(thrust::make_tuple(binIndex, arrayAddress, effFunc)),
                thrust::make_zip_iterator(thrust::make_tuple(binIndex + totalBins, arrayAddress, effFunc)),
                *(integrators[i][j]),
                dummy,
                complexSum);
            /*
            std::cout << "With resonance " << j << ": "
            << thrust::get<0>(*(integrals[i][j])) << " "
            << thrust::get<1>(*(integrals[i][j])) << " "
            << thrust::get<2>(*(integrals[i][j])) << " "
            << thrust::get<3>(*(integrals[i][j])) << " "
            << thrust::get<4>(*(integrals[i][j])) << " "
            << thrust::get<5>(*(integrals[i][j])) << std::endl;
            */
        }
    }

    // End of time-consuming integrals.

    fpcomplex integralA_2(0, 0);
    fpcomplex integralB_2(0, 0);
    fpcomplex integralABs(0, 0);

    for(unsigned int i = 0; i < decayInfo.resonances.size(); ++i) {
        // int param_i = parameters + resonanceOffset + resonanceSize * i;
        fpcomplex amplitude_i(host_parameters[parametersIdx + i * 2 + 4], host_parameters[parametersIdx + i * 2 + 5]);

        for(unsigned int j = 0; j < decayInfo.resonances.size(); ++j) {
            // int param_j = parameters + resonanceOffset + resonanceSize * j;
            fpcomplex amplitude_j(host_parameters[parametersIdx + j * 2 + 4],
                                  -host_parameters[parametersIdx + j * 2 + 5]); // Notice complex conjugation

            integralA_2 += (amplitude_i * amplitude_j
                            * fpcomplex(thrust::get<0>(*(integrals[i][j])), thrust::get<1>(*(integrals[i][j]))));
            integralABs += (amplitude_i * amplitude_j
                            * fpcomplex(thrust::get<2>(*(integrals[i][j])), thrust::get<3>(*(integrals[i][j]))));
            integralB_2 += (amplitude_i * amplitude_j
                            * fpcomplex(thrust::get<4>(*(integrals[i][j])), thrust::get<5>(*(integrals[i][j]))));

            /*
            if (cpuDebug & 1) {
            int idx = i * decayInfo.resonances.size() + j;
            if (0 == host_callnumber) std::cout << "Integral contribution " << i << ", " << j << " " << idx << " : "
                                << amplitude_i << " "
                                << amplitude_j << " ("
                                << real(amplitude_i * amplitude_j * complex<fptype>(thrust::get<0>(*(integrals[i][j])),
            thrust::get<1>(*(integrals[i][j])))) << ", "
                                << imag(amplitude_i * amplitude_j * complex<fptype>(thrust::get<0>(*(integrals[i][j])),
            thrust::get<1>(*(integrals[i][j])))) << ") ("
                                << real(amplitude_i * amplitude_j * complex<fptype>(thrust::get<2>(*(integrals[i][j])),
            thrust::get<3>(*(integrals[i][j])))) << ", "
                                << imag(amplitude_i * amplitude_j * complex<fptype>(thrust::get<2>(*(integrals[i][j])),
            thrust::get<3>(*(integrals[i][j])))) << ") ("
                                << real(amplitude_i * amplitude_j * complex<fptype>(thrust::get<4>(*(integrals[i][j])),
            thrust::get<5>(*(integrals[i][j])))) << ", "
                                << imag(amplitude_i * amplitude_j * complex<fptype>(thrust::get<4>(*(integrals[i][j])),
            thrust::get<5>(*(integrals[i][j])))) << ") "
                                << thrust::get<0>(*(integrals[i][j])) << ", "
                                << thrust::get<1>(*(integrals[i][j])) << ") ("
                                << thrust::get<2>(*(integrals[i][j])) << ", "
                                << thrust::get<3>(*(integrals[i][j])) << ") ("
                                << thrust::get<4>(*(integrals[i][j])) << ", "
                                << thrust::get<5>(*(integrals[i][j])) << ") ("
                                << real(integralA_2) << ", " << imag(integralA_2) << ") "
                                << std::endl;
                 }
                 */
        }
    }

    double dalitzIntegralOne = integralA_2.real(); // Notice that this is already the abs2, so it's real by
                                                   // construction; but the compiler doesn't know that.
    double dalitzIntegralTwo = integralB_2.real();
    double dalitzIntegralThr = integralABs.real();
    double dalitzIntegralFou = integralABs.imag();

    fptype tau     = host_parameters[parametersIdx + 1];
    fptype xmixing = host_parameters[parametersIdx + 2];
    fptype ymixing = host_parameters[parametersIdx + 3];

    fptype ret = resolution->normalization(
        dalitzIntegralOne, dalitzIntegralTwo, dalitzIntegralThr, dalitzIntegralFou, tau, xmixing, ymixing);

    double binSizeFactor = 1;
    binSizeFactor *= ((_m12.getUpperLimit() - _m12.getLowerLimit()) / _m12.getNumBins());
    binSizeFactor *= ((_m13.getUpperLimit() - _m13.getLowerLimit()) / _m13.getNumBins());
    ret *= binSizeFactor;

    host_normalizations[normalIdx + 1] = 1.0 / ret;
    cachedNormalization                = 1.0 / ret;
    // std::cout << "End of TDDP normalization: " << ret << " " << host_normalization[parameters] << " " <<
    // binSizeFactor << std::endl;
    return ret;
}
//#endif

SpecialDalitzIntegrator::SpecialDalitzIntegrator(int pIdx, unsigned int ri, unsigned int rj)
    : resonance_i(ri)
    , resonance_j(rj)
    , parameters(pIdx) {}

__device__ ThreeComplex SpecialDalitzIntegrator::operator()(thrust::tuple<int, fptype *, int> t) const {
    // Bin index, base address [lower, upper,getNumBins]
    // Notice that this is basically MetricTaker::operator (binned) with the special-case knowledge
    // that event size is two, and that the function to call is dev_Tddp_calcIntegrals.

    int globalBinNumber  = thrust::get<0>(t);
    fptype lowerBoundM12 = thrust::get<1>(t)[0];
    fptype upperBoundM12 = thrust::get<1>(t)[1];
    auto numBinsM12      = static_cast<int>(floor(thrust::get<1>(t)[2] + 0.5));
    int binNumberM12     = globalBinNumber % numBinsM12;
    fptype binCenterM12  = upperBoundM12 - lowerBoundM12;
    binCenterM12 /= numBinsM12;
    binCenterM12 *= (binNumberM12 + 0.5);
    binCenterM12 += lowerBoundM12;

    globalBinNumber /= numBinsM12;
    fptype lowerBoundM13 = thrust::get<1>(t)[3];
    fptype upperBoundM13 = thrust::get<1>(t)[4];
    auto numBinsM13      = static_cast<int>(floor(thrust::get<1>(t)[5] + 0.5));
    fptype binCenterM13  = upperBoundM13 - lowerBoundM13;
    binCenterM13 /= numBinsM13;
    binCenterM13 *= (globalBinNumber + 0.5);
    binCenterM13 += lowerBoundM13;

    ParameterContainer pc;

    fptype events[10];

    // increment until we are at tddp index
    while(pc.funcIdx < tddp)
        pc.incrementIndex();

    int id_m12 = pc.getObservable(2);
    int id_m13 = pc.getObservable(3);
    // if (0 == THREADIDX) cuPrintf("%i %i %i %f %f operator\n", thrust::get<0>(t), thrust::get<0>(t) % numBinsM12,
    // globalBinNumber, binCenterM12, binCenterM13);
    ThreeComplex ret = device_Tddp_calcIntegrals(binCenterM12, binCenterM13, resonance_i, resonance_j, pc);

    // fptype fakeEvt[10]; // Need room for many observables in case m12 or m13 were assigned a high index in an
    // event-weighted fit.
    events[0]      = 2;
    events[id_m12] = binCenterM12;
    events[id_m13] = binCenterM13;
    // unsigned int numResonances                               = indices[6];
    // int effFunctionIdx                                       = parIndexFromResIndex(numResonances);
    // if (thrust::get<0>(t) == 19840) {internalDebug1 = BLOCKIDX; internalDebug2 = THREADIDX;}
    // fptype eff = (*(reinterpret_cast<device_function_ptr>(device_function_table[indices[effFunctionIdx]])))(fakeEvt,
    // hipArray, paramIndices + indices[effFunctionIdx + 1]);
    while(pc.funcIdx < thrust::get<2>(t))
        pc.incrementIndex();

    fptype eff = callFunction(events, pc);
    // if (thrust::get<0>(t) == 19840) {
    // internalDebug1 = -1;
    // internalDebug2 = -1;
    // printf("Efficiency: %i %f %f %f %i\n", thrust::get<0>(t), binCenterM12, binCenterM13, eff, effFunctionIdx);
    // printf("Efficiency: %f %f %f %f %f %i %i\n", fakeEvt[0], fakeEvt[1], fakeEvt[2], fakeEvt[3], fakeEvt[4],
    // indices[indices[0] + 2 + 2], indices[indices[0] + 2 + 3]);
    //}

    // Multiplication by eff, not sqrt(eff), is correct:
    // These complex numbers will not be squared when they
    // go into the integrals. They've been squared already,
    // as it were.
    thrust::get<0>(ret) *= eff;
    thrust::get<1>(ret) *= eff;
    thrust::get<2>(ret) *= eff;
    thrust::get<3>(ret) *= eff;
    thrust::get<4>(ret) *= eff;
    thrust::get<5>(ret) *= eff;
    return ret;
}

SpecialWaveCalculator::SpecialWaveCalculator(int pIdx, unsigned int res_idx)
    : resonance_i(res_idx)
    , parameters(pIdx) {}

__device__ WaveHolder_s SpecialWaveCalculator::operator()(thrust::tuple<int, fptype *, int> t) const {
    // Calculates the BW values for a specific resonance.
    // The 'A' wave stores the value at each point, the 'B'
    // at the opposite (reversed) point.

    WaveHolder_s ret;
    ret.ai_real = 0.0;
    ret.ai_imag = 0.0;
    ret.bi_real = 0.0;
    ret.bi_imag = 0.0;

    int evtNum  = thrust::get<0>(t);
    int evtSize = thrust::get<2>(t);
    fptype *evt = thrust::get<1>(t) + (evtNum * evtSize);

    ParameterContainer pc;

    fptype events[10];

    for(int i = 0; i < evtSize; i++)
        events[i] = evt[i];

    // increment until we are at tddp index
    while(pc.funcIdx < tddp)
        pc.incrementIndex();

    int id_m12 = pc.getObservable(2);
    int id_m13 = pc.getObservable(3);

    // Read these values as tddp.
    fptype m12 = events[id_m12];
    fptype m13 = events[id_m13];

    if(!inDalitz(m12, m13, c_motherMass, c_daug1Mass, c_daug2Mass, c_daug3Mass))
        return ret;

    fptype m23 = c_motherMass * c_motherMass + c_daug1Mass * c_daug1Mass + c_daug2Mass * c_daug2Mass
                 + c_daug3Mass * c_daug3Mass - m12 - m13;

    // int parameter_i       = parIndexFromResIndex(resonance_i); // Find position of this resonance relative to TDDP
    // start  unsigned int functn_i = indices[parameter_i + 2];  unsigned int params_i = indices[parameter_i + 3];

    while(pc.funcIdx < resonance_i)
        pc.incrementIndex();

    ParameterContainer tmp = pc;
    fpcomplex ai           = getResonanceAmplitude(m12, m13, m23, tmp);
    tmp                    = pc;
    fpcomplex bi           = getResonanceAmplitude(m13, m12, m23, tmp);

    // printf("Amplitudes %f, %f => (%f %f) (%f %f)\n", m12, m13, ai.real, ai.imag, bi.real, bi.imag);

    ret.ai_real = ai.real();
    ret.ai_imag = ai.imag();
    ret.bi_real = bi.real();
    ret.bi_imag = bi.imag();

    return ret;
}

} // namespace GooFit
