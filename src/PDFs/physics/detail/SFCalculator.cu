#include <goofit/PDFs/physics/detail/SFCalculator.h>

#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/physics/SpinFactors.h>
#include <goofit/PDFs/physics/detail/Dim5.h>

namespace GooFit {

SFCalculator::SFCalculator() = default;

__device__ fpcomplex SFCalculator::operator()(thrust::tuple<int, fptype *, int> t) const {
    int evtNum  = thrust::get<0>(t);
    fptype *evt = thrust::get<1>(t) + (evtNum * thrust::get<2>(t));

    ParameterContainer pc;

    // Increment to DP
    while(pc.funcIdx < dalitzFuncId)
        pc.incrementIndex();

    int id_m12   = pc.getObservable(0);
    int id_m34   = pc.getObservable(1);
    int id_cos12 = pc.getObservable(2);
    int id_cos34 = pc.getObservable(3);
    int id_phi   = pc.getObservable(4);

    fptype m12   = evt[id_m12];
    fptype m34   = evt[id_m34];
    fptype cos12 = evt[id_cos12];
    fptype cos34 = evt[id_cos34];
    fptype phi   = evt[id_phi];

    fptype M  = pc.getConstant(0);
    fptype m1 = pc.getConstant(1);
    fptype m2 = pc.getConstant(2);
    fptype m3 = pc.getConstant(3);
    fptype m4 = pc.getConstant(4);

    fptype vecs[16];
    get4Vecs(vecs, m12, m34, cos12, cos34, phi, M, m1, m2, m3, m4);
    // printf("%i, %i, %f, %f, %f, %f, %f \n",evtNum, thrust::get<2>(t), m12, m34, cos12, cos34, phi );
    // printf("vec%i %f, %f, %f, %f\n",0, vecs[0], vecs[1], vecs[2], vecs[3]);
    // printf("vec%i %f, %f, %f, %f\n",1, vecs[4], vecs[5], vecs[6], vecs[7]);
    // printf("vec%i %f, %f, %f, %f\n",2, vecs[8], vecs[9], vecs[10], vecs[11]);
    // printf("vec%i %f, %f, %f, %f\n",3, vecs[12], vecs[13], vecs[14], vecs[15]);

    // loop until our appropriate spin factor
    while(pc.funcIdx < _spinfactor_i)
        pc.incrementIndex();

    auto func = reinterpret_cast<spin_function_ptr>(d_function_table[pc.funcIdx]);
    fptype sf = (*func)(vecs, pc);
    // printf("SpinFactors %i : %.7g\n",evtNum, sf );
    return {sf, 0.0};
}

} // namespace GooFit
