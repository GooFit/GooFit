#include "hip/hip_runtime.h"
#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/physics/DalitzPlotHelpers.h>
#include <goofit/PDFs/physics/detail/SpecialResonanceIntegrator.h>

namespace GooFit {

__device__ auto device_DalitzPlot_calcIntegrals(fptype m12, fptype m13, int res_i, int res_j, ParameterContainer &pc)
    -> fpcomplex {
    // Calculates BW_i(m12, m13) * BW_j^*(m12, m13).
    // This calculation is in a separate function so
    // it can be cached. Note that this function expects
    // to be called on a normalization grid, not on
    // observed points, that's why it doesn't use
    // cResonances. No need to cache the values at individual
    // grid points - we only care about totals.
    fptype motherMass = c_motherMass; // RO_CACHE(pc.constants[pc.constantIdx + 4]);
    fptype daug1Mass  = c_daug1Mass;  // RO_CACHE(pc.constants[pc.constantIdx + 5]);
    fptype daug2Mass  = c_daug2Mass;  // RO_CACHE(pc.constants[pc.constantIdx + 6]);
    fptype daug3Mass  = c_daug3Mass;  // RO_CACHE(pc.constants[pc.constantIdx + 7]);

    fpcomplex ret;

    if(!inDalitz(m12, m13, motherMass, daug1Mass, daug2Mass, daug3Mass))
        return ret;

    fptype m23
        = motherMass * motherMass + daug1Mass * daug1Mass + daug2Mass * daug2Mass + daug3Mass * daug3Mass - m12 - m13;

    ParameterContainer ipc = pc;
    while(ipc.funcIdx < res_i)
        ipc.incrementIndex();

    ret = getResonanceAmplitude(m12, m13, m23, ipc);

    ParameterContainer jpc = pc;
    while(jpc.funcIdx < res_j)
        jpc.incrementIndex();

    ret *= conj(getResonanceAmplitude(m12, m13, m23, jpc));

    return ret;
}

SpecialResonanceIntegrator::SpecialResonanceIntegrator(int pIdx, unsigned int ri, unsigned int rj)
    : resonance_i(ri)
    , resonance_j(rj)
    , parameters(pIdx) {}

__device__ auto SpecialResonanceIntegrator::operator()(thrust::tuple<int, fptype *, int> t) const -> fpcomplex {
    //(brad): new indexing plan: bin number, function id, parameter id (not required), fptype with actual
    // bins(needed???)
    // Bin index, base address [lower, upper,getNumBins]
    // Notice that this is basically MetricTaker::operator (binned) with the special-case knowledge
    // that event size is two, and that the function to call is dev_DalitzPlot_calcIntegrals.

    int globalBinNumber  = thrust::get<0>(t);
    fptype lowerBoundM12 = thrust::get<1>(t)[0];
    fptype upperBoundM12 = thrust::get<1>(t)[1];
    auto numBinsM12      = static_cast<int>(floor(thrust::get<1>(t)[2] + 0.5));
    int binNumberM12     = globalBinNumber % numBinsM12;
    fptype binCenterM12  = upperBoundM12 - lowerBoundM12;
    binCenterM12 /= numBinsM12;
    binCenterM12 *= (binNumberM12 + 0.5);
    binCenterM12 += lowerBoundM12;

    globalBinNumber /= numBinsM12;
    fptype lowerBoundM13 = thrust::get<1>(t)[3];
    fptype upperBoundM13 = thrust::get<1>(t)[4];
    auto numBinsM13      = static_cast<int>(floor(thrust::get<1>(t)[2] + 0.5));
    fptype binCenterM13  = upperBoundM13 - lowerBoundM13;
    binCenterM13 /= numBinsM13;
    binCenterM13 *= (globalBinNumber + 0.5);
    binCenterM13 += lowerBoundM13;

    ParameterContainer pc;

    fptype events[10];

    while(pc.funcIdx < dalitz_i)
        pc.incrementIndex();

    fpcomplex ret = device_DalitzPlot_calcIntegrals(binCenterM12, binCenterM13, resonance_i, resonance_j, pc);

    // TODO: read id's in in order to set them for the fake event.

    int id_m12 = pc.getObservable(0);
    int id_m13 = pc.getObservable(1);

    // fptype fakeEvt[10]; // Need room for many observables in case m12 or m13 were assigned a high index in an
    // event-weighted fit.
    // fakeEvt[0] = 2;
    // fakeEvt[id_m12] = binCenterM12;
    // fakeEvt[id_m13] = binCenterM13;

    events[0]      = 2;
    events[id_m12] = binCenterM12;
    events[id_m13] = binCenterM13;

    // unsigned int numResonances           = indices[2];
    // int effFunctionIdx                   = parIndexFromResIndex_DP(numResonances);

    // increment until we are on the efficiency function (17)
    int effFunc = thrust::get<2>(t);
    while(pc.funcIdx < effFunc)
        pc.incrementIndex();

    fptype eff = callFunction(events, pc);

    // Multiplication by eff, not sqrt(eff), is correct:
    // These complex numbers will not be squared when they
    // go into the integrals. They've been squared already,
    // as it were.
    ret *= eff;
    // printf("ret %f %f %f %f %f\n",binCenterM12, binCenterM13, ret.real, ret.imag, eff );
    return ret;
}

} // namespace GooFit
