#include "hip/hip_runtime.h"
#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/physics/DalitzPlotHelpers.h>
#include <goofit/PDFs/physics/detail/SpecialSqDpResonanceCalculator.h>
#include <goofit/PDFs/physics/resonances/Resonance.h>
#include <goofit/PDFs/physics/Amp3BodySqDP.h>

namespace GooFit {

SpecialSqDpResonanceCalculator::SpecialSqDpResonanceCalculator(int pIdx, unsigned int res_idx)
    : resonance_i(res_idx)
    , parameters(pIdx) {}

__device__ auto SpecialSqDpResonanceCalculator::operator()(thrust::tuple<int, fptype *, int> t) const -> fpcomplex {
    // Calculates the BW values for a specific resonance.
    fpcomplex ret(0.,0.);
    int evtNum  = thrust::get<0>(t);
    fptype *evt = thrust::get<1>(t) + (evtNum * thrust::get<2>(t));

    ParameterContainer pc;

    while(pc.funcIdx < dalitz_i)
        pc.incrementIndex();

    int id_mprime = pc.getObservable(0);
    int id_thetaprime = pc.getObservable(1);

    fptype mprime = RO_CACHE(evt[id_mprime]);
    fptype thetaprime = RO_CACHE(evt[id_thetaprime]);

    // if(thetaprime>0.5)
    //     thetaprime = 1.0-thetaprime;

    if(!inSqDalitz(mprime, thetaprime)){
        return ret;
    }

    // mprime, m23 and thetaprime stand for the squared invariant masses.
    // Now fixed.
    fptype m12 = calc_m12(mprime,c_motherMass,c_daug1Mass,c_daug2Mass,c_daug3Mass);
    fptype m13 = calc_m13(m12,cos(thetaprime*M_PI), c_motherMass,c_daug1Mass,c_daug2Mass,c_daug3Mass);
    fptype s12 = m12*m12;
    fptype s13 = m13*m13;
    fptype s23 = c_motherMass * c_motherMass + c_daug1Mass * c_daug1Mass + c_daug2Mass * c_daug2Mass
                 + c_daug3Mass * c_daug3Mass - s12 - s13;
  

     if(!inDalitz2(s13, s23,c_motherMass,c_daug1Mass,c_daug2Mass,c_daug3Mass ))
        return ret;

    while(pc.funcIdx < resonance_i)
        pc.incrementIndex();

    ret = getResonanceAmplitude(s13, s23 , s12 ,  pc);
    
    return ret/sqrt(norm);
    // return ret;

}

} // namespace GooFit
