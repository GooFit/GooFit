#include "hip/hip_runtime.h"
#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/physics/DalitzPlotHelpers.h>
#include <goofit/PDFs/physics/detail/SpecialSqDpResonanceIntegrator.h>
#include <goofit/PDFs/physics/Amp3BodySqDP.h>

namespace GooFit {

__device__ auto device_SqDalitzPlot_calcIntegrals(fptype mprime, fptype thetaprime, int res_i, int res_j, ParameterContainer &pc)
    -> fpcomplex {
    // Calculates BW_i(mprime, thetaprime) * BW_j^*(mprime, thetaprime).
    // This calculation is in a separate function so
    // it can be cached. Note that this function expects
    // to be called on a normalization grid, not on
    // observed points, that's why it doesn't use
    // cResonances. No need to cache the values at individual
    // grid points - we only care about totals.

//    printf("pc = %d \n",pc.funcIdx); == 1

    fpcomplex ret(0.,0.);

    if(!inSqDalitz(mprime, thetaprime))
        return ret;


    fptype m12 = calc_m12(mprime,c_motherMass,c_daug1Mass,c_daug2Mass,c_daug3Mass);
    fptype m13 = calc_m13(m12,cos(thetaprime*M_PI), c_motherMass,c_daug1Mass,c_daug2Mass,c_daug3Mass);
    fptype s12 = m12*m12;
    fptype s13 = m13*m13;
    fptype s23 = c_motherMass * c_motherMass + c_daug1Mass * c_daug1Mass + c_daug2Mass * c_daug2Mass
                 + c_daug3Mass * c_daug3Mass - s12 - s13;

    

    

    if(!inDalitz2(s13, s23,c_motherMass,c_daug1Mass,c_daug2Mass,c_daug3Mass ))
        return ret;

    ParameterContainer ipc = pc;
    while(ipc.funcIdx < res_i)
        ipc.incrementIndex();

    ret = getResonanceAmplitude(s13, s23 , s12 , ipc);

    

    ParameterContainer jpc = pc;
    while(jpc.funcIdx < res_j)
        jpc.incrementIndex();
    

    ret *= conj(getResonanceAmplitude(s13, s23 , s12 , jpc));

    return ret;
}

SpecialSqDpResonanceIntegrator::SpecialSqDpResonanceIntegrator(int pIdx, unsigned int ri, unsigned int rj)
    : resonance_i(ri)
    , resonance_j(rj)
    , parameters(pIdx) {}

__device__ auto SpecialSqDpResonanceIntegrator::operator()(thrust::tuple<int, fptype *, int, int> t) const -> fpcomplex {
   
    int evtNum  = thrust::get<0>(t);
    fptype *evt = thrust::get<1>(t) + (evtNum * thrust::get<2>(t));

    ParameterContainer pc;

    fptype events[20];

    while(pc.funcIdx < dalitz_i)
        pc.incrementIndex();

    int id_mprime = pc.getObservable(0);
    int id_thetaprime = pc.getObservable(1);

    fptype mprime = RO_CACHE(evt[id_mprime]);
    fptype thetaprime = RO_CACHE(evt[id_thetaprime]);

    if(thetaprime>0.5)
        thetaprime = 1.0-thetaprime;

    if(!inSqDalitz(mprime, thetaprime))
        return fpcomplex(0.,0.);

 
    fpcomplex ret = device_SqDalitzPlot_calcIntegrals(mprime, thetaprime, resonance_i, resonance_j, pc);

    // fptype m12 = calc_m12(mprime,c_motherMass,c_daug1Mass,c_daug2Mass,c_daug3Mass);
    // fptype m13 = calc_m13(m12,cos(thetaprime*M_PI), c_motherMass,c_daug1Mass,c_daug2Mass,c_daug3Mass);
    // fptype s12 = m12*m12;
    // fptype s13 = m13*m13;
    // fptype s23 = c_motherMass * c_motherMass + c_daug1Mass * c_daug1Mass + c_daug2Mass * c_daug2Mass
    //              + c_daug3Mass * c_daug3Mass - s12 - s13;


    events[0] = 2;
    events[id_mprime] = mprime;
    events[id_thetaprime] = thetaprime;

    int effFunc = thrust::get<3>(t);

    while(pc.funcIdx < effFunc)
         pc.incrementIndex();

    fptype eff = callFunction(events, pc);

    // printf("mp=%f  th=%f oi pj=%d  eff=%f \n",mprime,thetaprime,pc.funcIdx, eff);
    

    // Multiplication by eff, not sqrt(eff), is correct:
    // These complex numbers will not be squared when they
    // go into the integrals. They've been squared already,
    // as it were.
    fptype jacobian = calc_SqDp_Jacobian(mprime, thetaprime, c_motherMass, c_daug1Mass, c_daug2Mass, c_daug3Mass);
    if(m_no_eff)
        return ret*jacobian;
    else
        return ret*eff;

   
    // printf("ret %f %f %f %f %f\n",binCenterMPrime, binCenterThetaPrime, ret.real, ret.imag, eff );
    // return ret;
}

} // namespace GooFit
