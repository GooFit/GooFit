#include "hip/hip_runtime.h"
#include <goofit/PDFs/physics/lineshapes/NonRes.h>

#include <goofit/PDFs/ParameterContainer.h>

#include "Common.h"

namespace GooFit {

__device__ fpcomplex nonres_DP(fptype Mpair, fptype m1, fptype m2, ParameterContainer &pc) {
    fptype meson_radius  = pc.getConstant(0);
    unsigned int orbital = pc.getConstant(1);

    fptype mumsRecoMass2 = Mpair * Mpair;

    fptype mpsq       = (m1 + m2) * (m1 + m2);
    fptype mmsq       = (m1 - m2) * (m1 - m2);
    fptype num        = (mumsRecoMass2 - mpsq) * (mumsRecoMass2 - mmsq);
    fptype pABSq      = num / (4 * mumsRecoMass2);
    fptype formfactor = sqrt(BL2(pABSq * meson_radius * meson_radius, orbital));

    pc.incrementIndex(1, 0, 2, 0, 1);
    return fpcomplex(1., 0.) * formfactor;
}

__device__ resonance_function_ptr ptr_to_NONRES_DP = nonres_DP;

Lineshapes::NonRes::NonRes(
    std::string name, Variable mass, Variable width, unsigned int L, unsigned int Mpair, FF FormFac, fptype radius)
    : Lineshape(name, L, Mpair, FormFac, radius) {
    // TODO: Clean up signature

    registerConstant(radius);
    registerConstant(L);

    registerFunction("ptr_to_NONRES_DP", ptr_to_NONRES_DP);

    initialize();
}

} // namespace GooFit
