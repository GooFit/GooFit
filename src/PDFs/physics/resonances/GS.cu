#include "hip/hip_runtime.h"
#include <goofit/PDFs/physics/resonances/GS.h>

#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/physics/lineshapes/Lineshape.h>
#include <goofit/PDFs/physics/resonances/Resonance.h>

namespace GooFit {

__device__ fptype hFun(double s, double daug2Mass, double daug3Mass) {
    // Last helper function
    const fptype _pi = 3.14159265359;
    double sm        = daug2Mass + daug3Mass;
    double sqrt_s    = sqrt(s);
    double k_s       = twoBodyCMmom(s, daug2Mass, daug3Mass);

    return ((2 / _pi) * (k_s / sqrt_s) * log((sqrt_s + 2 * k_s) / (sm)));
}

__device__ fptype dh_dsFun(double s, double daug2Mass, double daug3Mass) {
    // Yet another helper function
    const fptype _pi = 3.14159265359;
    double k_s       = twoBodyCMmom(s, daug2Mass, daug3Mass);

    return hFun(s, daug2Mass, daug3Mass) * (1.0 / (8.0 * POW2(k_s)) - 1.0 / (2.0 * s)) + 1.0 / (2.0 * _pi * s);
}

__device__ fptype dFun(double s, double daug2Mass, double daug3Mass) {
    // Helper function used in Gronau-Sakurai
    const fptype _pi = 3.14159265359;
    double sm        = daug2Mass + daug3Mass;
    double sm24      = sm * sm / 4.0;
    double m         = sqrt(s);
    double k_m2      = twoBodyCMmom(s, daug2Mass, daug3Mass);

    return 3.0 / _pi * sm24 / POW2(k_m2) * log((m + 2 * k_m2) / sm) + m / (2 * _pi * k_m2)
           - sm24 * m / (_pi * POW3(k_m2));
}

__device__ fptype fsFun(double s, double m2, double gam, double daug2Mass, double daug3Mass) {
    // Another G-S helper function

    double k_s   = twoBodyCMmom(s, daug2Mass, daug3Mass);
    double k_Am2 = twoBodyCMmom(m2, daug2Mass, daug3Mass);

    double f = gam * m2 / POW3(k_Am2);
    f *= (POW2(k_s) * (hFun(s, daug2Mass, daug3Mass) - hFun(m2, daug2Mass, daug3Mass))
          + (m2 - s) * POW2(k_Am2) * dh_dsFun(m2, daug2Mass, daug3Mass));

    return f;
}

__device__ fpcomplex gouSak(fptype m12, fptype m13, fptype m23, ParameterContainer &pc) {
    unsigned int spin         = pc.getConstant(0);
    unsigned int cyclic_index = pc.getConstant(1);

    fptype resmass  = pc.getParameter(0);
    fptype reswidth = pc.getParameter(1);

    fptype rMassSq  = (PAIR_12 == cyclic_index ? m12 : (PAIR_13 == cyclic_index ? m13 : m23));
    fptype frFactor = 1;

    resmass *= resmass;
    // Calculate momentum of the two daughters in the resonance rest frame; note symmetry under interchange (dm1 <->
    // dm2).
    fptype measureDaughterMoms = twoBodyCMmom(rMassSq,
                                              (PAIR_23 == cyclic_index ? c_daug2Mass : c_daug1Mass),
                                              (PAIR_12 == cyclic_index ? c_daug2Mass : c_daug3Mass));
    fptype nominalDaughterMoms = twoBodyCMmom(resmass,
                                              (PAIR_23 == cyclic_index ? c_daug2Mass : c_daug1Mass),
                                              (PAIR_12 == cyclic_index ? c_daug2Mass : c_daug3Mass));

    if(0 != spin) {
        frFactor = dampingFactorSquare(nominalDaughterMoms, spin, c_meson_radius);
        frFactor /= dampingFactorSquare(measureDaughterMoms, spin, c_meson_radius);
    }

    // Implement Gou-Sak:

    fptype D = (1.0 + dFun(resmass, c_daug2Mass, c_daug3Mass) * reswidth / sqrt(resmass));
    fptype E = resmass - rMassSq + fsFun(rMassSq, resmass, reswidth, c_daug2Mass, c_daug3Mass);
    fptype F = sqrt(resmass) * reswidth * pow(measureDaughterMoms / nominalDaughterMoms, 2.0 * spin + 1) * frFactor;

    D /= (E * E + F * F);
    fpcomplex retur(D * E, D * F); // Dropping F_D=1
    retur *= sqrt(frFactor);
    retur *= spinFactor(spin, c_motherMass, c_daug1Mass, c_daug2Mass, c_daug3Mass, m12, m13, m23, cyclic_index);

    pc.incrementIndex(1, 2, 2, 0, 1);

    return retur;
}

__device__ resonance_function_ptr ptr_to_GOUSAK = gouSak;

namespace Resonances {

GS::GS(std::string name,
       Variable ar,
       Variable ai,
       Variable mass,
       Variable width,
       unsigned int sp,
       unsigned int cyc,
       bool sym)
    : ResonancePdf("GS", name, ar, ai) {
    registerParameter(mass);
    registerParameter(width);

    registerConstant(sp);
    registerConstant(cyc);

    registerFunction("ptr_to_GOUSAK", ptr_to_GOUSAK);
}

} // namespace Resonances
} // namespace GooFit
