#include "hip/hip_runtime.h"
#include <goofit/PDFs/physics/resonances/GS.h>

#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/physics/lineshapes/Lineshape.h>
#include <goofit/PDFs/physics/resonances/Resonance.h>

namespace GooFit {

__device__ auto gouSak(fptype m12, fptype m13, fptype m23, ParameterContainer &pc) -> fpcomplex {
    unsigned int spin         = pc.getConstant(0);
    unsigned int cyclic_index = pc.getConstant(1);
    bool norm                 = pc.getConstant(2);

    fptype resmass  = pc.getParameter(0);
    fptype reswidth = pc.getParameter(1);

    fptype rMassSq = (PAIR_12 == cyclic_index ? m12 : (PAIR_13 == cyclic_index ? m13 : m23));
    fptype bachelorMass
        = (PAIR_12 == cyclic_index ? c_daug3Mass : (PAIR_13 == cyclic_index ? c_daug2Mass : c_daug1Mass));
    fptype frFactor  = 1;
    fptype frFactorD = 1;

    resmass *= resmass;
    // Calculate momentum of the two daughters in the resonance rest frame; note symmetry under interchange (dm1 <->
    // dm2).
    fptype measureDaughterMoms = twoBodyCMmom(rMassSq,
                                              (PAIR_23 == cyclic_index ? c_daug2Mass : c_daug1Mass),
                                              (PAIR_12 == cyclic_index ? c_daug2Mass : c_daug3Mass));
    fptype nominalDaughterMoms = twoBodyCMmom(resmass,
                                              (PAIR_23 == cyclic_index ? c_daug2Mass : c_daug1Mass),
                                              (PAIR_12 == cyclic_index ? c_daug2Mass : c_daug3Mass));

    fptype measureDaughterMomsMother;
    fptype nominalDaughterMomsMother;

    if(norm) {
        // Mother momentum for normalized Blatt-Weisskopf form factors calculated in the resonance rest frame
        measureDaughterMomsMother = twoBodyCMMothermom(rMassSq, c_motherMass, bachelorMass);
        nominalDaughterMomsMother = twoBodyCMMothermom(resmass, c_motherMass, bachelorMass);
    } else {
        // Mother momentum for unnormalized Blatt-Weisskopf form factors calculated in mother rest frame
        measureDaughterMomsMother = twoBodyCMmom(c_motherMass * c_motherMass, sqrt(rMassSq), bachelorMass);
    }

    if(0 != spin) {
        if(norm) {
            frFactor = dampingFactorSquareNorm(nominalDaughterMoms, spin, c_meson_radius);
            frFactor /= dampingFactorSquareNorm(measureDaughterMoms, spin, c_meson_radius);

            frFactorD = dampingFactorSquareNorm(nominalDaughterMomsMother, spin, c_mother_meson_radius);
            frFactorD /= dampingFactorSquareNorm(measureDaughterMomsMother, spin, c_mother_meson_radius);
        }
        // unnormalized form factors
        else {
            frFactor  = dampingFactorSquare(measureDaughterMoms, spin, c_meson_radius);
            frFactorD = dampingFactorSquare(measureDaughterMomsMother, spin, c_mother_meson_radius);
        }
    }

    // Implement Gou-Sak:

    fptype D = (1.0 + dFun(resmass, c_daug2Mass, c_daug3Mass) * reswidth / sqrt(resmass));
    fptype E = resmass - rMassSq + fsFun(rMassSq, resmass, reswidth, c_daug2Mass, c_daug3Mass);
    fptype F = sqrt(resmass) * reswidth * pow(measureDaughterMoms / nominalDaughterMoms, 2.0 * spin + 1) * frFactor;

    D /= (E * E + F * F);
    fpcomplex retur(D * E, D * F); // Dropping F_D=1
    retur *= sqrt(frFactor);
    retur *= sqrt(frFactorD);
    retur *= spinFactor(spin, c_motherMass, c_daug1Mass, c_daug2Mass, c_daug3Mass, m12, m13, m23, cyclic_index);

    pc.incrementIndex(1, 2, 3, 0, 1);

    return retur;
}

__device__ resonance_function_ptr ptr_to_GOUSAK = gouSak;

namespace Resonances {

GS::GS(std::string name,
       Variable ar,
       Variable ai,
       Variable mass,
       Variable width,
       unsigned int sp,
       unsigned int cyc,
       bool norm,
       bool sym)
    : ResonancePdf("GS", name, ar, ai) {
    registerParameter(mass);
    registerParameter(width);

    registerConstant(sp);
    registerConstant(cyc);
    registerConstant(norm);

    registerFunction("ptr_to_GOUSAK", ptr_to_GOUSAK);
}

} // namespace Resonances
} // namespace GooFit
