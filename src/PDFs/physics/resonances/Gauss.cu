#include "hip/hip_runtime.h"
#include <goofit/PDFs/physics/resonances/Gauss.h>

#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/physics/lineshapes/Lineshape.h>
#include <goofit/PDFs/physics/resonances/Resonance.h>
#include <goofit/PDFs/physics/resonances/ResonanceUtils.h>
namespace GooFit {

__device__ auto gaussian(fptype m12, fptype m13, fptype m23, ParameterContainer &pc) -> fpcomplex {
    // indices[1] is unused constant index, for consistency with other function types.
    fptype resmass            = pc.getParameter(0);
    fptype reswidth           = pc.getParameter(1);
    unsigned int doSwap       = pc.getConstant(1);
    unsigned int cyclic_index = pc.getConstant(0);

    fpcomplex ret(0., 0.);

    // Notice sqrt - this function uses mass, not mass-squared like the other resonance types.
    for(int i = 0; i < 1 + doSwap; i++) {
        fptype massToUse = sqrt(PAIR_12 == cyclic_index ? m12 : (PAIR_13 == cyclic_index ? m13 : m23));
        massToUse -= resmass;
        massToUse /= reswidth;
        massToUse *= massToUse;
        fptype gauss = exp(-0.5 * massToUse);
        // Ignore factor 1/sqrt(2pi).
        gauss /= reswidth;
        ret += fpcomplex(gauss, 0.);

        if(doSwap) {
            fptype swpmass = m12;
            m12            = m13;
            m13            = swpmass;
        }
    }

    pc.incrementIndex(1, 2, 2, 0, 1);

    return ret;
}

__device__ resonance_function_ptr ptr_to_GAUSSIAN = gaussian;

namespace Resonances {

// Constructor for regular BW,Gounaris-Sakurai,LASS
Gauss::Gauss(std::string name, Variable ar, Variable ai, Variable mass, Variable width, unsigned int cyc, bool symmDP)
    : ResonancePdf("Gauss", name, ar, ai) {
    // Making room for index of decay-related constants. Assumption:
    // These are mother mass and three daughter masses in that order.
    // They will be registered by the object that uses this resonance,
    // which will tell this object where to find them by calling setConstantIndex.
    registerParameter(mass);
    registerParameter(width);

    registerConstant(cyc);
    registerConstant(symmDP);

    registerFunction("ptr_to_GAUSSIAN", ptr_to_GAUSSIAN);
}

} // namespace Resonances
} // namespace GooFit
