#include "hip/hip_runtime.h"
#include <goofit/PDFs/physics/resonances/NonRes3k.h>

#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/physics/resonances/Resonance.h>
#include <goofit/PDFs/physics/resonances/ResonanceUtils.h>
namespace GooFit {

__device__ auto nonres3k(fptype m13, fptype m23, fptype m12, ParameterContainer &pc) -> fpcomplex {
    fptype alpha = pc.getParameter(0);
    fptype beta = pc.getParameter(1);

    pc.incrementIndex(1, 2, 0, 0, 1);

    // if(m23 > 4.0 && m13> 4.0 ) {
        fptype exp13= exp(-alpha * m13);
        fptype exp23 = exp(-alpha * m23);
        fpcomplex amp_m13 = exp13*fpcomplex(cos(-beta*m13),sin(-beta*m13));
        fpcomplex amp_m23 = exp23*fpcomplex(cos(-beta*m23),sin(-beta*m23));
        return amp_m13+amp_m23;
    // } else {
    //    return fpcomplex(0.00001, 0.0);
    // }
}

__device__ resonance_function_ptr ptr_to_NONRES3k = nonres3k;

namespace Resonances {

NonRes3k::NonRes3k(std::string name, Variable ar, Variable ai, Variable alpha, Variable beta)
    : ResonancePdf("NonRes3k", name, ar, ai) {
    registerParameter(alpha);
    registerParameter(beta);
    registerFunction("ptr_to_NONRES3k", ptr_to_NONRES3k);
}

} // namespace Resonances
} // namespace GooFit
