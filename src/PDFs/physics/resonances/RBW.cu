#include "hip/hip_runtime.h"
#include <goofit/PDFs/physics/resonances/RBW.h>

#include <goofit/PDFs/ParameterContainer.h>
#include <goofit/PDFs/physics/lineshapes/Lineshape.h>
#include <goofit/PDFs/physics/resonances/Resonance.h>

namespace GooFit {

template <int I>
__device__ auto plainBW(fptype m12, fptype m13, fptype m23, ParameterContainer &pc) -> fpcomplex {
    unsigned int spin         = pc.getConstant(0);
    unsigned int cyclic_index = pc.getConstant(1);
    bool norm                 = pc.getConstant(2);

    fptype resmass  = pc.getParameter(0);
    fptype reswidth = pc.getParameter(1);

    fpcomplex result{0.0, 0.0};
    fptype resmass2 = POW2(resmass);

#pragma unroll
    for(size_t i = 0; i < I; i++) {
        fptype rMassSq    = (PAIR_12 == cyclic_index ? m12 : (PAIR_13 == cyclic_index ? m13 : m23));
        fptype mass_daug1 = PAIR_23 == cyclic_index ? c_daug2Mass : c_daug1Mass;
        fptype mass_daug2 = PAIR_12 == cyclic_index ? c_daug2Mass : c_daug3Mass;
        fptype bachelorMass
            = (PAIR_12 == cyclic_index ? c_daug3Mass : (PAIR_13 == cyclic_index ? c_daug2Mass : c_daug1Mass));

        fptype frFactor       = 1;
        fptype frFactorMother = 1;

        // Calculate momentum of the two daughters in the resonance rest frame
        // Note symmetry under interchange (dm1 <-> dm2)

        fptype measureDaughterMoms = twoBodyCMmom(rMassSq, mass_daug1, mass_daug2);
        fptype nominalDaughterMoms = twoBodyCMmom(resmass2, mass_daug1, mass_daug2);

        fptype measureDaughterMomsMother;
        fptype nominalDaughterMomsMother;

        if(norm) {
            // Mother momentum for normalized Blatt-Weisskopf form factors calculated in the resonance rest frame
            measureDaughterMomsMother = twoBodyCMMothermom(rMassSq, c_motherMass, bachelorMass);
            nominalDaughterMomsMother = twoBodyCMMothermom(resmass2, c_motherMass, bachelorMass);
        } else {
            // Mother momentum for unnormalized Blatt-Weisskopf form factors calculated in mother rest frame
            measureDaughterMomsMother = twoBodyCMmom(c_motherMass * c_motherMass, sqrt(rMassSq), bachelorMass);
        }
        if(0 != spin) {
            // D0 meson has same spin than resonance
            if(norm) {
                // normalized form factors
                frFactor = dampingFactorSquareNorm(nominalDaughterMoms, spin, c_meson_radius)
                           / dampingFactorSquareNorm(measureDaughterMoms, spin, c_meson_radius);

                frFactorMother = dampingFactorSquareNorm(nominalDaughterMomsMother, spin, c_mother_meson_radius)
                                 / dampingFactorSquareNorm(measureDaughterMomsMother, spin, c_mother_meson_radius);
            }
            // unnormalized form factors
            else {
                frFactor       = dampingFactorSquare(measureDaughterMoms, spin, c_meson_radius);
                frFactorMother = dampingFactorSquare(measureDaughterMomsMother, spin, c_mother_meson_radius);
            }
        }

        // RBW evaluation
        fptype A = (resmass2 - rMassSq);
        fptype B = resmass2 * reswidth * pow(measureDaughterMoms / nominalDaughterMoms, 2.0 * spin + 1) * frFactor
                   / sqrt(rMassSq);
        fptype C = 1.0 / (POW2(A) + POW2(B));

        fpcomplex ret(A * C, B * C); // Dropping F_D=1

        ret *= sqrt(frFactor);
        ret *= sqrt(frFactorMother);
        ret *= spinFactor(spin, c_motherMass, c_daug1Mass, c_daug2Mass, c_daug3Mass, m12, m13, m23, cyclic_index);

        result += ret;

        if(I > 1) {
            cyclic_index = cyclic_index + 1 % 3;
        }
    }
    pc.incrementIndex(1, 2, 3, 0, 1);
    return result;
}

__device__ resonance_function_ptr ptr_to_RBW     = plainBW<1>;
__device__ resonance_function_ptr ptr_to_RBW_Sym = plainBW<2>;

namespace Resonances {

RBW::RBW(std::string name,
         Variable ar,
         Variable ai,
         Variable mass,
         Variable width,
         unsigned int sp,
         unsigned int cyc,
         bool norm,
         bool sym)
    : ResonancePdf("RBW", name, ar, ai) {
    registerParameter(mass);
    registerParameter(width);

    registerConstant(sp);
    registerConstant(cyc);

    registerConstant(norm);

    if(sym)
        registerFunction("ptr_to_RBW_Sym", ptr_to_RBW_Sym);
    else
        registerFunction("ptr_to_RBW", ptr_to_RBW);
}

} // namespace Resonances

} // namespace GooFit
