// A small test program to debug a CUDA issue on Macs

#include <hip/hip_runtime.h>
#include <iostream>

__device__ double testDouble;
double hostDouble = 5.1;

int main (int argc, char** argv) {
 hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(testDouble), (void*) &hostDouble, sizeof(double));
 std::cout << hipGetErrorString(err) << std::endl;

 return 0;
}
