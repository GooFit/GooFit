#include <gtest/gtest.h>

#include <goofit/BinnedDataSet.h>
#include <goofit/FitManager.h>
#include <goofit/PDFs/basic/ExpPdf.h>
#include <goofit/PDFs/combine/ProdPdf.h>

#include <goofit/Variable.h>

#include <iostream>
#include <sys/time.h>
#include <sys/times.h>

#include <random>

using namespace GooFit;

TEST(BinnedFit, SimpleFit) {
    // Random number generation
    std::mt19937 gen(137);
    std::exponential_distribution<> d(1.5);

    // Independent variable.
    Observable xvar{"xvar", 0, 10};

    // Data set
    BinnedDataSet data(xvar);

    // Generate toy events.
    for(int i = 0; i < 100000; ++i) {
        double val = d(gen);
        if(val < 10) {
            xvar.setValue(val);
            data.addEvent();
        }
    }

    // Fit parameter
    Variable alpha{"alpha", -2, 0.1, -10, 10};

    // GooPdf object
    ExpPdf exppdf{"exppdf", xvar, alpha};
    exppdf.setData(&data);

    FitManager fitter{&exppdf};
    fitter.fit();

    EXPECT_TRUE(fitter);
    EXPECT_LT(alpha.getError(), .01);
    EXPECT_NEAR(-1.5, alpha.getValue(), alpha.getError() * 3);
}

TEST(BinnedFit, DualFit) {
    // Random number generation
    std::mt19937 gen(137);
    std::exponential_distribution<> dx(1.5);
    std::exponential_distribution<> dy(.75);

    // Independent variable.
    Observable xvar{"xvar", 0, 10};
    Observable yvar{"yvar", 0, 10};

    // Data set
    BinnedDataSet data{{xvar, yvar}};

    // Generate toy events.
    for(int i = 0; i < 200000; ++i) {
        double xval = dx(gen);
        double yval = dy(gen);
        if(xval < 10 && yval < 10) {
            xvar.setValue(xval);
            yvar.setValue(yval);
            data.addEvent();
        }
    }

    // Fit parameter
    Variable xalpha{"xalpha", -2, 0.1, -10, 10};
    // Fit parameter
    Variable yalpha{"yalpha", -2, 0.1, -10, 10};

    // GooPdf object
    ExpPdf xpdf{"xpdf", xvar, xalpha};
    ExpPdf ypdf{"ypdf", yvar, yalpha};
    ProdPdf totalpdf{"totalpdf", {&xpdf, &ypdf}};
    totalpdf.setData(&data);

    FitManager fitter{&totalpdf};
    fitter.fit();

    EXPECT_TRUE(fitter);
    EXPECT_LT(xalpha.getError(), .1);
    EXPECT_LT(yalpha.getError(), .1);
    EXPECT_NEAR(-1.5, xalpha.getValue(), xalpha.getError() * 3);
    EXPECT_NEAR(-.75, yalpha.getValue(), yalpha.getError() * 3);
}

TEST(BinnedFit, DifferentFitterVariable) {
    // Random number generation
    std::mt19937 gen(137);
    std::exponential_distribution<> dx(1.5);
    std::exponential_distribution<> dy(.75);

    // Independent variable.
    Observable xvar{"xvar", 0, 10};
    Observable yvar{"yvar", 0, 10};

    // Data set
    BinnedDataSet data{{xvar, yvar}, "Some name"};

    // Generate toy events.
    for(int i = 0; i < 200000; ++i) {
        double xval = dx(gen);
        double yval = dy(gen);
        if(xval < 10 && yval < 10) {
            xvar.setValue(xval);
            yvar.setValue(yval);
            data.addEvent();
        }
    }

    // Fit parameter
    Variable xalpha{"xalpha", -2, 0.1, -10, 10};
    // Fit parameter
    Variable yalpha{"yalpha", -2, 0.1, -10, 10};

    // GooPdf object
    ExpPdf ypdf{"ypdf", yvar, yalpha};
    ExpPdf xpdf{"xpdf", xvar, xalpha};
    ProdPdf totalpdf{"totalpdf", {&xpdf, &ypdf}};
    totalpdf.setData(&data);

    FitManager fitter{&totalpdf};
    fitter.fit();

    EXPECT_TRUE(fitter);
    EXPECT_LT(xalpha.getError(), .1);
    EXPECT_LT(yalpha.getError(), .1);
    EXPECT_NEAR(-1.5, xalpha.getValue(), xalpha.getError() * 3);
    EXPECT_NEAR(-.75, yalpha.getValue(), yalpha.getError() * 3);
}
